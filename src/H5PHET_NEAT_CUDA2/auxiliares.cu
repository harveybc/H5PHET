#include "hip/hip_runtime.h"
/** Funciones gen�ticas auxiliares (busqueda, diagn�stico, etc...) - C file
	funciones que no modifican el genoma ni la poblaci�n (estad�sticas, b�squedas).
*/

#ifndef PARAMS_H_INCLUDED
#include "params.h"
#define PARAMS_H_INCLUDED
#endif
#include "auxiliares.h"



float correlac(float* Vc,TConfig* conf)
// retorna el coeficiente de correlaci�n de dos arreglos de datos x e y de tama�o numDatos.
// la matriz de correlaciones correlacM debe inicializarse cn valores >10 la primera vez.
// par�metros:
//      x,y = vectores a comparar
//      xm,ym = media de x y media de y, se deben proporcionar por optimizaci�n de pre-calculo de medias
//      numDatos = tama�o de los vectores
{
    int i=0;
    float xm=0;
    float ym=0;
    float sum0=0;
    float sum1=0;
    float sum2=0;
    // calcula las medias
    for (i=0;i<conf->numDatos;i++)
    {
        xm+=conf->dataGTDf[i][conf->headerGTD.numEntradas];
        ym+=Vc[i];
    }
    xm/=conf->numDatos;
    ym/=conf->numDatos;
    // calcula sum0(0,nD,(Xi-Xm)*(Yi-Ym)),sum1(0,nD,sqr(Xi-Xm)) y sum2(0,nD,sqr(Yi-Ym)
    for (i=0;i<conf->numDatos;i++)
    {
        sum0+=((conf->dataGTDf[i][conf->headerGTD.numEntradas]-xm)*(Vc[i]-ym));
        sum1+=((conf->dataGTDf[i][conf->headerGTD.numEntradas]-xm)*(conf->dataGTDf[i][conf->headerGTD.numEntradas]-xm));
        sum2+=((Vc[i]-ym)*(Vc[i]-ym));
    }
    // retorna la correlaci�n
    return(sum0/(sqrt(sum1)*sqrt(sum2)));
}


void genSSNhdr1(int indexpob, TConfig * conf)
{
    conf->headerSNN[indexpob].fileID[0] = 'S';
    conf->headerSNN[indexpob].fileID[1] = 'N';
    conf->headerSNN[indexpob].fileID[2] = 'N';
    conf->headerSNN[indexpob].version = 1;
    conf->headerSNN[indexpob].usarSigned = conf->tSigma>=1000? 1 : 0;
    conf->headerSNN[indexpob].tamRegistros = conf->useFloat==0? 8 : 4;
    conf->headerSNN[indexpob].numEntradas = conf->numEntradas;
    conf->headerSNN[indexpob].numSalidas = conf->numSalidas;
    conf->headerSNN[indexpob].numBias = conf->numBias;
    conf->headerSNN[indexpob].numHiddens = conf->pob[indexpob].totalNodos-(conf->numEntradas+conf->numSalidas+conf->numBias);
    conf->headerSNN[indexpob].numConex = conf->pob[indexpob].totalConexiones;
    conf->headerSNN[indexpob].sigmaFactor = (double)conf->A;
    conf->headerSNN[indexpob].actThreshold = (double) conf->Fthreshold;
    conf->headerSNN[indexpob].lastFitness = (double) conf->pob[indexpob].fitness; // usado para programac distribuida
}

float randL(TConfig* conf)
{
    float valor;
    valor=(*conf->punteroRand).valor;
    conf->punteroRand=(tRandList*)(*conf->punteroRand).next;
    return(valor);
}

int genOrdenEvalF1g(int indexpob, TConfig* conf)
{
    int j,temp;
    int maxN=0;
    char* valCalculado;
    // inicializa los campos del header de SNN
    conf->headerSNN[indexpob].fileID[0] = 'S';
    conf->headerSNN[indexpob].fileID[1] = 'N';
    conf->headerSNN[indexpob].fileID[2] = 'N';
    conf->headerSNN[indexpob].version = 1;
    conf->headerSNN[indexpob].usarSigned = conf->tSigma>=1000? 1 : 0;
    conf->headerSNN[indexpob].tamRegistros = conf->useFloat==0? 8 : 4;
    conf->headerSNN[indexpob].numEntradas = conf->numEntradas;
    conf->headerSNN[indexpob].numSalidas = conf->numSalidas;
    conf->headerSNN[indexpob].numBias = conf->numBias;
    conf->headerSNN[indexpob].numHiddens = conf->pob[indexpob].totalNodos-(conf->numEntradas+conf->numSalidas+conf->numBias);
    conf->headerSNN[indexpob].numConex = conf->pob[indexpob].totalConexiones;
    conf->headerSNN[indexpob].sigmaFactor = (double)conf->A;
    conf->headerSNN[indexpob].actThreshold = (double) conf->Fthreshold;
    conf->headerSNN[indexpob].lastFitness = (double) conf->pob[indexpob].fitness; // usado para programac distribuida
    //calcula el n�mero de nodos
    maxN=conf->headerSNN[indexpob].numEntradas+conf->headerSNN[indexpob].numBias+conf->headerSNN[indexpob].numSalidas+conf->headerSNN[indexpob].numHiddens;
    // libera memoria de listaConexData[indexpob]
    free(conf->listaConexData[indexpob]);
    // reserva memoria para listaConexData[indexpob],tam=numConex[indexpob]*sizeof(tConexDataF)
    conf->listaConexData[indexpob]=(tConexDataF*)malloc((conf->pob[indexpob].totalConexiones)*sizeof(tConexDataF));
//printf("%d,",indexpob);
    if (!conf->listaConexData[indexpob])
    {
        printf("\nError 66.7 en genOrdenEvalF1g llamando a malloc()");
        return(0);
    }
//TODO: FALTA VER PORQUE NO SE PUDO LIBERAR?    // libera memoria de ordenEval
// TODO: HACER FUNCION EVALGENOM() para evaluar solo1
// TODO: guardar despu�s de evaalgenom para verificar y setear el leastfitness.
    valCalculado=(char*)malloc(maxN*sizeof(char));
    if (!valCalculado)
    {
        printf("\nError 61.1 en genOrdenEvalF1g llamando a malloc()");
        return(0);
    }
    // inicializa en 0 valcalculado para todos excepto bias
    for (j=0; j<maxN; j++)
    {
        valCalculado[j]=0;
    }
    // inicializa valcalculado en 1 para bias
    for (j=0; j<conf->headerSNN[indexpob].numBias; j++)
    {
        valCalculado[conf->headerSNN[indexpob].numEntradas+j]=1;
    }
    for (j=0; j<conf->pob[indexpob].totalConexiones; j++)
    {
        // genera listaConexData[indexpob][j]=tConexDataF[j]
        conf->listaConexData[indexpob][j].conexIn=conf->pob[indexpob].conex[j].indexIn;
        conf->listaConexData[indexpob][j].conexOut=conf->pob[indexpob].conex[j].indexOut;
        conf->listaConexData[indexpob][j].enabled=conf->pob[indexpob].conex[j].enabled;
        conf->listaConexData[indexpob][j].peso=conf->pob[indexpob].conex[j].peso;
    }
    // inicializa el contador global de posici�n
    conf->tamOrdenEval[indexpob]=0;
    temp=conf->headerSNN[indexpob].numEntradas+conf->headerSNN[indexpob].numBias;
    // para cada salida
    for(j=0; j<conf->headerSNN[indexpob].numSalidas; j++)
    {
        // llama recurOrganicer(indexNodoOut, tConexDataF* listaConexData, int* ordenEval)
        recurOrganicer(temp+j, conf->headerSNN[indexpob] ,conf->listaConexData[indexpob],  conf->ordenEval[indexpob], valCalculado,&(conf->tamOrdenEval[indexpob]));
    }
    // coloca tamListaConexPost[indexpob] en el valor obtenido del ordenamiento.
    conf->tamListaConexPost[indexpob]=ordenarListaConexF(conf->headerSNN[indexpob], conf->listaConexData[indexpob] ,conf->ordenEval[indexpob],conf->tamOrdenEval[indexpob]);
    //libera valcalculado
    free(valCalculado);
    return(1);
}

int ordenarListaConexF(hdrSNNv1 headerSNN, tConexDataF* listaConexData, int* ordenEval, int tamOrdenEval)
// usando el arreglo ordenEval, organiza la lista de conexiones para que queden primero las
// que est�n de �ltimas en el arreglo ordenEval
{
    int i,j;
    int tmp=0;
    tConexDataF tempConex;
    // para cada valor de ordenEval leyendolo desde el final.
    for (i=(tamOrdenEval-1); i>=0; i--)
    {
        // para todas las conexiones j
        for (j=0; j<headerSNN.numConex; j++)
        {
            // si conexOut = ordenEval[i]
            if (listaConexData[j].conexOut==ordenEval[i])
                // si est� enabled
                if (listaConexData[j].enabled)
                {
                    // hace swap
                    tempConex=listaConexData[j];
                    listaConexData[j]=listaConexData[tmp];
                    listaConexData[tmp]=tempConex;
                    // incrementa contador
                    tmp++;
                }
        }
    }
    return(tmp);
}

int recurOrganicer(int indexNodoOut, hdrSNNv1 headerSNN ,tConexDataF* listaConexData, int* ordenEval, char* valCalculado, int* cont)
// usa arreglo conf->valCalculado[] para marcar todos como no calculados al principio.
{
    int i;
    // si est� marcado como calculado, retorna 1
    if (valCalculado[indexNodoOut]) return(1);
    // marca indexNodoOut como Calculado
    valCalculado[indexNodoOut]=1;
    // adiciona indexNodoOut a ordenEval[*cont]
    ordenEval[*cont]=indexNodoOut;
    // incrementa *cont
    *cont=*cont+1;
    // busca entre las conex las que tengan como conexOut a indexNodoOut
    for (i=0; i<headerSNN.numConex; i++)
    {
        // si tiene conexOut=indexNodoOut
        if (listaConexData[i].conexOut==indexNodoOut)
            // si es enabled y no es bias (SOLO PUEDE HABER UN NODO BIAS)
            if ((listaConexData[i].enabled)&&(indexNodoOut!=headerSNN.numEntradas))
            {
                // si el nodo conexIn  est� calculado,
                if (!valCalculado[listaConexData[i].conexIn])
                    // llama a recurOrganicer
                    recurOrganicer(listaConexData[i].conexIn, headerSNN ,listaConexData, ordenEval, valCalculado, cont);
            }
    }
    //retorna 1
    return(1);
}

int genOrdenEvalF(int numGenomas, hdrSNNv1* headerSNN, tConexDataF** listaConexData, int** ordenEval, int* tamOrdenEval, TConfig* conf)
// genera ordenEval[], orden de evaluaci�n de los nodos de cada genoma (inverso a recursivo empezando de salida)
{
    int i;
    //determina el n�mero m�ximo de nodos
    for (i=0; i<numGenomas; i++)
    {
        genOrdenEvalF1g(i,conf);
    }
    // libera el vector temporal valCalculado
    return(1);
}

void imprimirSeleccion(TConfig* conf)
{
// Imprime los areglos: numGenomasPorEspecia, actNumGenomasPorEspecie, y la matriz listaordenFitness
// SOLO se puede usar dentro de seleccionCrossover debido a que al final se liberan los punteros.
    int i=0;
    int j=0;
    // imprime numGenomasPorEspecie
    fclose(conf->logFile);
    conf->logFile=fopen(conf->fileNameLog,"a+");
    fprintf(conf->logFile,"<br>\nnumGenomasPorEspecie:    ");
    for (i=0; i<conf->numEspecies; i++)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>%u=%u,",i,conf->numGenomasPorEspecie[i]);
    }
    // imprime actNumGenomasPorEspecie
    fclose(conf->logFile);
    conf->logFile=fopen(conf->fileNameLog,"a+");
    fprintf(conf->logFile,"<br>\nactNumGenomasPorEspecie: ");
    for (i=0; i<conf->numEspecies; i++)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>%u=%u,",i,conf->actNumGenomasPorEspecie[i]);
    }
    // imprime listaOrdenFitness
    fclose(conf->logFile);
    conf->logFile=fopen(conf->fileNameLog,"a+");
    fprintf(conf->logFile,"<br>\nlistaOrdenFitness:");
    for (j=0; j<conf->numEspecies; j++)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nE%2i = ",j);
        for (i=0; i<conf->actNumGenomasPorEspecie[j]; i++)
        {
            fclose(conf->logFile);
            conf->logFile=fopen(conf->fileNameLog,"a+");
            fprintf(conf->logFile,"<br>%u,",conf->listaOrdenFitness[j][i]);
        }
    }
    fclose(conf->logFile);
    conf->logFile=fopen(conf->fileNameLog,"a+");
    fprintf(conf->logFile,"<br>\n");
}

float fError(float valor, float salida)
{
    // retorna -1 si hay error.
    // funci�n de error para un genoma
    //TODO: falta implementar cuando rango de salidas es (-1,1)
    //TODO: PROBANDO Cambiado de versi�n 0.66
    float tmpA=2.435;
    float a=fabs(valor-salida)/2;
    //fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>Valor=%3.3f  salida=%3.3f\n",valor,salida);
    //calculo del error a partir de la diferencia de salidas (Diferente para cada problema)
    //a = a<=0.1? a: a<=0.2?sqrt(a):sqrt(sqrt(a));
    // A = -2.30258509299/((100.0-MinPorcentGanancia)/100);

    // a=1-exp(-10*a*a);

    // Error para forex con error de tres lineas -0.5,0,0.5:
    if ((valor>0.5)&&(salida<0.5)) tmpA*=7;
    if ((valor<0.5)&&(salida>0.5)) tmpA*=7;
    if ((valor>0)&&(salida<0)) tmpA*=7;
    if ((valor<0)&&(salida>0)) tmpA*=7;
    if ((valor>-0.5)&&(salida<-0.5)) tmpA*=7;
    if ((valor<-0.5)&&(salida>-0.5)) tmpA*=7;
    a=1-exp(-tmpA*a*a);



    //Comprueba que el error est� entre 0 y 1 YA que en selecci�n Crossover se parte de esto, valores superiores causan errores.

    //return( a>=0 ? a<=2 ? a : -1 : -1 );
    return(a);
}

unsigned buscarMejorFitness(TConfig* conf)  // OPTIMIZADA
{
//Busca el mejor fitness entre todos los elementos de la conf->poblaci�n
//Se debe llamar al final de laFuncion evaluarPob evaluarPob
//Resetea los representantes a valor -1 para indicar que no se han asignado, luego se recalculan
//Retorna el indexpob del genoma con el mejor fitness entre toda la conf->poblaci�n -1 si hay error
//Par�metros:	sPob = tama�o de la conf->poblaci�n.
    float temp=-100000;//fitness inicial =-100000
    unsigned i;
    unsigned j=0;
//TODO: probando qutando calculo de representantes y limitando la b�squeda a los representantes actuales.
    /*	for (i=0;i<conf->numEspecies;i++){	 //resetea todos los valores de los representantes a -1 para indicar que no se han asignado.
    		conf->representantes[i]=UINT_MAX;
    	}
    	for (i=0;i<conf->sizePob;i++){ //calcula los representantes
    		if (conf->representantes[conf->pob[i].especie]!=UINT_MAX){//si representantes de especie de i es diferente de -1 entonces: verifica si i es mejor que su representante.
    			if(conf->pob[conf->representantes[conf->pob[i].especie]].fitness<conf->pob[i].fitness){
    				conf->representantes[conf->pob[i].especie]=i;
    			}
    		}
    		else{ //si representantes de especie de i es -1 entonces: hace el representante de especie =i.
    			conf->representantes[conf->pob[i].especie]=i;
    		}
    	}
    */
    //busca el mejor entre todos y los backups de representantes.
    for (i=0; i<(conf->sizePob+conf->numEspecies); i++)
    {
        if(conf->pob[i].fitness>temp)
        {
            temp=conf->pob[i].fitness;
            j=i;
        }
    }
    return(j);
}

unsigned contarGPEsp(unsigned indEspecie, TConfig* conf)  // OPTIMIZADA
{
    // retorna el n�mero de genomas en la poblaci�n que pertenecen a la especie indEspecie
    unsigned i;
    unsigned temp=0;
    for (i=0; i<conf->sizePob; i++)
        if(conf->pob[i].especie==indEspecie)
            temp++;
    return temp;
}

unsigned verificarMejor(TConfig* conf)  // OPTIMIZADA, //TODO: se debe quitar cuando ya no se necesite
{
//Verificar mejor
//retorna 0 si error, se debe usar despu�s de cada especiaci�n de poblaci�n
    float a;
    a=conf->pob[buscarMejorFitness(conf)].fitness;
    if (conf->antMejor>a)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError en Funcion verificarMejor()");
        return(0);
    }
    else
        conf->antMejor=a;
//TODO: probando quitar esta comprobaci�n debido a que puede dar un falso positivo si muta el representante y todav�a no se actualiza el guardado en especiacion
    /*	for (i=0;i<conf->sizePob;i++){
    		if (conf->pob[i].fitness>conf->pob[conf->representantes[conf->pob[i].especie]].fitness){
    			fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 243 en Funcion VerificarMejor() comparando el fitness de cada genoma con el del representante de su esp.");
    			return(0);
    		}

    	}
    */
    return(1);
}

void swap(unsigned  int* a, unsigned  int* b)  // OPTIMIZADA
{
//unsigned  intercambia los valores apuntados por dos punteros.
    unsigned temp=*a;
    *a=*b;
    *b=temp;
}

unsigned cargarGenoma(unsigned indexpob, char *filename, TConfig* conf)  // OPTIMIZADA
{
//Lee un genoma desde un archivo y sobreescribe con �l un genoma de conf->pob , la memoria necesaria para el arreglo de nodos y conexiones se gestiona desde esta Funcion.
//Prerequisito:  debe haberse reservado memoria para el genoma en conf->pob[indexpob] (se hace con primeraGen())
//El formato de entrada es(sin separadores): Genoma, Genoma.nodo, genoma.conex las longitudes a escribir
//de cada estructura se basan en el tama�o de Genoma, GenNodoF, GenconexF y en los valores Genoma.totalNodos
//y Genoma.totalConexiones
//Par�metros:	indexpob = indice del arreglo de genomas conf->pob que se va a reemplazar por el le�do
//				filename = path y nombre de archivo del que se leer� el genoma
//Retorna 0 si hay error, 1 ok
    FILE *fileIn;
    size_t leidos=0;
    GenNodoF* temp;
    //Abre el archivo para lectura
    if ((fileIn=fopen(filename,"rb"))==NULL)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 49 en funcion guardarMejorFitness(%u,%s) llamando a fopen(%s,\"br\")\n",indexpob,filename,filename);
        return(0);
    }
    if (feof(fileIn)!=0)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 243-1 en Funcion cargarGenoma() llamando a la Funcion feof() antes de leer archivo, archivo vac�o.\n");
        fclose(fileIn);
        return(0);
    }
    leidos=fread(&(conf->pob[indexpob]),sizeof(Genoma),1,fileIn); //lee Genoma
    if (feof(fileIn)!=0)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 243-2 en Funcion cargarGenoma() llamando a la Funcion feof() despu�s de leer genoma, genoma incompleto, faltan nodos.\n");
        fclose(fileIn);
        return(0);
    }
    if ((temp=(GenNodoF *)malloc(sizeof(GenNodoF)*conf->pob[indexpob].totalNodos))==NULL)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 50 en funcion guardarMejorFitness(%u,%s) llamando a malloc(%u)\n",indexpob,filename,(unsigned  int)(conf->pob[indexpob].totalNodos*sizeof(GenNodoF)));
        fclose(fileIn);
        return(0);
    }
    //if (conf->pob[indexpob].nodo!=NULL)
    free(conf->pob[indexpob].nodo);
    if (temp!=NULL)
        conf->pob[indexpob].nodo=temp;
    leidos=fread(conf->pob[indexpob].nodo,1,conf->pob[indexpob].totalNodos*sizeof(GenNodoF),fileIn); //lee nodos NO es necesario leer listas de punteros ya que se actualizan entes de evaluaci�n.
    if (!feof(fileIn))
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 243-3 en Funcion cargarGenoma() llamando a la Funcion feof() despu�s de leer genoma, genoma incompleto, faltan conexiones.\n");
        fclose(fileIn);
        return(0);
    }
    if (conf->pob[indexpob].conex!=NULL) free((void *)conf->pob[indexpob].conex);
    if ((conf->pob[indexpob].conex=(GenConexF *)malloc(sizeof(GenConexF)*conf->pob[indexpob].totalConexiones))==NULL)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 51 en funcion guardarMejorFitness(%u,%s) llamando a malloc(%u)\n",indexpob,filename,(unsigned  int)(conf->pob[indexpob].totalConexiones*sizeof(GenConexF)));
        fclose(fileIn);
        return(0);
    }
    leidos=fread(conf->pob[indexpob].conex,1,sizeof(GenConexF)*conf->pob[indexpob].totalConexiones,fileIn); //lee conexiones
    if (leidos<conf->pob[indexpob].totalConexiones*sizeof(GenConexF))
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 243-4 en Funcion cargarGenoma() llamando a la Funcion feof() despu�s de leer genoma, genoma incompleto, conexiones incompletas.\n");
        fclose(fileIn);
        return(0);
    }
    //Cierra el archivo
    if (fclose(fileIn)!=0)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 50 en funcion cargarGenoma(%u,%s) llamando a fclose(fileout))\n",indexpob,filename);
        return(0);
    }
    return(1);
}

unsigned guardarRepresentantesNNP(char *filename, TConfig* conf)  // OPTIMIZADA
// Guarda los representantes en un archivo en formato NNPv1
// retorna 0 si hubo error, 1 si ok.
{
    hdrNNPv1 headerNNP;
    hdrSNNv1 header;
    int i,j,k,result;
    float tmpPesoF;
    double tmpPesoD;
    char tmpChar;
    FILE* fileNNP;
    unsigned escritos; // para verificar el n�mero de elementos escritos en el archivo
    unsigned* reps; //para ordenar por orden descendente de fitness

    // si el n�mero de especies es <1 retorna error;
    if (conf->numEspecies<1)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 51 en funcion guardarRepresentantesNPP(), numEspecies==0\n");
        return(0);
    }
    // abre el archivo para escritura
    if ((fileNNP=fopen(filename,"wb"))==NULL)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 51.36 en funcion guardarRepresentantesNPP() llamando a fopen()\n");
        return(0);
    }
    // inicializa los campos del header de SNN
    headerNNP.fileID[0] = 'N';
    headerNNP.fileID[1] = 'N';
    headerNNP.fileID[2] = 'P';
    headerNNP.version = 1;
    headerNNP.numGenomas = conf->numEspecies;
    // guarda el encabezado en el archivo NNP
    escritos=fwrite(&headerNNP, sizeof(hdrNNPv1),1,fileNNP);
    if (escritos<1)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 51.37 en guardarRepresentantesNNP() llamando a fwrite()\n");
        fclose(fileNNP);
        return(0);
    }
    // reserva memoria para reps
    reps = (unsigned*) malloc(conf->numEspecies*sizeof(unsigned));
    if (reps==NULL)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 51.38 en guardarRepresentantesNNP() llamando a malloc()\n");
        return(0);
    }
    // llena reps con los index de los representantes
    for (k=0; k<conf->numEspecies; k++)
    {
        reps[k]=conf->representantes[k];
    }
    // ordena por fitness los representantes en reps
    for (k=1; k<conf->numEspecies; k++)
    {
        for (i=0; i<(conf->numEspecies-k); i++)
        {
            if (conf->pob[reps[i]].fitness<conf->pob[reps[i+1]].fitness)
            {
                //intercambia elementos i y i+1 del arreglo reps
                j=reps[i];
                reps[i]=reps[i+1];
                reps[i+1]=j;
            }
        }
    }

    // para cada representante
    for (k=0; k<conf->numEspecies; k++)
    {
        // genera encabezado SNNv1
        header.fileID[0] = 'S';
        header.fileID[1] = 'N';
        header.fileID[2] = 'N';
        header.version = 1;
        header.usarSigned = conf->tSigma>=1000? 1 : 0;
        header.tamRegistros = conf->useFloat==0? 8 : 4;
        header.numEntradas = conf->numEntradas;
        header.numSalidas = conf->numSalidas;
        header.numBias = conf->numBias;
        header.numHiddens = conf->pob[reps[k]].totalNodos-(conf->numEntradas+conf->numSalidas+conf->numBias);
        header.numConex = conf->pob[reps[k]].totalConexiones;
        header.sigmaFactor = (double)conf->A;
        header.actThreshold = (double) conf->Fthreshold;
        header.lastFitness = (double) conf->pob[reps[k]].fitness;
        // guarda encabezado SNN
        escritos=fwrite(&header, sizeof(hdrSNNv1),1,fileNNP);
        if (escritos<1)
        {
            fclose(conf->logFile);
            conf->logFile=fopen(conf->fileNameLog,"a+");
            fprintf(conf->logFile,"<br>\nError 51.38 en guardarRepresentantesNNP() llamando a fwrite()\n");
            fclose(fileNNP);
            return(0);
        }
        // Escribe los arreglos en orden: int conexIn[numConex],conexOut[numConex],enabled[],double peso[numConex]
        // para conexIn
        for (i=0; i<conf->pob[reps[k]].totalConexiones; i++)
        {
            // escribe los 3 arrays con los datos de la conex
            escritos=fwrite(&(conf->pob[reps[k]].conex[i].indexIn), sizeof(unsigned),1,fileNNP);
        }
        // para conexOut
        for (i=0; i<conf->pob[reps[k]].totalConexiones; i++)
        {
            // escribe los 4 arrays con los datos de la conex
            escritos=fwrite(&(conf->pob[reps[k]].conex[i].indexOut), sizeof(unsigned),1,fileNNP);
        }
        // para enabled
        for (i=0; i<conf->pob[reps[k]].totalConexiones; i++)
        {
            tmpChar = conf->pob[reps[k]].conex[i].enabled;
            escritos = fwrite(&tmpChar, sizeof(char),1,fileNNP);
        }
        // tama�o de lista de orden de evaluaci�n de conexiones
        escritos = fwrite(&(conf->tamListaConexPost[reps[k]]), sizeof(int),1,fileNNP);
        if (escritos==0)
        {
            printf("ERROR");
            exit(0);
        }
        // genera y escribe lista de evaluaci�n de conexiones
        for (i=0; i<conf->tamListaConexPost[reps[k]]; i++)
        {
            for(j=0; j<conf->pob[reps[k]].totalConexiones; j++)
            {
                if ((conf->listaConexData[reps[k]][i].conexIn==conf->pob[reps[k]].conex[j].indexIn)&&(conf->listaConexData[reps[k]][i].conexOut==conf->pob[reps[k]].conex[j].indexOut))
                    result=j;
            }
            escritos = fwrite(&result, sizeof(int),1,fileNNP);
        }
        // escribe  listaConexData[indexpob][tamListaConexPost[indexpob]] que es la lista de orden de evaluaci�n de conex.
        //escritos = fwrite(conf->listaConexData[conf->representantes[i]], conf->tamListaConexPost[conf->representantes[i]]*sizeof(int),1,fileNNP);
        // para peso
        for (i=0; i<conf->pob[reps[k]].totalConexiones; i++)
        {
            if (header.tamRegistros==4)//para float
            {
                //hace cast para sacar double a partir de float.
                tmpPesoF=(float)conf->pob[reps[k]].conex[i].peso;
                // escribe los 3 arrays con los datos de la conex
                escritos=fwrite(&tmpPesoF, sizeof(float),1,fileNNP);
            }
            else //para double
            {
                //hace cast para sacar double a partir de float.
                tmpPesoD=(double)conf->pob[reps[k]].conex[i].peso;
                // escribe los 3 arrays con los datos de la conex
                escritos=fwrite(&tmpPesoD, sizeof(double),1,fileNNP);
            }
        }
    }
    free(reps);
    //cierra el archivo NNP
    if (fclose(fileNNP)!=0)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 50 en funcion guardarRepresentantesNNP() llamando a fclose()");
        return(0);
    }
    return(1);
}

long unsigned calcularMemoriaUsada(unsigned sPob, TConfig* conf)  //OPTIMIZADA - //TODO Adicionar consumo de arreglos de punteros a nodos, conexiones, enabled y peso en estructura GenNodoF
{
//Calcula la memoria usada por los arreglos del programa en bytes
//Par�metros:	sPob
//Retorna el n�mero de bytes usados por el programa en memoria.
    unsigned  long int acum;
    unsigned i;
    acum=(unsigned  int)(sPob*sizeof(Genoma));//Calcula bytes usados por genomas
    for (i=0; i<sPob; i++) //calcula bytes usados por arreglos de nodos y conexiones
        acum=acum+(unsigned  int)((conf->pob[i].totalNodos*sizeof(GenNodoF))+(conf->pob[i].totalConexiones*sizeof(GenConexF)));
    acum=(unsigned  long int)(acum+(conf->numEspecies*sizeof(unsigned  int)));//lista de conf->representantes
    acum=(unsigned  long int)(acum+(conf->contInnovNodo*sizeof(TListaInnov)));//lista de innovaciones de nodos
    acum=(unsigned  long int)(acum+(conf->contInnovNodo*sizeof(TNodoOut)));//y sus nodos de salida
    acum=(unsigned  long int)(acum+(conf->contInnovNodo*sizeof(TListaInnov)));
    acum=(unsigned  long int)(acum+(conf->contInnovCon*sizeof(TNodoOut)));//lista de innovaciones de conexiones
    acum=(unsigned  long int)(acum+(conf->numEspecies*sizeof(unsigned  int)));//lista de conf->contGeneracSinMejora
    acum=(unsigned  long int)(acum+(conf->numConservacion*sizeof(unsigned  int)));//lista de conservacion

    return(acum);
}

void calcularD(TConfig* conf)  //TODO, falta calcular params paa aproximaciones
{
    // para el tsigma escogido, calcula los par�metros (coeficientes ) de la funci�n fSigma seleccionada.
    if (conf->tSigma==0)  //con y(-1)=0, y(0)=1 y y(0.5)=0.5
    {
        conf->A = 10;
        conf->D = 1.04;
        conf->F = -0.2;
    }
    if (conf->tSigma==4)  //con y(-1)=0, y(0)=1 y y(0.5)=0.5
    {
        conf->A = 2.435;
        conf->D = 1.09626166044;
        conf->F = -0.09626166044;
    }
    if (conf->tSigma==1003)  //con y(-1)=0, y(0)=1 y y(0.5)=0.5
    {
        conf->A = 2.435;
        conf->D = 1.09626166044;
        conf->F = -0.09626166044;
    }

}

float fSigma(float fX, unsigned param, float fD, TConfig* conf)  //NO OPTIMA//TODO: usar INLINE si es posible
{
//fSigma, retorna un float corespondiente a la funcion de activaci�n seleccionada con param para una entrada X
//TODO: verificar rangos de entrada y salida de cada aprox sigma
    float y=0;
    //Param =	0 = sigma(0,1),		y = 1 / (1 + exp (- D * x)) -> corregido
    //		1 = sigma aprox (0,1)	y = 0.5 + x * (1 - abs(x) / 2), y=0 si x<=-1, y=1 si x>=1
    //		2 = elliot (0,1)	y = (x / 2) / (1 + |x|) + 0.5
    //		3 = binario (0,1)	y = x>=0 ? 1:0
    //		4 = gauss(0,1),		y = exp(- x * x)
    //		1000 = tanh(-1,1),		y = 2 / (1 + exp(-2 * x)) - 1
    //		1000 = elliot,(-1,1)	y = x / (1 + |x|)
    //		1002 = binario (-1,1) 	y = x=0 ? 1: -1;
    //		1003 = gaussAn (-1,1)	y= 2*exp(- x*x))-1
    if (param==0)
        return ((conf->D / (1.0 + exp (- conf->A * (fX+0.5))))+conf->F);
    if (param==1)
    {
        if (fX<=-1)
        {
            return (0);
        }
        if (fX>=1.0)
        {
            return (1.0);
        }
        if ((fX!=-1.0)&&(fX<1.0))
            y = 0.5 + fX * (1.0 - (fabs(fX) / 2.0));
        return (y);
    }
    if (param==2)
        return ((fX / 2.0) / (1.0 + fabs(fX)) + 0.5);
    if (param==3)
    {
        return (fX>=0 ? fX>=1 ? 0: cos(3.14*fX): fX<=-1 ? 0: fX+1) ;
        //return (fX>=0 ? fX>=1 ? 0: -fX+1: fX<=-1 ? 0: fX+1) ;  TODO: probando funci�n de activaci�n sinusoidal, podr�a usarse con valores fasoriales de entradas
    }
    if (param==4)
        //return (conf->D*exp(-conf->A*(fX * fX))+conf->F);
        return(exp(-conf->A*(fX * fX)));
    //return (exp(-3*(fX * fX)));
    if (param==1000)
        return (2.0 / (1.0 + exp(fX*-2.0)) - 1.0);

    if (param==1001)
        return (fX / (1.0 + fabs(fX)));
    if (param==1002)
    {
        return (fX>=0 ? 1.0:-1.0);
    }
    if (param==1003)
        return 2*(exp(-conf->A*(fX * fX)))-1;
    return(0);
}

void imprimirGenoma(unsigned index, TConfig* conf)  //OPTIMIZADA - //TODO: generar bmp del genoma
{
//imprime la principal informaci�n del genoma incluyendo nodos y conexiones
    unsigned i;
    printf("\n ordenEval\n");
    /*   for (i=0;i<conf->contO[index];i++)
       {
           printf("\nOrdenEval[%2d] =  %2d",i,conf->ordenEval[index][i]);
       }*/
    fclose(conf->logFile);
    conf->logFile=fopen(conf->fileNameLog,"a+");
    fprintf(conf->logFile,"<br>\nN�mero de nodos = %u, MaxInnovNumNodo = %u\nN�mero de conexiones = %u, MaxInnovNumConex = %u\nListado de nodos Nindex=function,valor,threshold\n",conf->pob[index].totalNodos,conf->pob[index].maxInnovNumNodo,conf->pob[index].totalConexiones,conf->pob[index].maxInnovNumConex);
    for (i=0; i<conf->pob[index].totalNodos; i++)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>N%u=%u,%1.1f,%1.1f,",i,conf->pob[index].nodo[i].nodeFunction,conf->pob[index].nodo[i].valor,conf->pob[index].nodo[i].thNodo);
    }
    fclose(conf->logFile);
    conf->logFile=fopen(conf->fileNameLog,"a+");
    fprintf(conf->logFile,"<br>\nListado de conexiones Cindex = IndexIn,IndexOut,peso,enabled\n");
    for (i=0; i<conf->pob[index].totalConexiones; i++)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>C%u=%u,%u,%1.1f,%u,",i,conf->pob[index].conex[i].indexIn,conf->pob[index].conex[i].indexOut,conf->pob[index].conex[i].peso,conf->pob[index].conex[i].enabled);
    }
    /*fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nEstructura de punteros. Nindex=indexHijo0,indexHijo1....\n");
    for (i=0; i<conf->pob[index].totalNodos; i++)
    {
        fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br> N%u(%u)=",i,conf->pob[index].nodo[i].contHijos);
        for (j=0; j<conf->pob[index].nodo[i].contHijos; j++)
        {
            fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>%u,",conf->pob[index].nodo[i].conexHijo[j]->indexIn);
        }
    }*/
    fclose(conf->logFile);
    conf->logFile=fopen(conf->fileNameLog,"a+");
    fprintf(conf->logFile,"<br>\n");
}

unsigned verificarGenoma(unsigned index, TConfig* conf)  // OPTIMIZADA - //TODO: quitarla cuando no haya errores
{
// Verificar Genoma, verifica que los m�ximos innov num y m�ximos numNodo y conex correspondan con los que se encuentran en el genoma
// retorna 0 si hay error, 1 si OK
    unsigned i;
    // verifica maxInnovnumNodo
    for (i=0; i<conf->pob[index].totalNodos; i++)
    {
        if (i>0)
            if (conf->pob[index].nodo[i].innovNum==0)
            {
                fclose(conf->logFile);
                conf->logFile=fopen(conf->fileNameLog,"a+");
                fprintf(conf->logFile,"<br>\nError en pob[%u] verificando innovNum de gen nodo[%u]!=0  es igual a 0 el maxInnovNumNodo",index,i);
                return(0);
            }
        if (conf->pob[index].nodo[i].innovNum>conf->pob[index].maxInnovNumNodo)
        {
            fclose(conf->logFile);
            conf->logFile=fopen(conf->fileNameLog,"a+");
            fprintf(conf->logFile,"<br>\nError en pob[%u] verificando no corresponde el maxInnovNumNodo[%u]",index,i);
            return(0);
        }
    }
    // verifica maxInnovNumConex
    for (i=0; i<conf->pob[index].totalConexiones; i++)
    {
        /*	if (i>0)
        		if (conf->pob[index].nodo[i].innovNum==0){
        			fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError en nodo %u verificando innovNum de gen nodo!=0  es igual a 0 el maxInnovNumNodo\n",index);
        			return(0);
        		} */
        if (conf->pob[index].conex[i].innovNum>conf->pob[index].maxInnovNumConex)
        {
            fclose(conf->logFile);
            conf->logFile=fopen(conf->fileNameLog,"a+");
            fprintf(conf->logFile,"<br>\nError en pob[%u] verificando no corresponde el maxInnovNumConex[%u]",index,i);
            return(0);
        }
    }

    return(1);
}

unsigned verificarPob(TConfig* conf)  //OPTIMIZADA - //TODO: quitarla cuando no haya errores
{
//VerificarPob: verifica los innovnum de todos los genomas de la pob
//retorna 0 si hay error, 1 si OK
    unsigned i;
    for (i=0; i<conf->sizePob; i++)
    {
        if (verificarGenoma(i,conf)==0)
            return(0);
    }
    fclose(conf->logFile);
    conf->logFile=fopen(conf->fileNameLog,"a+");
    fprintf(conf->logFile,"<br>Verificado ");
    return(1);
}

unsigned buscarInnovNodo(unsigned indexpob, unsigned innovNum, TConfig* conf)  // OPTIMIZADA
{
//Retorna el index del arreglo de nodos en un genoma en la conf->poblaci�n, retorna -1 si no lo encuentra.
//Par�metros: 	indexpob 	= �ndice del genoma en el arreglo de conf->poblaci�n.
//				innovNum 	= n�mero de innovaci�n buscado.
    unsigned i;
    //para cada i entre 0 y maxnodos busca el que tenga nodo.innovnum == al buscado
    unsigned totalNodos = conf->pob[indexpob].totalNodos;
    for (i=0; i<totalNodos; i++)
        if(conf->pob[indexpob].nodo[i].innovNum==innovNum)
            return(i);
    return(UINT_MAX);
}

unsigned buscarInnovConex(unsigned indexpob, unsigned innovNum, TConfig* conf)  //OPTIMIZADA
{
//Retorna el index del arreglo de conexiones en un genoma en la conf->poblaci�n, retorna -1 si no lo encuentra.
//Par�metros: 	indexpob 	= �ndice del genoma en el arreglo de conf->poblaci�n.
//				innovNum 	= n�mero de innovaci�n buscado.
    unsigned i;
    unsigned totalConex = conf->pob[indexpob].totalConexiones; //para acelerar evaluaci�n en for.
    //para cada i entre 0 y maxnodos busca el que tenga nodo.innovnum == al buscado
    for (i=0; i<totalConex; i++)
        if(conf->pob[indexpob].conex[i].innovNum==innovNum)
            return(i);
    return(UINT_MAX);
}

unsigned buscarInnovConexPorNodos(unsigned indexpob, unsigned innovIn, unsigned innovOut, TConfig* conf)  //OPTIMIZADA
{
//Retorna el index del arreglo de conexiones en un genoma en la conf->poblaci�n, retorna -1 si no lo encuentra.
//Par�metros: 	indexpob 	= �ndice del genoma en el arreglo de conf->poblaci�n.
//				innovIn 	= n�mero de innovaci�n de nodo de Entrada Buscado.
//				innovOut 	= n�mero de innovaci�n de nodo de Salida Buscado.
    unsigned i;
    unsigned totalConex=conf->pob[indexpob].totalConexiones; //para acelerar evaluaci�n de for.
    //para cada i entre 0 y maxnodos busca el que tenga nodo.innovnum == al buscado
    for (i=0; i<totalConex; i++)
        if (conf->pob[indexpob].conex[i].nodoIn==innovIn)
            if (conf->pob[indexpob].conex[i].nodoOut==innovOut)
                return(i);
    return(UINT_MAX);
}
// sudor y l�grimas.

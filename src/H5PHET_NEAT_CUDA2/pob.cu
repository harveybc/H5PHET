#include "hip/hip_runtime.h"
/** Archivo de manejo de poblaci�n, incluye ciclo principal - C file
	usan/modifican la poblacion �nicamente (pueden usarse operaciones de especies, genomas o genes).
*/

#ifndef PARAMS_H_INCLUDED
#include "params.h"
#define PARAMS_H_INCLUDED
#endif
#ifndef AUXILIARES_H_INCLUDED
#include "auxiliares.h"
#define AUXILIARES_H_INCLUDED
#endif
#ifndef GEN_H_INCLUDED
#include "gen.h"
#define GEN_H_INCLUDED
#endif
#ifndef GENOMA_H_INCLUDED
#include "genoma.h"
#define GENOMA_H_INCLUDED
#endif
#ifndef ESPECIE_H_INCLUDED
#include "especie.h"
#define ESPECIE_H_INCLUDED
#endif
#include "pob.h"
#include <hip/hip_runtime.h>

#define THREADSPB 96
#define MAXCONEX 1000

//kernel de cuda que toma datos de entrenamiento y lista de conexiones, genera vector de fitness.
__global__ void evalPobCUDA(int numEntradas, int numSalidas, int numBias, int numDatos, float* dataGTDf, int** conexIn, int** conexOut, float** conexPeso, int* tamListaConexPost, float** valorC, float** valorTr, float* fitness)
{
	int i,j,k;
    // vectores en registros para conexiones: valor[], conexIn[] conexOut[] conexPeso[]
    int r_conexIn[16];
    int r_conexOut[16];
    float r_conexPeso[16];
	// vector en registros para los nodos (HASTA 96 ENTRADAS,6 salidas,192 Nodos en total)
    float valor[16];
    // vector de salida de entrenamiento.
    float r_valorTr[4];
	// vector de tama�os de lista de conexiones.
	int r_tamListaConexPost;
    // calcula el id del thread
    int idx=blockDim.x*blockIdx.x+threadIdx.x;
	// calcula el n�mero de repeticiones para floor de todos los datos/buffSize
    int numReps=(numDatos*(numEntradas+numSalidas))/THREADSPB;
	// THREADSPB tambi�n es el n�mero de datos completos leidos de global a shared.
	// vector de datos de entrenamiento leidos en shared de tama�o THREADSPB
	__shared__ float s_dataGTDf[THREADSPB];
	// vector de fitness resultante para cada genoma por repetici�n entre grbaciones a global.
	__shared__ float s_fitness[THREADSPB];
    // vector de valor calculado para para calculo de fitness por correlaci�n.
	__shared__ float s_valorC[THREADSPB];
    // vector de valor de entrenamiento para para calculo de fitness por correlaci�n.
	__shared__ float s_valorTr[THREADSPB];

    // inicializa el fitness del genoma  en 0
    s_fitness[idx]=0;
    // calcula l�mites de for para optimizaci�n.
    int indSalidas=(numEntradas+numBias+numSalidas);
    int indBias=(numEntradas+numBias);
    int indInS=numEntradas*4;
    int g_indexGTD=0;
    int s_indexGTD=0;
    int numRepsShared=THREADSPB/(numEntradas+numSalidas);
    float A =-2.435;
    float acum=0;
    float vCm=0; //para las medias
    float vTm=0;
    int contDato=0;
    float sum0=0;
    float sum1=0;
    float sum2=0;
	int r_tamListaConexPostAnt;
    __syncthreads();
	// coloca en 0 valor[]
	for (i=0;i<16;i++)
    {
        valor[i]=0;
    }

	// vector de tama�os de lista de conexiones.
    __syncthreads();
    r_tamListaConexPost=tamListaConexPost[idx];
	r_tamListaConexPostAnt=r_tamListaConexPost-1;
    // lee las conexiones en los arreglos de registros (LENTO)
    __syncthreads();
	//for (i=0;i<r_tamListaConexPost;i++)
	if (r_tamListaConexPost>16) r_tamListaConexPost=16;
	for (i=0;i<r_tamListaConexPost;i++)
    {
        r_conexIn[i]=conexIn[idx][i];
        r_conexOut[i]=conexOut[idx][i];
        r_conexPeso[i]=conexPeso[idx][i];
    }

	// para el n�mero de repeticiones:
    for (i=0;i<numReps;i++)
    {
		
		// coloca los valores de entrada desde dataGTDf[indexGTD] en el buffer s_dataGTDf[]
        __syncthreads();
        s_dataGTDf[idx]=dataGTDf[g_indexGTD+idx];
       // __syncthreads();
        // para cada uno de los datos en s_dataGTDf[]
        s_indexGTD=0;

		for (j=0;j<numRepsShared;j++)
        {

            // coloca bias en 1 TODO: se puede colocar solo na vez al principio (VERIFICAR).
            valor[numEntradas]=1;
			// coloca entradas en sus respectivos nodos valor[].

			//			for (k=0;k<numEntradas;k++)

			for (k=0;k<numEntradas;k++)
            {
                __syncthreads();
                valor[k]=s_dataGTDf[s_indexGTD];
				s_indexGTD++;
            }
			// coloca las salidas de entrenamiento en el vector valorTr
            //for (k=0;k<numSalidas;k++)
			for (k=0;k<numSalidas;k++)
            {
                __syncthreads();
				r_valorTr[k]=s_dataGTDf[s_indexGTD];
                __syncthreads();
                s_valorTr[s_indexGTD]=r_valorTr[k];
                // acumula para calulo de medias
                vTm=vTm+r_valorTr[k];
				__syncthreads();
                s_indexGTD++;
            }
            // calcula el fSigma de las entradas
            for (k=0;k<numEntradas;k++)
            {
                valor[k]=2*(expf(A*((valor[k]-1)*(valor[k]-1))))-1;
            }

			// inicializa el acumulador.
            acum=0;
            for (k=0;k<r_tamListaConexPost;k++)
            {
                acum=acum+(r_conexPeso[k]*valor[r_conexIn[k]]);
                // si el conexOut no es el �ltimo
				// OPTIMIZABLE: r_tamListaConexPost-1 puede ser una cte o reg
                if (k<(r_tamListaConexPostAnt))
                {
                    // si el ConexOut siguiente es diferente al actual
                    if (r_conexOut[k]!=r_conexOut[k+1])
                    {
                        // calcula el sigma
                        valor[r_conexOut[k]]=2*(expf(A*((acum-1)*(acum-1))))-1;
                        //reinicializa el acumulador
                        acum=0;
                    }
                }
            }
            // saca el Fsigma del �ltimo nodo.// OPTIMIZABLE: r_tamListaConexPost-1 puede ser una cte o reg
            valor[r_conexOut[r_tamListaConexPostAnt]]=2*(expf(A*(acum-1)*(acum-1)))-1;
            // guarda los valores calculados en el arreglo valoresC[]
            s_valorC[idx]=valor[indBias];
            // acumula para calculo de fitness con correlaci�n
            vCm=vCm+valor[indBias];
            // incrementa el index del dato leido
            contDato++;

        }
        //copia s_valorC y s_valorTr a global
        for (j=0;j<numRepsShared;j++)
        {
            //copia s_valorC y s_valorTr a global
            __syncthreads();
            valorTr[idx][g_indexGTD+j]=s_valorTr[idx];
            __syncthreads();
            valorC[idx][g_indexGTD+j]=s_valorC[idx];
        }
        // incrementa index de dataGTDf
        __syncthreads();
        g_indexGTD+=THREADSPB;
	}
    // calcula el fitness como el coeficiente de correlaci�n de Pearson de los 2 vectores (1 si =es,-1 si inversos, 0 si diferentes)
    //fitness[idx]=correlacCUDA(valorC[idx],valorTr[idx]);
    vCm/=contDato;
    vTm/=contDato;
    // calcula sum0(0,nD,(Xi-Xm)*(Yi-Ym)),sum1(0,nD,sqr(Xi-Xm)) y sum2(0,nD,sqr(Yi-Ym)
	for (i=0;i<numDatos;i++)
    {
        // lee en s_valorC y s_valorTr los vectores globales
        __syncthreads();
        s_valorC[idx]=valorC[idx][i];
        __syncthreads();
        s_valorTr[idx]=valorTr[idx][i];
        __syncthreads();
        sum0+=((s_valorTr[idx]-vTm)*(s_valorC[idx]-vCm));
        __syncthreads();
        sum1+=((s_valorTr[idx]-vTm)*(s_valorTr[idx]-vTm));
        __syncthreads();
        sum2+=((s_valorC[idx]-vCm)*(s_valorC[idx]-vCm));
        __syncthreads();
        g_indexGTD+=THREADSPB;
    }
    // retorna la correlaci�n
    __syncthreads();
    fitness[idx]=(sum0/(sqrtf(sum1)*sqrtf(sum2)));
	fitness[idx]=0.5;
    __syncthreads();
}

int evalPob(int numGenomas, int numDatos, hdrGTDv1 headerGTD, hdrSNNv1* headerSNN, float** dataGTD, tConexDataF** listaConexData, float* fitness, int** ordenEval, int* tamListaConexPost, TConfig* conf)
// genera un vector de fitness al evaluar los datos GTD con la lista de conexiones de los genomas SNN
// asume que todos los SNN tienen iguales valores de encabezado excepto el n�mero de conex.
// retorna 0 si hubo error
{
    int i,j;
    int threadsPorBlock=THREADSPB;
    int blocksPorGrid=(1+threadsPorBlock-1)/threadsPorBlock;//?
	float fitness_h[THREADSPB];
	int tempCin[MAXCONEX];
	int tempCout[MAXCONEX];
	float tempCpeso[MAXCONEX];
    // inicializa vectores de evaluaci�n.
    for (i=0;i<numGenomas;i++)
    {
        // reserva memoria para ordenEval
        conf->ordenEval[i]=(int*)malloc(conf->pob[i].totalNodos*sizeof(int));
        if (!conf->ordenEval[i])
        {
            printf("\nError 66.0.7 en genOrdenEvalF1g llamando a malloc()");
            return(0);
        }
        // actualiza el headerSNN;
        genSSNhdr1(i,conf);
        // genera la lista de conex para este genoma
        genOrdenEvalF1g(i, conf);
        // reordena listaConexData para que est�n primero los nodos necesarios y elimina enableds (optimizaci�n)
        conf->tamListaConexPost[i]=ordenarListaConexF(conf->headerSNN[i], conf->listaConexData[i] ,conf->ordenEval[i],conf->tamOrdenEval[i]);
        // CUDA: reserva memoria en device para listaConexData[i][tamConexPost[i]]
        hipMalloc(&(conf->cu_conexIn[i]),conf->tamListaConexPost[i]*sizeof(int));
		// CUDA: reserva memoria en device para listaConexData[i][tamConexPost[i]]
        hipMalloc(&(conf->cu_conexOut[i]),conf->tamListaConexPost[i]*sizeof(int));
		// CUDA: reserva memoria en device para listaConexData[i][tamConexPost[i]]
        hipMalloc(&(conf->cu_conexPeso[i]),conf->tamListaConexPost[i]*sizeof(float));
		// genera vector conexIn para transferir
		for (j=0;j<conf->tamListaConexPost[i];j++) 
		{
			tempCin[j]=conf->listaConexData[i][j].conexIn;
			tempCout[j]=conf->listaConexData[i][j].conexOut;
			tempCpeso[j]=conf->listaConexData[i][j].peso;
		}
        // CUDA:  transfiere listaConexData[i] ordenada a device
        hipMemcpy(conf->cu_conexIn[i], tempCin, conf->tamListaConexPost[i]*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(conf->cu_conexOut[i], tempCout, conf->tamListaConexPost[i]*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(conf->cu_conexPeso[i], tempCpeso, conf->tamListaConexPost[i]*sizeof(float), hipMemcpyHostToDevice);
    }
	printf("Llamando kernel de CUDA\n");
    // llama el kernel de CUDA para los 96 threads.
    evalPobCUDA<<<blocksPorGrid,threadsPorBlock>>>(conf->numEntradas,conf->numSalidas,conf->numBias,conf->numDatos,conf->cu_dataGTDf,conf->cu_conexIn,conf->cu_conexOut,conf->cu_conexPeso,conf->cu_tamListaConexPost,conf->cu_valorC, conf->cu_valorTr, conf->cu_fitness);
    // CUDA: transfiere el vector fitness[sizePob] de device a host
    hipMemcpy(fitness_h, conf->cu_fitness, conf->sizePob*sizeof(float), hipMemcpyDeviceToHost);
    // libera memoria de ordenEval y de device de listaConexData
    for (i=0;i<numGenomas;i++)
    {
		printf("F%3.3f",fitness_h[i]);
        free(conf->ordenEval[i]);
        // CUDA: libera memoria de  listaconexData ordenada
        hipFree(conf->cu_conexIn[i]);
		hipFree(conf->cu_conexOut[i]);
		hipFree(conf->cu_conexPeso[i]);
    }
    printf("\n");
    return(1);
}

unsigned distribProc(TConfig* conf)
// lee los archivos NNP con el prefix fileNameDistrib y compara cada genoma con los
// representantes actuales, si el m�s cercano de distrib es mejor, lo deja.
// usado para procesamiento distribuido.
// retorna 0 si error, 1 si ok.
{
    int i, j, k;
    char tmpString[512];
    FILE* fileIn=NULL;
    int leidos;
    hdrNNPv1 headerNNP;
    unsigned buscado1;
    unsigned buscado2;
    unsigned especieCercana;
    // copia el fileNameDistrib a tmpstring;
    strcpy(tmpString, conf->fileNameDistrib);
    // para i= el n�mero cargarDistrib
    for (i=0; i<conf->cargarDistrib; i++)
    {
        // genera el nombre de archivo
        sprintf(tmpString, "%s%i.nnp", conf->fileNameDistrib, i);
        // abre el archivo
        fileIn=fopen(tmpString,"rb");
        if (fileIn==NULL)
        {
            fclose(conf->logFile);
            conf->logFile=fopen(conf->fileNameLog,"a+");
            fprintf(conf->logFile,"<br>\nError 1.1 en funcion distribProc() llamando a fopen\n");
            return(1);
        }
        else
        {
            // lee encabezado NNP
            leidos = fread(&headerNNP,sizeof(hdrNNPv1),1,fileIn);
            if (leidos<1)
            {
                fclose(conf->logFile);
                conf->logFile=fopen(conf->fileNameLog,"a+");
                fprintf(conf->logFile,"<br>\nError 1.2 en funcion distribProc() llamando a fread\n");
                return(1);
            }
            else
            {
                // busca primer genoma no representante
                k=0;
                while ((k==conf->representantes[conf->pob[k].especie])&&(k<conf->sizePob))
                    k++;
                buscado1=k;
                if (buscado1>=conf->sizePob)
                {
                    fclose(conf->logFile);
                    conf->logFile=fopen(conf->fileNameLog,"a+");
                    fprintf(conf->logFile,"<br>\nError 1.3 en funcion distribProc() genoma no encontrado\n");
                    return(0);
                }
                // guarda buscado1 en tmpgenoma
                copiarGenoma(buscado1,conf->tmpIndexPob,conf);
                // para cada SNN hasta numGenomas
                for (j=0; j<headerNNP.numGenomas; j++)
                {
                    // carga SNN en busca1
                    if (snnDataLoader(buscado1,0,fileIn,conf)==0)
                    {
                        fclose(conf->logFile);
                        conf->logFile=fopen(conf->fileNameLog,"a+");
                        fprintf(conf->logFile,"<br>\nError 1.4 en funcion distribProc() llamando a snnDataLoader(%d)\n",buscado1);
                        return(0);
                    }
                    // busca la especie mas cercana a pob[busca1]
                    especieCercana=especieMinDist(buscado1, conf->c1,conf->c2,conf->c3,conf->eG_t,conf);
                    // actualiza la especie
                    conf->pob[buscado1].especie=especieCercana;
                    // si headerSNN.fitness > representantes[especieCercana].fitness
                    if ((conf->pob[buscado1].fitness>conf->pob[conf->representantes[especieCercana]].fitness)&&(conf->pob[buscado1].fitness>conf->pob[conf->sizePob+especieCercana].fitness))
                    {
                        // busca2 el que NO sea representante y de  especieCercana
                        k=0;
                        while (((k==conf->representantes[conf->pob[k].especie])||(conf->pob[k].especie!=especieCercana)||(k==buscado1))&&(k<conf->sizePob))
                            k++;
                        buscado2=k;
                        // si no lo encontr�, muestra error
                        if (buscado2>=conf->sizePob)
                        {
                            fclose(conf->logFile);
                            conf->logFile=fopen(conf->fileNameLog,"a+");
                            fprintf(conf->logFile,"<br>\nError 1.5 en funcion distribProc()\n");
                            return(0);
                        }
                        // si busca1.fitness>busca2.fitness //nunca se sabe :)
                        if (conf->pob[buscado1].fitness>conf->pob[buscado2].fitness)
                        {
                            // copiarGenoma(busca1,busca2)
                            fclose(conf->logFile);
                            conf->logFile=fopen(conf->fileNameLog,"a+");
                            fprintf(conf->logFile,"D(%d,%d)%d",i,conf->pob[buscado1].especie,buscado2);
                            copiarGenoma(buscado1,buscado2,conf);
                        }
                        // para debugging
                    }
                }
                // restaura tmp a busca1
                copiarGenoma(conf->tmpIndexPob,buscado1,conf);
                // cierra el archivo de entrada.
                fclose(fileIn);
            }
        }
    }
    return(1);
}

unsigned cargarDesdeNNP(char* filename, unsigned especieDestino, unsigned cantidad,TConfig* conf)
// Carga desde un archivo NNP (Neural Network Population) los genomas que se colocan
// en la especie deseada.
// retorna 0 si hubo error, 1 si ok
{
    FILE *fileIn;
    size_t leidos=0;
    hdrNNPv1 headerNNP;
    int i,k;
    // abre el archivo NNP para lectura
    if ((fileIn=fopen(filename,"rb"))==NULL)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 49 en funcion cargarDesdeNNP() llamando a fopen\n");
        return(0);
    }
    // lee el encabezado NNP
    leidos=fread(&headerNNP,sizeof(hdrNNPv1),1,fileIn);
    // verifica leidos
    if (leidos<1)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>Error 343 en cargarDesdeNNP() llamando a fread()");
        return(0);
    }
    // verifica fileID y versi�n de header NNP.
    if ((headerNNP.fileID[0]!='N')||(headerNNP.fileID[1]!='N')||(headerNNP.fileID[2]!='P')||(headerNNP.version!=1)||(headerNNP.numGenomas>32000))
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 344 en cargarDesdeNNP() error en encabezado NNPv1 %c %c %c %d ,%d",headerNNP.fileID[0],headerNNP.fileID[1],headerNNP.fileID[2],headerNNP.version,headerNNP.numGenomas);
        return(0);
    }
    // cuenta i=0 hasta numGenomas
    if (cantidad>conf->numEspecies)
        cantidad=conf->numEspecies;
    if (cantidad>headerNNP.numGenomas)
        cantidad=headerNNP.numGenomas;
    fclose(conf->logFile);
    conf->logFile=fopen(conf->fileNameLog,"a+");
    fprintf(conf->logFile,"<br>NNP%d=",cantidad);
    for (i=0; i<cantidad; i++)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"%d,",i);
        // Lee SNN
        if (snnDataLoader(conf->representantes[i],i,fileIn,conf)==0)
        {
            fclose(conf->logFile);
            conf->logFile=fopen(conf->fileNameLog,"a+");
            fprintf(conf->logFile,"<br>Error 343.1 en cargarDesdeNNP() llamando a snnDataLoader()");
            return(0);
        }
        // copia el nuevo genoma sobre todos los de su especie.
        for (k=0; k<conf->sizePob; k++)
        {
            if ((conf->pob[k].especie==i)&&(k!=conf->representantes[i]))
                copiarGenoma(conf->representantes[i],k,conf);
        }

    }
    // cierra el archivo NNP
    fclose(fileIn);
    return(1);
}

// todas las funciones de este archivo deben cumplir con que no modifican a los representantes (NMR)
unsigned cicloPrincipal(TConfig* conf)  // OPTIMIZADA    //TODO verificar por NMR todas las funcs en el ciclo, adicionar par�metros al llamar a Funcion perturbarPeso con pert no uniforme, //TODO Funcion perturbarTh.
{
    // Prerequisisto: Tener una conf->poblaci�n inicial usando primeraGen()
    // Realiza el ciclo principal de NEAT, realiza  evaluaci�n, especiaci�n,seleccion(y cruce) y mutaci�n AN + AC (con probabilidades de entrada)
    // hasta que se cumpla maxconf->iteracion o minFitness se alcance.
    // Par�metros:
    //			probMutAN = entre 0 y 1 prob de mutaci�n AN
    //			probMutAC = entre 0 y 1 prob de mutaci�n AC
    //			maxconf->iteracion = m�ximo n�mero de veces que debe correrse el ciclo
    //			minFitness = entre 0 y 1 m�nimo fitness (promediado durante el arch de entrenamiento) necesario para detener el ciclo
    //			maxMemoriaUsada = Max memoria que se puede utilizar en MBytes (si se supera, sale del ciclo)
    //			fileNameGTDv1 = Path para el archivo de datos de entrenamiento en formato GTDv1.
    //			filenameMejor = Path  para el mejor genoma encontrado al  omento de terminar el ciclo principal.
    //			repTrain = n�mero de repeticiones del archivo de entrenamiento(para cada genoma).
    //			maxBufferSize = tam��o m�ximo del buffer de lectura de archivos de entrada., memoria usada = maxBufferSize*4 bytes
    // retorna -1 si hay error, indexpob de genoma con mayor fitness si se cumple una de las condiciones de parada.
    unsigned mejor=0;
    long unsigned memoriaUsada=0;
    unsigned i=0;
    float fitMejor=0;
    int tmpUsarD=conf->usarDistrib;
    // imprime el mejor genoma o de la poblaci�n
    // regla de los \n : uno controla como termina, no como empieza. (solo se puede o no colocar \n al final de cada linea, nunca al principio)
    //						para los errores se hace lo contrario: ellos controlan como empiezan, pero se supone que todo lo que se imprima despu�s
    //						ser�n m�s errores, por tanto, se debe colocar el \n al principio. y un \n al final del error de cicloprincipal./	fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>unsigned  introduciendo GenomaPerfecto como elemento 15");
    /*if (genomaPerfecto(15,conf)==0){ //necesaria esta evaluaci�n antes de primera especiaci�n?
    	fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 59 en funcion cicloPrincipal() llamando a evaluarPob(1,%u,%u,%u)",conf->maxBufferSize,conf->numEntradas,conf->numSalidas);
    	return(UINT_MAX);
    }*/
    // coloca iteracion en 0;
    conf->iteracion=0;

    calcularD(conf);
    fclose(conf->logFile);
    conf->logFile=fopen(conf->fileNameLog,"a+");
    fprintf(conf->logFile,"<br>evaluando genoma inicial \n");
    // coloca el n�mero de especies en 1 ya que genoma inicial le asigna la especie 0 en primeraGen();
    conf->numEspecies=1;
    // realiza primera evaluaci�n //TODO: probar si funciona mejor inicializando (primer par�metro)
    if (evaluarPob(1,1,conf->maxBufferSize,conf->fileNameGTDv1, conf)==0)  //necesaria esta evaluaci�n antes de primera especiaci�n?
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 59 en funcion cicloPrincipal() llamando a evaluarPob(1,%u,%u,%u)",conf->maxBufferSize,conf->numEntradas,conf->numSalidas);
        return(UINT_MAX);
    }
    // llama a calcularLimTh de gen.h para ccontrolar el max y min Th de las neuronas de toda la pob
    calcularLimTh(conf);
    // especiaci�n (necesita haberse avaluado al menos una vez el genoma.)
    fclose(conf->logFile);
    conf->logFile=fopen(conf->fileNameLog,"a+");
    fprintf(conf->logFile,"<br>Primera especiaci�n.\n");
    if (especiacion(conf)==0)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 60 en funcion cicloPrincipal() llamando a especiacion(1,%u,%u,%u)",conf->maxBufferSize,conf->numEntradas,conf->numSalidas);
        return(UINT_MAX);
    }


    // calcula el mejor y su fitness
    if ((mejor=buscarMejorFitness(conf))==UINT_MAX)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 59_144 en cicloPrincipal() llamando a buscarMejorFitness()");
        return(UINT_MAX);
    }
    fitMejor=conf->pob[mejor].fitness;
    // mientras no se cumplan las tres condiciones de parada
    fclose(conf->logFile);
    conf->logFile=fopen(conf->fileNameLog,"a+");
    fprintf(conf->logFile,"<br>Comenzando iteraci�n,\n");
    while((conf->pob[mejor].fitness<conf->minFitness)&&(conf->iteracion<conf->maxIteraciones)&&(memoriaUsada<conf->maxMemoriaUsada))
    {
		printf("\nWorkz!!");
        // TODO: en ninguna etapa se deben modificar los representantes excepto en la de perturbar peso (VERIFICAR)
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>Iteraci�n %4u =",conf->iteracion);
        // Imprime los mejores de cada especie
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile," M(%2u) =%3u(%9.9f)e:%2u ",conf->pob[mejor].especie,mejor,fitMejor,conf->numEspecies);
        // TODO: si hay m�s de 3 especies (debido a bug), se realiza sc, sinn� no!!!
        // realiza selecci�n y cruces Funcion seleccionCrossover() pag 54 - 55 phd y otras fuentes,maneja conservaci�n, eliminaci�n y vector de

//TODO: dividir sc en selecci�n y reproducci�n.
        if (conf->numEspecies>2)
        {
            fclose(conf->logFile);
            conf->logFile=fopen(conf->fileNameLog,"a+");
            fprintf(conf->logFile,"sc");
            if (seleccionCrossover(conf)==0)
            {
                fclose(conf->logFile);
                conf->logFile=fopen(conf->fileNameLog,"a+");
                fprintf(conf->logFile,"<br>\nError 61 en funcion cicloPrincipal() llamando a  seleccionCrossover");
                return(UINT_MAX);
            }
        }
        // realiza mutaciones AN y AC		VERIFICADA
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"m");
        for (i=0; i<conf->sizePob; i++)
        {
            // realiza mutaci�n AN con rconf->pobabilidad probMutAN
            // probabilidad de mutaci�n
            if (((float)randL(conf))<=conf->probMutAN)
            {
                // if (conf->representantes[conf->pob[i].especie]!=i){//Muta si no es el representante
                if (mutarAN(i,conf)==0)
                {
                    fclose(conf->logFile);
                    conf->logFile=fopen(conf->fileNameLog,"a+");
                    fprintf(conf->logFile,"<br>\nError 62 en funcion cicloPrincipal() llamando a mutarAN(%u)",i);
                    return(UINT_MAX);
                }
            }
            //}
            // realiza mutaci�n AC con probabilidad porbMutAC
            // probabilidad de mutaci�n
            if (((float)randL(conf))<=conf->probMutAC)
            {
                // if (conf->representantes[conf->pob[i].especie]!=i)//Muta si no es el representante
                if (mutarAC(i,conf->maxIntentosMutarAC,conf)==0)
                {
                    fclose(conf->logFile);
                    conf->logFile=fopen(conf->fileNameLog,"a+");
                    fprintf(conf->logFile,"<br>\nError 63 en funcion cicloPrincipal() llamando a mutarAC(%u)",i);
                    return(UINT_MAX);
                }
            }
        }
        // realiza perturbaci�n de pesos 		VERIFICADA
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"w");
// Hace las perturbaciones de peso en paralelo
//#pragma omp parallel for
        for (i=0; i<conf->sizePob; i++)
        {
            // if (conf->representantes[conf->pob[i].especie]!=i){
            // perturbarPesoYth(i,conf->porcentMutPeso,conf->porcentMutNTh,conf->probMutTh,conf->probMutPeso, conf);
            perturbarPeso(i,conf->porcentMutPeso,conf->probMutPeso,0,0,conf->tipoPert,0, conf);
            //}
        }
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"N");
        // introduce poblaci�n desde archivo NNP en diferentes especies
        // FALTA: poner como param cargar al inicio o cargar al final
        // if ((conf->cargarNNP>0)&&(conf->numEspecies==conf->spEspecies))
        if ((conf->cargarNNP>0)&&(conf->numEspecies==conf->cargarNNP))
        {
            fclose(conf->logFile);
            conf->logFile=fopen(conf->fileNameLog,"a+");
            fprintf(conf->logFile,"<br>\nCargando Genomas desde NNP\n");
            if (cargarDesdeNNP(conf->fileNameNNPv1Load,0,conf->cargarNNP,conf)==0)
            {
                fclose(conf->logFile);
                conf->logFile=fopen(conf->fileNameLog,"a+");
                fprintf(conf->logFile,"<br>\nError 355 en cicloPrincipal() llamando a cargarDesdeNNP()");
                return(UINT_MAX);
            }
            conf->cargarNNP=0;
        }
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"D");
        // implementaci�n de procesamiento distribuido(justo antes de la eval)
        if ((conf->cargarDistrib>0)&&(conf->numEspecies==conf->spEspecies)&&(tmpUsarD>0))
        {
            // TODO: FALTA: param para seleccionar si descargar antes  o despu�s.
            // realiza el procesamiento distribuido en los archivo actuales
            if (distribProc(conf)==0)
            {
                fclose(conf->logFile);
                conf->logFile=fopen(conf->fileNameLog,"a+");
                fprintf(conf->logFile,"<br>\nError 355.1 en cicloPrincipal() llamando a distribProc()");
                return(UINT_MAX);
            }
            tmpUsarD--;
        }

/*        // deja descargando en paralelo archivos de procesamiento distrib mientras hace evaluaci�n
        if ((tmpUsarD==0)&&(conf->cargarDistrib>0))
        {
            system(conf->distriCmd);
            tmpUsarD=conf->usarDistrib;
        }
*/
        // evaluaci�n
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"e");
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"C");
        if (evaluarPob(1,0,conf->maxBufferSize,conf->fileNameGTDv1, conf)==0)  //// TODO EL ANTERIORMANTE DICHO PARAMETRO para conf->primero para las evaluaciones y hacer inicializaciones a 0 de los valores cuando se crean las neuronas.
        {
            fclose(conf->logFile);
            conf->logFile=fopen(conf->fileNameLog,"a+");
            fprintf(conf->logFile,"<br>\nError 59 en funcion cicloPrincipal() llamando a evaluarPob(1,%u,%u,%u)",conf->maxBufferSize,conf->numEntradas,conf->numSalidas);
            return(UINT_MAX);
        }
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"s");
        // especiaci�n (necesita haberse evaluado al menos una vez el genoma.)
        if (especiacion(conf)==0)
        {
            fclose(conf->logFile);
            conf->logFile=fopen(conf->fileNameLog,"a+");
            fprintf(conf->logFile,"<br>\nError 60 en funcion cicloPrincipal() llamando a especiacion(1,%u,%u,%u)",conf->maxBufferSize,conf->numEntradas,conf->numSalidas);
            return(UINT_MAX);
        }
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"Sv");
        // guarda el mejor Genoma
        guardarGenomaSNN(conf->sizePob+(conf->pob[buscarMejorFitness(conf)].especie),conf->fileNameSNNv1,conf);
        // guarda representantes en NNP
        guardarRepresentantesNNP(conf->fileNameNNPv1, conf);
        // si cargarNNP==0 guarda en filenameNNPv1Load (para poder continuar de nuevo si se interrumpe)
        if (conf->cargarNNP==0)
            guardarRepresentantesNNP(conf->fileNameNNPv1Load, conf);
        guardarRepresentantesNNP(conf->fileNameNNPv1, conf);
//Todo: quitar cuando se ahay solucionado problama de delay en especiaci�n
        if (verificarListasInnov(conf)==0)
        {
            fclose(conf->logFile);
            conf->logFile=fopen(conf->fileNameLog,"a+");
            fprintf(conf->logFile,"<br>Error 60.1 en cicloPrincipal llamando a verificarListaInnov()\n");
            return(UINT_MAX);
        }
        conf->iteracion++; // incrementa conf->iteracion
        //realiza competencia (si el porcentCompetencia es negativo, no hace nada)
        if (conf->porcentCompetencia>0)
        {
            fclose(conf->logFile);
            conf->logFile=fopen(conf->fileNameLog,"a+");
            fprintf(conf->logFile,"C");
            i=competencia(conf);
        }
        // verifica  y  busca el mejor y su fitness
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"v\n");
        // calcula m�ximo fitness entre los conf->representantes de la conf->pob obtiene el indexpob en mejor
        if ((mejor=buscarMejorFitness(conf))==UINT_MAX)
        {
            fclose(conf->logFile);
            conf->logFile=fopen(conf->fileNameLog,"a+");
            fprintf(conf->logFile,"<br>\nError 59_144 en cicloPrincipal() llamando a buscarMejorFitness()");
            return(UINT_MAX);
        }
        fitMejor=conf->pob[mejor].fitness;
        memoriaUsada=calcularMemoriaUsada(conf->sizePob,conf)/(1048576); // calcula memoria usada en MB
        /*for (j=0;j<conf->numEspecies;j++)
        	fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>E%u[%u]=%3d(%2.2f), ",j,contarGPEsp(j, conf),conf->representantes[j],conf->pob[conf->representantes[j]].fitness);
        //Se llama a calcularLimTh(conf); de gen.h para ccontrolar el max y min Th de las neuronas de toda la pob
        fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\n");*/
        calcularLimTh(conf);
//while((conf->pob[mejor].fitness<conf->minFitness)&&(conf->iteracion<conf->maxconf->iteracion)&&(memoriaUsada<conf->maxMemoriaUsada)){
        if (conf->pob[mejor].fitness>=conf->minFitness)
        {
            fclose(conf->logFile);
            conf->logFile=fopen(conf->fileNameLog,"a+");
            fprintf(conf->logFile,"<br>Salida de ciclo principal debido a MinFitness alcanzado.\n");
        }
        if (conf->iteracion>=conf->maxIteraciones)
        {
            fclose(conf->logFile);
            conf->logFile=fopen(conf->fileNameLog,"a+");
            fprintf(conf->logFile,"<br>Salida de ciclo principal debido a m�ximo n�mero de conf->iteracion.\n");
        }
        if (memoriaUsada>=conf->maxMemoriaUsada)
        {
            fclose(conf->logFile);
            conf->logFile=fopen(conf->fileNameLog,"a+");
            fprintf(conf->logFile,"<br>Salida de ciclo principal debido a m�xima memoria usada=%lu\n",memoriaUsada);
        }

    }

    return(mejor);
}

//Nueva evaluarPob, versi�n antigua al final.
unsigned evaluarPob(unsigned inicializar,unsigned primero,unsigned maxBufferSize, char *fileNameGTDv1, TConfig* conf)  // OPTIMIZADA, NMR
// lee los archivos SNN y GTD en memoria dentro de los buffers de entrada de la funci�n evalPob
// , luego se llama a la funci�n que eval�a toda la poblaci�n  y finalmente se actualiza el
// fitness de cada genoma de la pob desde el vector de retorno de evalPob();.
// evalPob(int numGenomas, int numDatos, hdrGTDv1 headerGTD, hdrSNNv1* headerSNN, float** dataGTD, tConexDataF** listaConexData, float* fitness)
// retora 0 si error,1 si OK
{
    int numGenomas=conf->sizePob;
    int i,leidos;
    FILE* fileIn;
    // si primero==1
    if (primero==1)
    {
        // abre archivo GTDv1
        fileIn=fopen(conf->fileNameGTDv1,"rb");
        if (!fileIn)
        {
            printf("\nError 57.1 en evaluarPob() llamando a fopen()");
            return(0);
        }
        // lee headerGTD
        leidos=fread(&conf->headerGTD,sizeof(hdrGTDv1),1,fileIn);
        // busca el filesize
        if (leidos<1)
        {
            printf("\nError 58 en evaluarPob() llamando a fread()");
            return(0);
        }
        // averigua el fileSize y calcula conf->numDatos
        fseek(fileIn, 0, SEEK_END);
        conf->numDatos = (ftell(fileIn) - sizeof(hdrGTDv1))/(conf->headerGTD.tamRegistros*(conf->headerGTD.numEntradas+conf->headerGTD.numSalidas));
        //printf("\nFileSize=%d",ftell(fileIn));
        rewind(fileIn);
        leidos=fread(&conf->headerGTD,sizeof(hdrGTDv1),1,fileIn); // para ubicarlo en posici�n de lectura de datos
        // reserva memoria en headerSNN[numGenomas]
        conf->headerSNN=(hdrSNNv1*)malloc(numGenomas*sizeof(hdrSNNv1));
        if (!conf->headerSNN)
        {
            printf("\nError 60 en evaluarPob() llamando a malloc()");
            return(0);
        }
        //reserva memoria para el vector de salidas calculadas para usar en c�lculo de fitness en evalGenom()
        conf->valoresC=(float*)malloc(conf->numDatos*sizeof(hdrSNNv1));
        if (!conf->valoresC)
        {
            printf("\nError 60 en evaluarPob() llamando a malloc()");
            return(0);
        }
        // reserva memoria para conf->ordenEval[i<numGenomas][numNodos[i]]
        conf->ordenEval=(int**)malloc(numGenomas*sizeof(int*));
        if (!conf->ordenEval)
        {
            printf("\nError 60.5 en evaluarPob() llamando a malloc()");
            return(0);
        }
        // reserva memoria para conf->tamOrdenEval[numGenomas]
        conf->tamOrdenEval=(int*)malloc(numGenomas*sizeof(int));
        if (!conf->tamOrdenEval)
        {
            printf("\nError 60.6 en evaluarPob() llamando a malloc()");
            return(0);
        }
        // tamListaConexPost[numGenomas]
        conf->tamListaConexPost=(int*)malloc(numGenomas*sizeof(int));
        if (!conf->tamListaConexPost)
        {
            printf("\nError 60.7 en evaluarPob() llamando a malloc()");
            return(0);
        }
        for (i=0;i<numGenomas;i++)
        {
            conf->ordenEval[i]=(int*)malloc(conf->maxNodos*sizeof(int));
            if (!conf->ordenEval[i])
            {
                printf("\nError 60.7 en evaluarPob() llamando a malloc()");
                return(0);
            }
        }
        // si no son datos tipo float
        if (conf->headerGTD.tamRegistros==4)
        {
            // reserva memoria en fitness[numGenomas]
            conf->fitness=(float*)malloc(numGenomas*sizeof(float));
            if (!conf->fitness)
            {
                printf("\nError 61 en evaluarPob() llamando a malloc()");
                return(0);
            }
            // reserva memoria en listaConexData[i<numGenomas][numConex[i]]
            conf->listaConexData=(tConexDataF**)malloc(numGenomas*sizeof(tConexDataD*));
            if (!conf->listaConexData)
            {
                printf("\nError 62 en evaluarPob() llamando a malloc()");
                return(0);
            }
            for (i=0;i<numGenomas;i++)
            {
                conf->listaConexData[i]=(tConexDataF*)malloc(conf->maxConex*sizeof(tConexDataF));
                if (!conf->listaConexData[i])
                {
                    printf("\nError 63.11 en evaluarPob() llamando a malloc()");
                    exit(0);
                }
            }
            // reserva memoria en conf->dataGTD[conf->numDatos][numEntradas+numSalidas]
            // para todo el buffer, ojo! obtiene toda la memoria para el index 0 para que sean consecutivos
            // durante la lectura
            conf->dataGTDf=(float**)malloc(conf->numDatos*sizeof(float*));
printf("NumDatos=%d\n",conf->numDatos);
            if (!conf->dataGTDf)
            {
                printf("\nError 64 en evaluarPob() llamando a malloc()");
                return(0);
            }
            conf->dataGTDf[0]=(float*)malloc((conf->headerGTD.numEntradas+conf->headerGTD.numSalidas)*conf->numDatos*sizeof(float));
            if (!conf->dataGTDf[0])
            {
                printf("\nError 65 en evaluarPob() llamando a malloc((%d+%d)*%d)",conf->headerGTD.numEntradas,conf->headerGTD.numSalidas,conf->numDatos*sizeof(float));
                return(0);
            }
            // coloca las filas de la matriz del buffer GTD apuntando al inicio de cada grupo de datos.
            for (i=1;i<conf->numDatos;i++)
            {
                //TODO: OJO!!!! Verificar si es sizeof(float)*(nIn+nOut)*i o (nIn+nOut)*i
                conf->dataGTDf[i]=conf->dataGTDf[0]+((conf->headerGTD.numEntradas+conf->headerGTD.numSalidas)*i);
            }
            // lee de GTD (numEntradas+numSalidas)*sizeof(float) en conf->dataGTD[i];
            leidos=fread(conf->dataGTDf[0],sizeof(float)*(conf->headerGTD.numEntradas+conf->headerGTD.numSalidas),conf->numDatos,fileIn);
            // verifica lectura.
            if (leidos<conf->numDatos)
            {
                printf("\nError 66 en evaluarPob() llamando a fread()");
                return(0);
            }
           // for (i=0;i<conf->numDatos;i++)
            //{
             //   printf("E0=%3.3f E1=%3.3f E2=%3.3f E3=%3.3f S0=%3.3f\n",conf->dataGTDf[i][0],conf->dataGTDf[i][1],conf->dataGTDf[i][2],conf->dataGTDf[i][4],conf->dataGTDf[i][5]);
            //}
        }
        // CUDA: reserva memoria en device CUDA para los datos de entrenamiento.
        hipMalloc(&(conf->cu_dataGTDf),(conf->headerGTD.numEntradas+conf->headerGTD.numSalidas)*conf->numDatos*sizeof(float));
        // coloca las filas de la matriz del buffer GTD apuntando al inicio de cada grupo de datos.
/*        for (i=1;i<conf->numDatos;i++)
        {
//TODO: OJO: SE DEBE HACER EN EL KERNEL.
            //TODO: OJO!!!! Verificar si es sizeof(float)*(nIn+nOut)*i o (nIn+nOut)*i
            conf->dataGTDf[i]=conf->dataGTDf[0]+((conf->headerGTD.numEntradas+conf->headerGTD.numSalidas)*i);
        }
*/
        // CUDA: reserva memoria en device para listaConexData[sizePob]
        hipMalloc(&(conf->cu_conexIn),conf->sizePob*sizeof(int*));
        // CUDA: reserva memoria en device para listaConexData[sizePob]
        hipMalloc(&(conf->cu_conexOut),conf->sizePob*sizeof(int*));
        // CUDA: reserva memoria en device para listaConexData[sizePob]
        hipMalloc(&(conf->cu_conexPeso),conf->sizePob*sizeof(float*));
        // CUDA: reserva memoria en device para tamListaConexData[sizePob]
        hipMalloc(&(conf->cu_tamListaConexPost),conf->sizePob*sizeof(int));
        // CUDA: reserva memoria en device para fitness[sizePob]
        hipMalloc(&conf->cu_fitness,conf->sizePob*sizeof(float));
        // CUDA: reserva memoria en device para valorC[sizePob] usado para calculo de fitness con correlaci�n.
        hipMalloc(&(conf->cu_valorC),conf->sizePob*sizeof(float*));
        for (i=0;i<conf->sizePob;i++)
            hipMalloc(&(conf->cu_valorC[i]),conf->numDatos*sizeof(float));
        // CUDA: reserva memoria en device para valorC[sizePob] usado para calculo de fitness con correlaci�n.
        hipMalloc(&(conf->cu_valorTr),conf->sizePob*sizeof(float*));
        for (i=0;i<conf->sizePob;i++)
            hipMalloc(&(conf->cu_valorTr[i]),conf->numDatos*sizeof(float));
        // CUDA: transfiere los datos de entrenamiento.dataGTDf.
        printf("Copiando datos de entrenamiento a dispositivo CUDA.\n");
        hipMemcpy(conf->cu_dataGTDf, conf->dataGTDf[0], sizeof(float)*(conf->headerGTD.numEntradas+conf->headerGTD.numSalidas)*conf->numDatos, hipMemcpyHostToDevice);
    }
    // si es o no primero
    // para i=0;i<numGenomas;i++
    for (i=0;i<numGenomas;i++)
    {
        // genera headerSNN[i]
        conf->headerSNN[i].actThreshold=conf->Fthreshold;
        conf->headerSNN[i].numEntradas=conf->numEntradas;
        conf->headerSNN[i].numSalidas=conf->numSalidas;
        conf->headerSNN[i].numBias=conf->numBias;
        conf->headerSNN[i].numHiddens=conf->pob[i].totalNodos-(conf->numEntradas+conf->numSalidas+conf->numBias);
        conf->headerSNN[i].numConex=conf->pob[i].totalConexiones;
        conf->headerSNN[i].sigmaFactor=conf->A;
        conf->headerSNN[i].tamRegistros=(conf->useFloat==1?4:8);
        conf->headerSNN[i].usarSigned=conf->usarSigned;
    }
    // llama a evalPob() (PARALELIZABLE(secci�n dentro de ella))
    evalPob(numGenomas, conf->numDatos, conf->headerGTD, conf->headerSNN, conf->dataGTDf, conf->listaConexData, conf->fitness, conf->ordenEval, conf->tamListaConexPost , conf);
    // coloca el fitness en cada genoma, para i=0;i<numGenomas;i++
    for (i=0;i<numGenomas;i++)
         conf->pob[i].fitness=conf->fitness[i];
       // printf("F%3.3f",conf->pob[i].fitness=conf->fitness[i]);
	
    return(1);
}

/*
unsigned evaluarPob(unsigned inicializar,unsigned primero,unsigned maxBufferSize, char *fileNameGTDv1, TConfig* conf)  // OPTIMIZADA, NMR
{
// eval�a toda la poblaci�n y deja el valor post fSigma en cada nodo.
// y calcula el fitness basado en el que se va acumulando con cada evaluaci�n de cada genoma.
// Los archivos de entrada y salida deben estar previamente abiertos para lectura binaria br
// retorna 0 si hay error, 1 si ok.
// //TODO: URGENTE PARA FX par�metro repeticiones para pasar los archivos de entrada repetidas veces por las redes neuronales al realizar las evaluaciones.
    unsigned i;
    unsigned j;
    int leidos=0;
    double pasadas=0; // usado para normalizar el fitness
    FILE* fileIn;
    //coloca los valores de cada neurona en 0 para comenzar la evaluaci�n. si incializar=1;
    //tambi�n Actualiza los punteros a conexHijo de cada nodo necesarios para evaluarGenoma
    if (primero==1)
    {
        //reserva memoria para sistema de ordenEval[][]
        // reserva memoria para conf->ordenEval[i<numGenomas][numNodos[i]]
        conf->ordenEval=(int**)malloc(conf->realSizePob*sizeof(int*));
        if (!conf->ordenEval)
        {
            printf("\nError 61.5 en .evaluarPob() llamando a malloc()");
            return(0);
        }
        // reserva memoria para m�ximo conf->maxNodos nodos por genoma
        for (i=0;i<conf->realSizePob;i++)
        {
            conf->ordenEval[i]=(int*)malloc(conf->maxNodos*sizeof(int));
            if (!conf->ordenEval[i])
            {
                printf("\nError 61.6 en .evaluarPob() llamando a malloc()");
                return(0);
            }
        }
        // reserva memoria para conf->contO[numGenomas] usado como contador para dentro de la funci�n recursiva
        conf->contO=(int*)malloc(conf->realSizePob*sizeof(int));
        if (!conf->contO)
        {
            printf("\nError 61.8 en .evaluarPob() llamando a malloc()");
            return(0);
        }
        // abre archivo GTDv1
        fileIn=fopen(conf->fileNameGTDv1,"rb");
        if (!fileIn)
        {
            printf("\nError 57.1 en evaluarPob() llamando a fopen()");
            return(0);
        }
        // lee headerGTD
        leidos=fread(&conf->headerGTD,sizeof(hdrGTDv1),1,fileIn);
        // busca el filesize
        if (leidos<1)
        {
            printf("\nError 58 en evaluarPob() llamando a fread()");
            return(0);
        }
        // averigua el fileSize y calcula conf->numDatos
        fseek(fileIn, 0, SEEK_END);
        conf->numDatos = (ftell(fileIn) - sizeof(hdrGTDv1))/(conf->headerGTD.tamRegistros*(conf->headerGTD.numEntradas+conf->headerGTD.numSalidas));
        if (conf->numDatos<1)
        {
            printf("\nError 65 en evaluarPob()llamando a ftell()");
            return(0);
        }
        //printf("\nFileSize=%d",ftell(fileIn));
        rewind(fileIn);
        // ubica en posici�n de lectura de datos despu�s del header.
        leidos=fread(&conf->headerGTD,sizeof(hdrGTDv1),1,fileIn);
        // reserva memoria en conf->dataGTD[conf->numDatos][numEntradas+numSalidas]
        // para todo el buffer, ojo! obtiene toda la memoria para el index 0 para que sean consecutivos
        // durante la lectura
        conf->dataGTDf=(float**)malloc(conf->numDatos*sizeof(float*));
        if (!conf->dataGTDf)
        {
            printf("\nError 64 en evaluarPob() llamando a malloc()");
            return(0);
        }
        conf->dataGTDf[0]=(float*)malloc((conf->headerGTD.numEntradas+conf->headerGTD.numSalidas)*conf->numDatos*sizeof(float));
        if (!conf->dataGTDf[0])
        {
            printf("\nError 65 en evaluarPob() llamando a malloc((%d+%d)*%d)",conf->headerGTD.numEntradas,conf->headerGTD.numSalidas,conf->numDatos*sizeof(float));
            return(0);
        }
        // coloca las filas de la matriz del buffer GTD apuntando al inicio de cada grupo de datos.
        for (i=1;i<conf->numDatos;i++)
        {
            //TODO: OJO!!!! Verificar si es sizeof(float)*(nIn+nOut)*i o (nIn+nOut)*i
            conf->dataGTDf[i]=conf->dataGTDf[0]+((conf->headerGTD.numEntradas+conf->headerGTD.numSalidas)*i);
        }
        // lee de GTD (numEntradas+numSalidas)*sizeof(float) en conf->dataGTD[i];
        leidos=fread(conf->dataGTDf[0],sizeof(float)*(conf->headerGTD.numEntradas+conf->headerGTD.numSalidas),conf->numDatos,fileIn);
        // verifica lectura.
        if (leidos<conf->numDatos)
        {
            printf("\nError 66 en evaluarPob() llamando a fread()");
            return(0);
        }
        fclose(fileIn);
    }
    if (inicializar==1)
    {
        for (i=0; i<conf->sizePob; i++)
        {
            //Actualiza los punteros a conecHijo de los nodos del genoma.
            if (actualizarPNodos(i,conf)==0)
            {
                fclose(conf->logFile);
                conf->logFile=fopen(conf->fileNameLog,"a+");
                fprintf(conf->logFile,"<br>\nError 1150 en fnci�n evaluarPob() llamando a actualizarPNodos()\n");
                return(0);
            }
            for (j=0; j<conf->pob[i].totalNodos; j++)
            {
                if (conf->pob[i].nodo[j].nodeFunction!=3)
                    conf->pob[i].nodo[j].valor=0;
            }
        }
    }
    //Inicializa el acumulador de fitness
    for (i=0; i<conf->sizePob; i++)
    {
        conf->pob[i].fitness=0;
    }
    fclose(conf->logFile);
    conf->logFile=fopen(conf->fileNameLog,"a+");
    fprintf(conf->logFile,"0");
    //hace repeticiones deben ser 0 o m�s
    for (i=0; i<conf->numDatos; i++)
    {
        pasadas = pasadas+1;
        //eval�a entradas y salidas para todos los genomas
        //TODO: ES m�s r�pido si a evaluargenoma se le pasa todo el buffer en lugar de dato a dato
        //con pragme 0,1,2=1:38s,  SIN=48s
        //#pragma omp parallel for
        for (j=0; j<conf->sizePob; j++)
        {
            if(evaluarGenoma(j,&(conf->pob[j]),primero, &(conf->dataGTDf[i][0]) ,&(conf->dataGTDf[i][conf->headerGTD.numEntradas]),conf)==0) ////TODO: HACER PARAMETRO GLOBAL conf->PRIMERO PARA DIFERENTES APLICACIONES
            {
                fclose(conf->logFile);
                conf->logFile=fopen(conf->fileNameLog,"a+");
                fprintf(conf->logFile,"<br>\nError 59 en funcion evaluarPob() llamando a evaluarGenoma(genoma=%u,i=%u )\n",j,i);
            }
        }
    }
    fclose(conf->logFile);
    conf->logFile=fopen(conf->fileNameLog,"a+");
    fprintf(conf->logFile,"1");
    // calcula los fitness a partir del acumulado de cada genoma
    if (pasadas>0)
        for (i = 0; i < conf->sizePob; i++)
        {
            //conf->pob[i].fitness /= (conf->tSigma > 1000 ? 2*(conf->numSalidas*pasadas) : conf->numSalidas*pasadas);
            conf->pob[i].fitness /= (conf->numSalidas*pasadas);
            conf->pob[i].fitness = 1.0 - conf->pob[i].fitness;
        }
    fclose(conf->logFile);
    conf->logFile=fopen(conf->fileNameLog,"a+");
    fprintf(conf->logFile,"2");
    // TODO: colocar como par�metro si copiar en cada evaluaci�n el mejor genoma de una especie en luagar del peor de la especie para que pueda mutar pesos,etc...
    // cierra archivos de entrada y salida
    fclose(conf->fIn);
    // libera los punteros usados por los punteros a las conexhijos usados durante eval (deben ser rearmados nuevamente con actualizarPNodos).
    for (i=0; i<conf->sizePob; i++)
    {
        for (j=0; j<conf->pob[i].totalNodos; j++)
        {
            if ((conf->pob[i].nodo[j].conexHijo!=NULL)&&(conf->pob[i].nodo[j].contHijos>0)&&(conf->pob[i].nodo[j].nodeFunction!=3))
            {
                free(conf->pob[i].nodo[j].conexHijo);
            }
        }
    }
    //libera memoria de las listas inputs y outputs
    fclose(conf->logFile);
    conf->logFile=fopen(conf->fileNameLog,"a+");
    fprintf(conf->logFile,"3");
    ////TODO: hacer Funcion bufferize para leer las entradas y salidas en arreglos globales para que se lean una sola vez del disco durante todo el ciclo ppal
    return(1);
}
*/
unsigned seleccionCrossover(TConfig* conf)  // OPTIMIZADA, NMR (No modifica representantes) TODO: falta mirar si formula de numHijos funcionam bi�n
{
    // realiza selecci�n y cruce en toda la poblaci�n
    // Inicializa el arreglo conf->numGenomasPorEspecie (conf->fInal de pag 394(426) de AI game prog) y lo modifica para reducir a la mitad el n�mero de especies
    // en conservaci�n. Y reparte el resto entre las especies que n� est�n en conservaci�n.
    // Requiere haber evaluado fitness y haber realizado especiacion.
    // Retorna 0 si hay error, 1 si ok
    // Par�metros:	porcentRedConserv = (entre 0 y 1) porcentaje de genomas que se quitan al n�mero calclulado para especies en conserv. recom=0.3
    // porcentElim = (entre 0 y 1) porcentaje de genomas que se eliminan en cada generaci�n, el restante porcentaje se reproduce por cruce.
    // unsigned  intentosPareja= n�mero de unsigned  intentos para buscar pareja aleatoriamente entre los padres antes de "matrimonio forzoso :) "
    // super = float entre 0 y 1, Probabilidad de heredar los exess y disjounsigned  ints del menos apto (aparte de los que se heredan normalmente del m�s apto)
    // promendiarProb = float entre 0 y 1 = probabilidad de que en caso de matching, se promedien los pesos en lugar de
    // seleccionarlos aleatoriamente entre los padres.
    unsigned i;
    unsigned j;
    unsigned dif=0;
    unsigned k=0;
    float avgTotal=0; // para almacenar la suma de los promedios de fitness de todas las especies
    unsigned verificacion =0; // usado para verificar si la sumatoria de conf->numGenomasPorEspecie==conf->sizePob
    unsigned variacion=0; // IGUAL QUE VERIFICACI�N
    unsigned tmp;
    unsigned huboSwap=1;// para reliazr ordenamiento de listaOrden
    unsigned sentido=1;// usado para ordenar listaOrden
    unsigned x;
    unsigned Ge=30005; // n�mero de genomas por especie (total de hijos de toda la especie)
    unsigned Gp=7;// n�mero de genomas padre.
    float m;
    unsigned y;
    unsigned acum=0;
    float correc=0.0001;
    unsigned sum=0;
    unsigned huboReprod=1; // usado para verificar reproducci�n de todos los padres
    unsigned posLMadre=0; // posici�n en la lista Ordenada de la madre
    unsigned posHijo=0; // indexpob del hijo
    unsigned especie1=0;
    unsigned especieCercana=0;

    // reserva memoria para *conf->fitnessAvgPorEspecie
    if ((conf->fitnessAvgPorEspecie=(float*)calloc(conf->spEspecies,(unsigned  int)sizeof(float)))==NULL)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 51 en funcion seleccionCruce() llamando a calloc(%u,%u)",conf->numEspecies,(unsigned  int)sizeof(float));
        return(0);
    }
    // inicializa  a a0
    for (i=0; i<conf->numEspecies; i++)
    {
        conf->fitnessAvgPorEspecie[i]=0;
    }
    // reserva memoria para conf->actNumGenomasPorEspecie de tama�o conf->numEspecies
    if ((conf->actNumGenomasPorEspecie=(unsigned  int*)calloc(conf->spEspecies,(unsigned  int)sizeof(unsigned  int)))==NULL)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 52 en funcion seleccionCruce() llamando a calloc(%u,%u)",conf->numEspecies,(unsigned  int)sizeof(unsigned  int));
        return(0);
    }
    // inicializa  a 0
    for (i=0; i<conf->numEspecies; i++)
    {
        conf->actNumGenomasPorEspecie[i]=0;
    }
    // reserva memoria para la matriz de indexpob por especie[f][c]
    // memoria para filas:
    if((conf->listaOrdenFitness=(unsigned  int**)calloc(conf->spEspecies,(unsigned  int)sizeof(unsigned  int*)))==NULL)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 53 en funcion seleccionCruce() llamando a malloc(%u)",conf->numEspecies*(unsigned int)sizeof(unsigned  int*));
        return(0);
    }
    // memoria para columnas y coloca el n�mero de hijos de todos los genomas en 0
    // coloca el n�mero de hijos para todos los genomas de la conf->poblaci�n en 0 antes de calcular su n�mero de hijos.
    for (i=0; i<conf->numEspecies; i++)
    {
        if((conf->listaOrdenFitness[i]=(unsigned  int*)calloc(1,sizeof(unsigned  int)*conf->sizePob))==NULL)
        {
            fclose(conf->logFile);
            conf->logFile=fopen(conf->fileNameLog,"a+");
            fprintf(conf->logFile,"<br>\nError 54 en funcion seleccionCruce() en asignacion conf->listaOrdenFitness[%u]=(unsigned  int*)malloc(%u)",i,conf->sizePob*(unsigned int)sizeof(unsigned  int));
            return(0);
        }
    }
    // inicializa actNumGenomas por epecie a 0
    // se calcula con formula el conf->numGenomasPorEspecie para cada especie, adem�s se llena la matriz de orden y conf->actNumGenomasPorEspecie
    for (i=0; i<conf->sizePob; i++) //se barre conf->pob
    {
        conf->listaOrdenFitness[conf->pob[i].especie][conf->actNumGenomasPorEspecie[conf->pob[i].especie]]=i; //lena el indexpob correspondiente en la lista de orden
        conf->fitnessAvgPorEspecie[conf->pob[i].especie]+=conf->pob[i].fitness; //acumula fitness por especie
        conf->actNumGenomasPorEspecie[conf->pob[i].especie]++; //se actualiza conf->actNumGenomasPorEspecie
    }
    // para el calculo se divide conf->fitnessAvgPorEspecie entre conf->actNumGenomasPorEspecie correspondiente para obtener el conf->fitnessAvgPorEspecie
    for (i=0; i<conf->numEspecies; i++) //se barren las especies
    {
        conf->fitnessAvgPorEspecie[i]/=(float)conf->actNumGenomasPorEspecie[i];
        avgTotal+=conf->fitnessAvgPorEspecie[i];
    }
    // verifica si alguno de los genomas tiene el m�nimo de fitness requerido para tener minPorcentGenomasPorEspecie y si no lo asigna y recalcula el avgTotal
    for (i=0; i<conf->numEspecies; i++)
    {
        if ((conf->fitnessAvgPorEspecie[i]/avgTotal)<conf->minPorcentGenomasPorEspecie)
        {
            conf->fitnessAvgPorEspecie[i]=conf->minPorcentGenomasPorEspecie;
        }
    }

    /** PROBANDO!!!! CAMBIADO El fitness sharing para distribuci�n de poblaci�n, ahora depende del fitness solo del rep no del prom de todos
        // recalcula el n�mero de genomas por especie.
        avgTotal=0;
        for (i=0; i<conf->numEspecies; i++) //se barren las especies
        {
            avgTotal+=conf->fitnessAvgPorEspecie[i];
        }
            //calcula el fitness para cada especie dependiendo de el fitness de la especie y el fitness total


    //TODO:formula cambiada buscando bug de ciclo infinito         conf->numGenomasPorEspecie[i]=(unsigned  int)floor(((conf->fitnessAvgPorEspecie[i]/avgTotal)*(float)conf->sizePob)+0.5); //calcula el n�mero de genomas para cada especie

        for (i=0; i<conf->numEspecies; i++) //se barren las especies
        {
            conf->numGenomasPorEspecie[i]=(unsigned  int)floor((conf->fitnessAvgPorEspecie[i]/avgTotal)*(float)conf->sizePob)+0.5; //calcula el n�mero de genomas para cada especie
            verificacion+=conf->numGenomasPorEspecie[i];
        }
    */
//INICIO DE PRUEBA
    // recalcula el n�mero de genomas por especie.
    avgTotal=0;
    for (i=0; i<conf->numEspecies; i++) //se barren las especies
    {
        avgTotal+= conf->pob[conf->representantes[i]].fitness;
    }
    //calcula el fitness para cada especie dependiendo de el fitness de la especie y el fitness total


//TODO:formula cambiada buscando bug de ciclo infinito         conf->numGenomasPorEspecie[i]=(unsigned  int)floor(((conf->fitnessAvgPorEspecie[i]/avgTotal)*(float)conf->sizePob)+0.5); //calcula el n�mero de genomas para cada especie

    for (i=0; i<conf->numEspecies; i++) //se barren las especies
//TODO: probando para debugging de sigerr, antes no se le restaba numespecies a sizePob y solo se le sumaba 0.5 en lubgar de 1.5
    {
        conf->numGenomasPorEspecie[i]=(unsigned  int)floor((conf->pob[conf->representantes[i]].fitness/avgTotal)*(float)(conf->sizePob-(3*conf->numEspecies)))+3.5; //3 porque 1 champios,2 distrib, 3 worker//calcula el n�mero de genomas para cada especie
        verificacion+=conf->numGenomasPorEspecie[i];
    }
//FIN DE PRUEBA
    //verifica que la sumatoria de numgenomas por especie coincida con conf->sizePob, si es superior, resta genomas barriendo a cada especie en conf->numGenomasPorEspecie
    i=0;
    if (verificacion>conf->sizePob) //si hay m�s que los debidos
    {
        variacion=verificacion-conf->sizePob;
        if (conf->numGenomasPorEspecie[i]>1)
            while(variacion--)  //Disminuye uniformemente el conf->numGenomasPorEspecie variaci�n veces.
            {
                conf->numGenomasPorEspecie[i++]--;
                if(i==conf->numEspecies)
                    i=0;
            }
    }
    if (verificacion<conf->sizePob) //si hay menos que los debidos
    {
        variacion=conf->sizePob-verificacion;
        while(variacion--)  //Incrementa uniformemente el conf->numGenomasPorEspecie variaci�n veces.
        {
            conf->numGenomasPorEspecie[i++]++;
            if(i==conf->numEspecies)
                i=0;
        }
    }

    // imprime n�mero de genomas por especie
    /*   for (j=0; j<conf->numEspecies; j++)
       {
           fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>E%i=%u,",j,conf->numGenomasPorEspecie[j]);
       } */
    // se ordena la conf->listaOrdenFitness para cada especie con prioridad parael fitness y segunda prio la simplicidad (numConex)
    // poco sofisticado, pero funciona.

    for (i=0; i<conf->numEspecies; i++) //barre las especies
    {
        huboSwap=1;//para que entre la primera vez
        while(huboSwap)
        {
            huboSwap=0;
            // TODO: optimizar con punteros y variables toda esta secci�n
            if(sentido==1) //para ordenamiento descendente desde el primer elemento
            {
                for (j=0; j<(conf->actNumGenomasPorEspecie[i]-1); j++) //Barre los index de la especie
//TODO: quitar cuando corrijamos bug hang por max uint (desbordamiento)
                    if ((conf->actNumGenomasPorEspecie[i]-1)<=conf->sizePob)
                    {
                        if (conf->pob[conf->listaOrdenFitness[i][j]].fitness<conf->pob[conf->listaOrdenFitness[i][j+1]].fitness)
                        {
                            huboSwap=1;
                            swap(&(conf->listaOrdenFitness[i][j]),&(conf->listaOrdenFitness[i][j+1]));
                        }
                        if (conf->pob[conf->listaOrdenFitness[i][j]].fitness==conf->pob[conf->listaOrdenFitness[i][j+1]].fitness)
                        {
                            if(conf->pob[conf->listaOrdenFitness[i][j]].totalConexiones>conf->pob[conf->listaOrdenFitness[i][j+1]].totalConexiones)
                            {
                                huboSwap=1;
                                swap(&(conf->listaOrdenFitness[i][j]),&(conf->listaOrdenFitness[i][j+1]));
                            }
                        }
                    }
            }
            else //para ordenamiento ascendente desde el �ltimo elemento
            {
                for (j=(conf->actNumGenomasPorEspecie[i]-1); j>=1; j--) //Barre los index de la especie
//TODO: quitar cuando corrijamos bug hang por max uint (desbordamiento)
                    if (((conf->actNumGenomasPorEspecie[i]-1)<conf->sizePob)&&(j<=conf->sizePob))
                    {

                        if (conf->pob[conf->listaOrdenFitness[i][j-1]].fitness<conf->pob[conf->listaOrdenFitness[i][j]].fitness)
                        {
                            huboSwap=1;
                            swap(&(conf->listaOrdenFitness[i][j-1]),&(conf->listaOrdenFitness[i][j]));
                        }

                        if (conf->pob[conf->listaOrdenFitness[i][j-1]].fitness==conf->pob[conf->listaOrdenFitness[i][j]].fitness)
                        {
                            if(conf->pob[conf->listaOrdenFitness[i][j-1]].totalConexiones>conf->pob[conf->listaOrdenFitness[i][j]].totalConexiones)
                            {
                                huboSwap=1;
                                swap(&(conf->listaOrdenFitness[i][j-1]),&(conf->listaOrdenFitness[i][j]));
                            }
                        }

                    }

            }
            sentido*=-1; //invierte el sentido de la b�squeda
        }

    }


    //reorganiza listaorden: SOBREESCRIBE genomas faltantes en para que numgenomasPE y actnum sean iguales //TODO: verificar si esto es necesario.
    for (i=0; i<conf->numEspecies; i++)
    {
        if(conf->actNumGenomasPorEspecie[i]<conf->numGenomasPorEspecie[i])
        {
            dif=conf->numGenomasPorEspecie[i]-conf->actNumGenomasPorEspecie[i];
            while(dif)
            {
                for (j=0; j<conf->numEspecies; j++)
                {
                    if(conf->actNumGenomasPorEspecie[j]>conf->numGenomasPorEspecie[j])
                    {
                        dif--;
                        conf->actNumGenomasPorEspecie[j]--;
                        conf->listaOrdenFitness[i][conf->actNumGenomasPorEspecie[i]]=conf->listaOrdenFitness[j][conf->actNumGenomasPorEspecie[j]];
                        copiarGenoma(conf->listaOrdenFitness[i][0],conf->listaOrdenFitness[i][conf->actNumGenomasPorEspecie[i]], conf);
                        conf->actNumGenomasPorEspecie[i]++;
                        if (dif==0)
                            j=conf->numEspecies;
                    }
                }
            }
        }
    }

    //TODO: quitar esta parte SIGUIENTE  cuando funcione bi�n
    //verifica que la suma de numGenomasPorEspecie sea igual a sizePob
    acum=0;
    for (i=0; i<conf->numEspecies; i++)
    {
        acum+=conf->actNumGenomasPorEspecie[i];

    }
    if (acum!=conf->sizePob)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError an SeleccionCrossover() tama�o de la pob (%u) no coincide con suma de numGenomasxespecie(%u).",conf->sizePob,acum);
        return(0);
    }
    // AsignarNumHijos: se coloca descargando del total de hijos por especie calculado, el numHijos para cada genoma
    // Con porcentElim calcula el n�mero de genomas por especie que se deben reproducir y les asigna su respectivo numHijos.
    // corresponde a un segmento de linea con pendiente negativa entre x=0 y x=genomasPadre y con unsigned  integral=conf->numGenomasPorEspecie.
    // se hace barriendo la conf->listaOrdenFitness para cada especie y asignando el numHijos descontando un tanto de totalHijosPorEspecie-1 (debido a que el came�n se conserva y el numhijos se disminuye en 1) y si =0,
    //TODO: comprobar si esta formula funciona y leer y remover si es neceario el comment de arriba
    for (i=0; i<conf->numEspecies; i++) //barre las especies
    {
        Ge=conf->numGenomasPorEspecie[i]-1;// era -1 al finaln�mero de genomas para reemplazar (todos los de la especie menos el champion)
        if (Ge==-1)
        {
            fclose(conf->logFile);
            conf->logFile=fopen(conf->fileNameLog,"a+");
            fprintf(conf->logFile,"<br>\nError 23 en seleccionCrossover(), numGenomasPorEspecie[%d]=%d\n",i,conf->numGenomasPorEspecie[i]);
            return(0);
        }
        sum=0;
        Gp=(unsigned  int)floor((float)Ge*(1-conf->porcentElim)+0.5);//n�mero padres por especie;
        for (x=1; x<=Gp; x++) //calcula el decremento de hijos por genoma.
        {
            sum+=x;
        }
        m=(float)Ge/(float)sum;// decremento de hijos por genoma
        acum=0;// para realizar verificaci�n de n�mero correcto de genomas.
        for (x=Gp; x>0; x--) //calcula el n�mero de hijos por genoma
        {
            y=(unsigned  int)floor(((m*(float)x)+0.5)+correc); //n�mero de hijos para genoma Gp-x de especie i
            conf->pob[conf->listaOrdenFitness[i][Gp-x]].numHijos=y;
            // fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>y(%u) = %u = %3.5f, ",Gp-x,y,(m*(float)x+0.5)+b-0.5+correc);
            acum+=y;
        }
        acum=Ge-acum;//calcula cuantos genomas sobraron o faltaron y los descuenta del champion.
        if(acum!=0) conf->pob[conf->listaOrdenFitness[i][0]].numHijos+=acum;
    }

    //imprimirSeleccion(conf);

// VERIFICAR QUE LOS HIJOS SUMEN FITNESS CORRECTO
    //for (x=0;x<Gp)
    //Se hace reproducci�n :))))  :)
    //para esto se barre la conf->listaOrdenFitness por cada especie y se realiza cruce entre padre y una madre aleatoria  (n unsigned  intentos) que tenga
    //el numHijos>0 si no se encuentra aleatorio, se barre de izq a der en busqueda.
    //Si el numHijos del PADRE es 1, se reemplaza el indexpob del padre por el hijo y se coloca numHijos=-1.
    //sin�, se coloca el resultado reemplazando a un genoma que tenga numHijos=0 EN TODA LA conf->pob (si despu�s del barrido no se encuentra (pej �ltimogenoma))
    //se toma como madre el genoma campe�n (index 0 de listaorden) y se reemplaza el del padre.
    //con cada reproducci�n se disminuye el numHijos del padre. si es =0 se pasa al pr�ximo genoma de listaOrden.
    //Si el genoma es un campe�n de especie, y su numHijos=1 se reproduce, se bua un numHijos=0 en conf->pob y se coloca ahi a su hijo, el numHijos
    //del campe�n se hace -1 para evitar que sea reemplazado por otros cruces y se conserve unsigned  intacto.
    //si no se encuentra madre con numHijos>0, se reproduce con champion y hijo sustituye a padre.
    for (i=0; i<conf->numEspecies; i++) //barre especies
    {
        if (conf->contGeneracSinMejora[i]<conf->maxGenParaNoCruce)
        {
            fclose(conf->logFile);
            conf->logFile=fopen(conf->fileNameLog,"a+");
            fprintf(conf->logFile,"%d",i);
            Gp = (unsigned  int)floor(((float)conf->numGenomasPorEspecie[i]-1)*(1-conf->porcentElim));//n�mero padres por especie;
            huboReprod=1; //inicializa para cada especie el huboReprod.
            // numHijos=0 para todos los dem�s genomas de listaOrdenFitness a partir de Gp
            for (k=Gp; k<conf->numGenomasPorEspecie[i]; k++)
                conf->pob[conf->listaOrdenFitness[i][k]].numHijos=0;


            while(huboReprod) //si hubo reproducci�n en el ciclo anterior
            {
                huboReprod=0;
                for (j=0; j<Gp; j++) //barre padres
                {
                    if (conf->pob[conf->listaOrdenFitness[i][j]].numHijos>0) //si el padre elegido tiene numHijos>0
                    {
                        tmp=conf->intentosPareja;
                        posLMadre=(unsigned  int)floor((Gp*((float)randL(conf)))+0.5000001);//posMadre=busca aleatoriamente entre 0 y Gp una madre se coloca por si unsigned  intentos=0
                        while (tmp--)  //mientras haya unsigned  intentos disponibles
                        {
                            if (Gp>1) // para evitar ciclo infinito y para evitar que los dos padres sean el mismo genoma
                                while((posLMadre=(unsigned  int)floor((Gp*((float)randL(conf)))+0.5000001))==j);//mientras ((posMadre=aleatoriio(0,Gp))==PosPadre);
                            if (conf->pob[conf->listaOrdenFitness[i][posLMadre]].numHijos>0)//si madre.numHijos>0 tmp=0
                                tmp=0;
                        }
                        if (conf->pob[conf->listaOrdenFitness[i][posLMadre]].numHijos==0) //si madre.numHijos==0
                        {
                            for (k=0; k<Gp; k++) //para k=0;k<Gp;k++ si no encontr� madre aleatoriamente, la busca secuencialmente
                            {
                                if (conf->pob[conf->listaOrdenFitness[i][k]].numHijos>0)//si si posK.numHijos>0
                                    if (j!=k)//si posK!=j,posMadre=k
                                        posLMadre=k;
                            }
                        }
                        if (conf->pob[conf->listaOrdenFitness[i][posLMadre]].numHijos>0) //si madre.numHijos>0
                        {
                            posHijo=UINT_MAX;//posHijo=-1
                            if (conf->pob[conf->listaOrdenFitness[i][j]].numHijos==1) //si padre.numHijos==1
                            {
                                if (j>0) //si posPadre>0 posHijo=listaOrden[i][j]
                                    posHijo = conf->listaOrdenFitness[i][j];
                                else // coloca el n�mero de hijos en -1 para el representante
                                    conf->pob[conf->listaOrdenFitness[i][j]].numHijos=-1;//sino padre.numHijos=-1 porque ee el champion
                            }

                            // adicionado para  competencia: busca posHijo y verifica que el index de un representante no sea escogido como hijo
                            // si posHijo==-1 si a�n no se tiene posici�n para el hijo busca en especie
                            if (posHijo==UINT_MAX)
                            {
                                if ((conf->numGenomasPorEspecie[i]-1)<conf->sizePob)
                                    for (k=(conf->numGenomasPorEspecie[i]-1); k>1; k--)
                                    {
                                        //para no sobreescribir el champion, se hace hasta k>0 no=0
                                        if(conf->pob[conf->listaOrdenFitness[i][k]].numHijos==0)
                                        {
                                            posHijo=conf->listaOrdenFitness[i][k];
                                            k=1; //sale del for
                                        }
                                    }
                            }

                            // si posHijo==-1 si a�n no se tiene posici�n para el hijo, busca en pob
                            if (posHijo==UINT_MAX)
                            {
                                for (k=0; k<conf->sizePob; k++) //busca primero entre los de su especie, luego busca entre toda la conf->poblaci�n un genoma que tenga numHijos=0 y guarda su indexpob en posHijo
                                    if(conf->pob[k].numHijos==0)
                                    {
                                        if(conf->representantes[conf->pob[k].especie]!=k)
                                        {
                                            posHijo=k;
                                            k=conf->sizePob; //sale del for
                                        }
                                    }
                            }

                            if (posHijo!=UINT_MAX) //si posHijo!=-1 es decir, si ya se tiene posici�n para el hijo.
                            {
                                if(crossover(conf->listaOrdenFitness[i][j],conf->listaOrdenFitness[i][posLMadre],posHijo,conf->super,conf->promediarPeso,conf->porcentEnableds, conf)==0) //crossover(indexpobPadre,indexpobMadre,indexpobHijo,super,promediarPob);
                                {
                                    fclose(conf->logFile);
                                    conf->logFile=fopen(conf->fileNameLog,"a+");
                                    fprintf(conf->logFile,"<br>\nError 55 en funcion seleccionCruce() llamando a crossover(%u,%u,%u,%1.1f,%1.1f)",conf->listaOrdenFitness[i][j],conf->listaOrdenFitness[i][posLMadre],posHijo,conf->super,conf->promediarPeso);
                                    return(0);
                                }

                                huboReprod=1;//huboReproducci�n=1
                                conf->pob[conf->listaOrdenFitness[i][j]].numHijos--;//padre.numHijos--

                                conf->pob[posHijo].numHijos=-1;//genomaindexpobHijo.numhijos=-1
                            }

                        }
                        else //sino (no se encontr� en especie madre con numhijos>0)
                        {
                            posLMadre=0;//posMadre=champion
                            if (j>0)
                            {
                                posHijo=conf->listaOrdenFitness[i][j];//posHijo=posPadre
                                //crossover(indexpobPadre,indexpobMadre,indexpobHijo,super,promediarPob);
                                if(crossover(conf->listaOrdenFitness[i][j],conf->listaOrdenFitness[i][posLMadre],posHijo,conf->super,conf->promediarPeso,conf->porcentEnableds, conf)==0) //crossover(indexpobPadre,indexpobMadre,indexpobHijo,super,promediarPob);
                                {
                                    fclose(conf->logFile);
                                    conf->logFile=fopen(conf->fileNameLog,"a+");
                                    fprintf(conf->logFile,"<br>\nError 56 en funcion seleccionCruce() llamando a crossover(%u,%u,%u,%1.1f,%1.1f)",conf->listaOrdenFitness[i][j],conf->listaOrdenFitness[i][posLMadre],posHijo,conf->super,conf->promediarPeso);
                                    return(0);
                                }

                                huboReprod=1;//huboReproducci�n=1
                                conf->pob[conf->listaOrdenFitness[i][j]].numHijos--;//padre.numHijos--
                                conf->pob[posHijo].numHijos=-1;//genomaindexpobHijo.numhijos=-1
                            }//MODIFICADO PARA EVITAR REEMPLAZAR A CHAMPION
                        }
                    }
                }
            }
        }
    }
    // realiza cruce inter-especies.
    if ((((float)randL(conf))<(conf->probInterSp*conf->sizePob))&&(conf->numEspecies>1))
    {
        // escoge una especie al azar
        especie1 = (conf->numEspecies-1)*((float)randL(conf));
        // busca su especie m�s cercana
        especieCercana = especieMinDist(conf->representantes[especie1],conf->c1,conf->c2,conf->c3,conf->eG_t,conf);
        // busca en pob un genoma que no sea el rep de espCercana
        i = 0;
        while (((i < conf->sizePob)&&(conf->pob[i].especie!=especieCercana))||(i==conf->representantes[especieCercana]))
            i++;
        if (i>=conf->sizePob)
        {
            fclose(conf->logFile);
            conf->logFile=fopen(conf->fileNameLog,"a+");
            fprintf(conf->logFile,"<br>\nError 55.9 en seleccionCruce() no hay suficientes integrantes de la especie.");
        }
        else
        {
            // hace cruce de rep[esp1] y buscadoesp2 y deja al hijo en buscadoesp2
            if(crossover(conf->representantes[especie1],i,i,conf->super,conf->promediarPeso,conf->porcentEnableds, conf)==0) //crossover(indexpobPadre,indexpobMadre,indexpobHijo,super,promediarPob);
            {
                fclose(conf->logFile);
                conf->logFile=fopen(conf->fileNameLog,"a+");
                fprintf(conf->logFile,"<br>\nError 55.92 en funcion seleccionCruce() llamando a crossover(%u,%u,%u,%1.1f,%1.1f)",conf->listaOrdenFitness[i][j],conf->listaOrdenFitness[i][posLMadre],posHijo,conf->super,conf->promediarPeso);
                return(0);
            }
            // busca en pob un genoma que no sea el rep de especie1
            i = 0;
            while (((i < conf->sizePob)&&(conf->pob[i].especie!=especie1))||(i==conf->representantes[especie1]))
                i++;
            if (i>=conf->sizePob)
            {
                fclose(conf->logFile);
                conf->logFile=fopen(conf->fileNameLog,"a+");
                fprintf(conf->logFile,"<br>\nError 55.93 en seleccionCruce() durante cruce intersp.");
                return(0);
            }
            // hace cruce de rep[espCercana] y buscadoEsp1 y deha al hijo en buscadoEsp1
            if(crossover(conf->representantes[especieCercana], i, i, conf->super, conf->promediarPeso,conf->porcentEnableds, conf)==0) //crossover(indexpobPadre,indexpobMadre,indexpobHijo,super,promediarPob);
            {
                fclose(conf->logFile);
                conf->logFile=fopen(conf->fileNameLog,"a+");
                fprintf(conf->logFile,"<br>\nError 55.94 en funcion seleccionCruce() llamando a crossover(%u,%u,%u,%1.1f,%1.1f)",conf->listaOrdenFitness[i][j],conf->listaOrdenFitness[i][posLMadre],posHijo,conf->super,conf->promediarPeso);
                return(0);
            }
            // imprimeISP
            fclose(conf->logFile);
            conf->logFile=fopen(conf->fileNameLog,"a+");
            fprintf(conf->logFile,"I");
        }
    }
    // libera memoria usada por todos los punteros
    for (i=0; i<conf->numEspecies; i++)
    {
        if (conf->listaOrdenFitness[i]!=NULL) free(conf->listaOrdenFitness[i]);
    }
    if (conf->listaOrdenFitness!=NULL) free(conf->listaOrdenFitness);
    if (conf->fitnessAvgPorEspecie!=NULL) free(conf->fitnessAvgPorEspecie);
    if (conf->actNumGenomasPorEspecie!=NULL) free(conf->actNumGenomasPorEspecie);
    // si no hubo errores, retorna 1
    return(1);
}

float especiacion(TConfig* conf)  // OPTIMIZADA, NMR //TODO: optimizar punteros
{
//realiza la asignaci�n de especies para toda la pob, el exterminio y tambi�n actualiza la liste de representantes de especies y el contador de generaciones sin mejora.
//copia el representante de cada especie a la posici�n sizePob+especie de la pob, sie el fitness es superior
//en caso contrario, copia el representante guardado en lugar del representante actual para conservar los cambios.
//para permitir su mutaci�n y reproducci�n normal en la poblaci�n sin perder info del mejor fitness.
//Se debe realizar DESPUES de la evaluaci�n.
//Altera el valor de threshold en un m�ximo porcentaje especificado para alcanzar el n�mero de especies requerido.
//si el n�mero de especies requerido ya se alcanz�,  no crea nuevas especies sino que asigna a caga genoma la especie m�s cercana.
//actualiza en cada ejecuci�n la lista de conf->representantes y el n�mero de generaciones sin mejora de fitness por especie.
//tambi�n actualiza la lista de especies en conservaci�n.
//Par�metros:	sPob = n�mero de genomas en la conf->poblaci�n
//				numEspDeseadas = n�mero de especies deseadas
//				numConserv = n�mero de especies en conservaci�n.
//				threshold = threshold inicial
//				porcentVarTh = entre 0 y 1 = porcentaje de variaci�n del threshold si no se ha alcanzado el m�ximo n�mero de especies.
//				c1= constante de proporcionalidad en distencia para n�mero de genes excess entre los padres
//				c2= constante de proporcionalidad en distancia para n�mero de genes disjounsigned entre los padres
//				c3= constante de proporcionalidad en distancia para promedio de diferencias de pesos en matching genes de los padres
//				eG_t=(creo que no es necesario REVISAR) n�mero de genes necesarios para considerar el genoma suficientemente grande y hacer n=1
//retorna 0 si hay error, 1 si ok
    unsigned i;
    unsigned j=0;
    unsigned mejor=0;
    //TODO : FALTA: Garantizar randomizando pesos que la distancia al m�s cercano sea mayora la m�nima
    //      se puede utilizar u ciclo hasta m�ximo de veces o >th y retornar el m�s lejano encontrado
    //      y si es mayor del m�nimo aumenta el threshold, sin� lo disminuyen para mentener
    //      el threshold de distancia �ptimo para cualquier n�mero de especies.
    // quitar este decremento,  solo copiar el genoma a cada (ver primeraGEn y exterminio)
    // si conf->numEspecies<conf->numEspecies deseadas decrementa el threshold en porcentVarTh (para que aparezcan como nuevas al ser diferentes a las actuales)
    if (conf->numEspecies < conf->spEspecies)
    {
        conf->threshold = conf->threshold * (1-conf->porcentVarTh);
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br> Eth=%2.2f",conf->threshold);
    }
    // TODO: Verificar si esto es necesario, sino, quitarlo.
    if (conf->numEspecies > conf->spEspecies) 	//AGREGAO HOY TAMBI�N: //TODO:manejar sin modificaci�n del threshold cuando conf->numEspecies>spEspecies para conservar fijo el n�mero de especies si se extermina una especie y se distribuye su poblaci�n de alguna manera/
    {
        conf->threshold = conf->threshold * (1+conf->porcentVarTh);
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nIncrementando threshold de compatibilidad de especies, nuevo valor = %3.3f",conf->threshold);
    }
    //TODO: verificar si se puede hacer el n�mero de especies completamente din�mico controlando el n�mero de especies con el threshold y agrgando o extinguiendo una especie.
    // incrementa el n�mero de generaciones sin mejora para todas las especies
    for (i = 0; i < conf->numEspecies; i++)
        conf->contGeneracSinMejora[i]++;
    // recalcula los representantes
    for (i = 0; i < conf->sizePob; i++)
    {
        if (conf->pob[i].fitness>conf->pob[conf->representantes[conf->pob[i].especie]].fitness)
        {
            //conf->representantes[conf->pob[i].especie]=i
            conf->representantes[conf->pob[i].especie]=i;
            conf->contGeneracSinMejora[conf->pob[i].especie]=0;
        }
    }

    // para toda la pob us asignarEspecies() para actualizar la especie a la mas cercana para el genoma, excepto para los reps.
    for (i=0; i<conf->sizePob; i++)
    {
        //si conf->numEspecies<conf->spEspecies
        if (conf->numEspecies<conf->spEspecies)
        {
            //asignarEspecie(i)
            //si i no es un representante de especie:
            if (conf->representantes[conf->pob[i].especie]!=i)
                conf->pob[i].especie=asignarEspecie(i,conf->threshold,conf->c1,conf->c2,conf->c3,conf->eG_t,conf);
        }
        else
        {
            //sino
            //conf->pob[i].especie = epecieMindist
            //si i no es un representante de especie:
            if (conf->representantes[conf->pob[i].especie]!=i)
                conf->pob[i].especie=especieMinDist(i,conf->c1,conf->c2,conf->c3,conf->eG_t,conf);
            //si conf->pob[i].fitness>conf->pob[conf->representantes[conf->pob[i].especie]].fitness
        }
    }

//	for (i=0;i<conf->numEspecies;i++)
//		conf->conservacionEsp[i]=i;
    //Ordena por fitness el arreglo conf->representantes en el arreglo conservaci�n, donde el index 0 es el de mayor fitness
    //para obtener en conservaci�nEsp en el index 0 el index de especie que tenga m�s fitness y decrementalmente hasta conf->numEspecies-1
    //si dos conf->representantes tienen el m�smo fitness, la mejor posici�n es la del que tenga menor totalConexiones
    /*
    	while(huboSwap){ //repite hasta que el arreglo conf->conservacionEsp est� ordenado (por fitness)
    		//recorre haciendo burbuja down de 0 a conf->numEspecies-1
    		huboSwap=0;
    		if (sentido==1){
    			for (i=0;i<(conf->numEspecies-1);i++){
    				if (conf->pob[conf->representantes[conf->conservacionEsp[i]]].fitness<conf->pob[conf->representantes[conf->conservacionEsp[i+1]]].fitness){
    					huboSwap=1;
    					swap(&(conf->conservacionEsp[i]),&(conf->conservacionEsp[i+1]));
    				}
    				if (conf->pob[conf->representantes[conf->conservacionEsp[i]]].fitness==conf->pob[conf->representantes[conf->conservacionEsp[i+1]]].fitness){
    					if(conf->pob[conf->representantes[conf->conservacionEsp[i]]].totalConexiones>conf->pob[conf->representantes[conf->conservacionEsp[i+1]]].totalConexiones){
    						huboSwap=1;
    						swap(&(conf->conservacionEsp[i]),&(conf->conservacionEsp[i+1]));
    					}
    				}
    			}
    		}
    		else{
    			//recorre haciendo burbuja up de conf->numEspecies-1 hasta 0
    			for (i=(conf->numEspecies-2);i>=0;i--){
    				if (conf->pob[conf->representantes[conf->conservacionEsp[i]]].fitness<conf->pob[conf->representantes[conf->conservacionEsp[i+1]]].fitness){
    					huboSwap=1;
    					swap(&(conf->conservacionEsp[i]),&(conf->conservacionEsp[i+1]));
    				}
    				if (conf->pob[conf->representantes[conf->conservacionEsp[i]]].fitness==conf->pob[conf->representantes[conf->conservacionEsp[i+1]]].fitness){
    					if(conf->pob[conf->representantes[conf->conservacionEsp[i]]].totalConexiones>conf->pob[conf->representantes[conf->conservacionEsp[i+1]]].totalConexiones){
    						huboSwap=1;
    						swap(&(conf->conservacionEsp[i]),&(conf->conservacionEsp[i+1]));
    					}
    				}
    			}

    		}
    		sentido=sentido*-1;

    	}
    		//coloca en 0 el conf->contGeneracSinMejora[] para las especies que se encuentran en conservaci�n, para que cuando salgan de conservaci�n
    		//tengan mejores probabilidades de mejorar antes de ser eliminadas.
    		//fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nEspecies en Conservacion = ");
    		for (i=0;i<conf->numConservacion;i++){
    			conf->contGeneracSinMejora[conf->conservacionEsp[i]]=0;
    			fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>%u(%2.5f), ",conf->conservacionEsp[i],conf->pob[conf->representantes[conf->conservacionEsp[i]]].fitness);
    		}
    		//Si las especies superan maxGenSinMejora coloca contGeneracSinMejora en 0 y coloca todos los genomas dde la espeie i como genomas Iniciales
    		*/
    // verifica si alguna especie debe se exterminada excepto la que tiene el mejor fitness.
    if ((mejor=buscarMejorFitness(conf)) == UINT_MAX)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 34.45 en Funcion especiacion() llamando a buscarMejorFitness()");
        return(0);
    }
    for (i=0; i<conf->numEspecies; i++)
    {
        if (conf->pob[mejor].especie!=i)
        {
            if (conf->contGeneracSinMejora[i]>conf->maxGeneracSinMejora)
            {
                // coloca generacSinMejora en 0
                fclose(conf->logFile);
                conf->logFile=fopen(conf->fileNameLog,"a+");
                fprintf(conf->logFile,"<br>X%2u ",i);
                conf->contGeneracSinMejora[i]=0;
                // sustituye el representante de la especie por un genoma inicial
                if (genomaInicial(conf->representantes[i],conf->numEntradas,conf->numSalidas,conf->numBias,0,i,conf)==0)
                {
                    fclose(conf->logFile);
                    conf->logFile=fopen(conf->fileNameLog,"a+");
                    fprintf(conf->logFile,"<br>\nError 34.5 en Funcion especiacion() llamando a genomaInicial(%u,%u,%u)",conf->numEntradas,conf->numSalidas,conf->numBias);
                    return(0);
                }
                //TODO : FALTA: Garantizar randomizando pesos que la distancia al m�s cercano sea mayora la m�nima
                //      se puede utilizar u ciclo hasta m�ximo de veces o >th y retornar el m�s lejano encontrado
                //      y si es mayor del m�nimo aumenta el threshold, sin� lo disminuyen para mentener
                //      el threshold de distancia �ptimo para cualquier n�mero de especies.
                if (randomizarPesos(conf->representantes[i],conf->pesoMinInicial,conf->pesoMaxInicial,conf)==0)
                {
                    fclose(conf->logFile);
                    conf->logFile=fopen(conf->fileNameLog,"a+");
                    fprintf(conf->logFile,"<br>\nError 35 en Funcion primeraGen() llamando a randomizarPesos(%u,%1.1f,%1.1f)\n",j,conf->pesoMinInicial,conf->pesoMaxInicial);
                    return(0);
                }
                // muta el genoma inicial AN+AC mutacionesPorExterminio Veces.
                for (j=0; j<conf->mutacionesPorExterminio; j++)
                {
                    // aplica mutaci�n AN al genoma copiado
                    if (mutarAN(conf->representantes[i],conf)==0)
                    {
                        fclose(conf->logFile);
                        conf->logFile=fopen(conf->fileNameLog,"a+");
                        fprintf(conf->logFile,"<br>\nError 34.6 en funcion especiacion() llamando a mutarAN(%u)\n",i);
                        return(0);
                    }
                    // aplica mutaci�n AC al genoma copiado
                    if (mutarAC(conf->representantes[i],conf->maxIntentosMutarAC,conf)==0)
                    {
                        fclose(conf->logFile);
                        conf->logFile=fopen(conf->fileNameLog,"a+");
                        fprintf(conf->logFile,"<br>\nError 34.7 en funcion especiacion() llamando a mutarAC(%u)\n",i);
                        return(0);
                    }
                }
                // coloca la especie = a la desaparecida
                conf->pob[conf->representantes[i]].especie=i;
                // copia el genoma representante de la nueva especie en todos los dem�s genomas con especie == i
                for (j=0; j<conf->sizePob; j++)
                {
                    if((conf->pob[j].especie==i)&&(j!=conf->representantes[i]))  //si no es el represeentante
                    {
                        //copia el genoma del representante a cada genoma de la poblaci�n perteneciente a la especie i

                        copiarGenoma(conf->representantes[i],j,conf);

                        //perturba pesos aleatoriamente para los genomas copiados
                        //TODO: probando mutar pesos totalmente despu�s de extinsi�n en lugar de perturbar los del genoma rep.
                        perturbarPeso(i,conf->porcentMutPeso,conf->probMutPeso,0,0,0,0, conf);
                        /*       if (randomizarPesos(j,conf->pesoMinInicial,conf->pesoMaxInicial,conf)==0){
                                   fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 35 en Funcion primeraGen() llamando a randomizarPesos(%u,%1.1f,%1.1f)\n",j,conf->pesoMinInicial,conf->pesoMaxInicial);
                                   return(0);
                               }   */

                    }
                }

                //guarda una nueva copia del genoma representante.
                copiarGenoma(conf->representantes[i],conf->sizePob+i,conf);
                // TODO: es necesiario que si hubo exterminio de una especie, realiza evaluaci�n de la poblaci�n para alistarse para el cruce.
                // EvaluarEspecie
                // TODO: quitar si se resolvi� error de fitness>1
/*                if (evaluarEspecie(1,i,conf->maxBufferSize,conf->fileNameGTDv1, conf)==0) ////TODO EL ANTERIORMANTE DICHO PARAMETRO para conf->primero para las evaluaciones y hacer inicializaciones a 0 de los valores cuando se crean las neuronas.
                {
                    fclose(conf->logFile);
                    conf->logFile=fopen(conf->fileNameLog,"a+");
                    fprintf(conf->logFile,"<br>\nError 59.55 en funcion especiacion() llamando a evaluarEspecie()");
                    return(0);
                }
                */
            }
        }

        // Actualiza representante de nueva especie
        // Compara entre el genoma guardado y el representante ANTERIOR de cada especie, deja el mejor guardado y en el index del rep.
        if (conf->pob[conf->representantes[i]].fitness < conf->pob[conf->sizePob+i].fitness)
        {
            copiarGenoma(conf->sizePob+i,conf->representantes[i],conf);
        }
        // Busca entre toda la pob los nuevos representantes.
        for (j = 0; j < conf->sizePob; j++)
        {
            if (conf->pob[j].especie == i)
            {
                if (conf->pob[j].fitness > conf->pob[conf->representantes[i]].fitness)
                {
                    conf->representantes[i]=j;
                }
            }
        }
        // Compara entre el genoma guardado y el representante ACTUAL de cada especie, deja el mejor guardado y en el index del rep.
        // Espto se hace por si el index del representante var�a entre una generaci�n y otra mejorando al anterior rep mutado(que debe haber disminuido fitness) y as� no sobreescribir el representantte nuevo.
        if (conf->pob[conf->representantes[i]].fitness > conf->pob[conf->sizePob+i].fitness)
        {
            copiarGenoma(conf->representantes[i],conf->sizePob+i,conf);
        }
    }
    // fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>,numEspecies=%u",conf->numEspecies);
    return(1);
}

unsigned primeraGen(unsigned tamPob, unsigned nEntradas, unsigned nSalidas, unsigned nBias, float minPeso, float maxPeso, unsigned maxMutacionesAN,unsigned maxMutacionesAC,unsigned maxIntentosMutAC, short unsigned useMutarAC, short unsigned useMutarAN,unsigned useRandomization, TConfig* conf)  //OPTIMIZADA
{
//genera una poblaci�n inicial de genomas de nIn entradas, nOut salidas y nBias bias a  partir de la mutaci�n
// (AN + AC) de un genoma inicial (en indice 0) totalmente conectado y que es el representante de la especie 0
//Adem�s muta aleatoriamente los pesos de llas conexiones de todos los genomas.
//retorna 0 si hubo alg�n error en el posicionamiento en memoria de las estructuras de los genomas.
//necesita: funcion (//TODO) mutarAN(), mutar(AC), genomaInicial(), nuevoNodo(), nuevaConex(), especieMinDist(),asignarEspecie()
////TODO: se deben hacer dos arreglos para los genomas de los conf->representantes de cada especie de la generaci�n actual y para
//los conf->representantes de las especies de la generaci�n anterior.
//En cada generaci�n se debe comparar cada genoma con los conf->representantes de la generaci�n anterior para determinar la especie a la que pertenecen
//y se toma el genoma con menor error como representante de la especie en el arreglo actual.
//Se debe llevar un record para cada representante de cada especie del n�mero de generaciones que lleva sin mejorar, para poder eliminaar de esta
//manera especies que se quedaron estancadas(EXCEPTO LA MEJOR(o n mejores?)).
//El n�mero de hijos que puede producir cada especie depende del fitness de sus individuos comparado con el promedio de fitness total como se
//muestra en la pag 394 del libro de AI game programming. (great tool)
//Para la primera generaci�n: Averiguar
//Algoritmo para especiar toda la conf->poblaci�n despu�s de mutaci�n y cruce, en pag 54 de disertaci�n de PhD
    unsigned i;
    unsigned j;
    unsigned numMutacionesAN;
    unsigned numMutacionesAC;
    unsigned mejor=conf->maxIntentosDist;
//TODO : FALTA: Garantizar randomizando pesos que la distancia al m�s cercano sea mayora la m�nima
//      se puede utilizar u ciclo hasta m�ximo de veces o >th y retornar el m�s lejano encontrado
//      y si es mayor del m�nimo aumenta el threshold, sin� lo disminuyen para mentener
//      el threshold de distancia �ptimo para cualquier n�mero de especies.
// para esto llenar la poblaci�n de genomas, randomizar pesos de todos escoge como rep de esp 0 a 0
// luego busca entre los que no son reps de ninguna especie el que tenga mayor m�nima distancia hasta todos los representantes
// y lo hace representante de la siguiente especie hasta spEspecies. En cada iteraci�n se DEBE hacer una randomizaci�n de los
// no-representantes (pues son m�s cercanos que el nuevo rep). DEJAR el thresold de distancia inicializado en la m�nima (porque puede haber distancias muy grandes si hay mutaciones)
// encontrada entre cada rep y los dem�s, luego hacer control sobre el th durante cada exterminio para mantenerse en el m�ximo posible
// realizando conf->iterSelRand iteraciones de randomizaci�n de pesos (y mutaciones Iniciales), manteniendo el genoma de
// MAYOR distancia a cada representante y ubicandolo como representante de su especie
    //busca el mayor entre maxIntentosDist ymaxIntentosDistInicial
    if (conf->maxIntentosDist<conf->maxIntentosDistInicial)
        mejor=conf->maxIntentosDistInicial;
    // reserva memoria para vector de genomas conf->pob lo crea de tama�o tamPob+spEspecies para poder guardar en los �ltimos elementos los representantes de cada especie(adicionado en versi�n 0.59).
    // se adiciona 1 para usar el �ltimo como genoma
    conf->realSizePob=mejor+(2*tamPob)+conf->spEspecies+1;
    if (inicializarPob(conf->realSizePob,nEntradas,nSalidas,nBias,conf)==0)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 33 en Funcion primeraGen() llamando a incializarconf->pob(%u,%u,%u,%u)\n",tamPob,nEntradas,nSalidas,nBias);
        return(0);
    }
    //marca la posici�n del genoma temporal (usado para procesamiento distribuido.)
    conf->tmpIndexPob=mejor+(2*tamPob)+conf->spEspecies;
    //Crea el genoma inicial en conf->pob[0]
    if (genomaInicial(0, nEntradas, nSalidas, nBias, 1, 0, conf)==0)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 34 en Funcion primeraGen() llamando a genomaInicial(%u,%u,%u)\n",nEntradas,nSalidas,nBias);
        return(0);
    }
    //randomiza pesos de genoma inicial
    if (useRandomization==1)
        if (randomizarPesos(0,minPeso,maxPeso,conf)==0)
        {
            fclose(conf->logFile);
            conf->logFile=fopen(conf->fileNameLog,"a+");
            fprintf(conf->logFile,"<br>\nError 35 en Funcion primeraGen() llamando a randomizarPesos(%u,%1.1f,%1.1f)\n",0,minPeso,maxPeso);
            return(0);
        }

    //imprimirGenoma(0,conf);
    fclose(conf->logFile);
    conf->logFile=fopen(conf->fileNameLog,"a+");
    fprintf(conf->logFile,"<br>\nClonando genoma inicial...\n");
    //Para los dem�s genomas copia del original, randomiza y hace las dos mutaciones (DEBE incluir los genomas guardados de reps de cada especie).
    for (i=1; i<((tamPob*2)+conf->spEspecies); i++)
    {
        //Copia el genoma inicial(0) a los dem�s desde 1 hasta tamPob-1
        if (copiarGenoma(0,i,conf)==0)
        {
            fclose(conf->logFile);
            conf->logFile=fopen(conf->fileNameLog,"a+");
            fprintf(conf->logFile,"<br>\nError 36 en Funcion primeraGen() llamando a copiarGenoma(0,%u)\n",i);
            return(0);
        }
    }
//PROBANDO MAXDISTANCIA, quitar si ok.
    //genomaMasLejano(1,conf->maxIntentosDistInicial,conf);
    //fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>dist=%7.7f, espCerc=%7.7f",calcularDist(1, conf->representantes[conf->pob[1].especie], conf->c1, conf->c2, conf->c3, conf->eG_t,conf)
    //       ,distEspecieCercana( 1,conf->c1, conf->c2, conf->c3, conf->eG_t,conf)/2);

    fclose(conf->logFile);
    conf->logFile=fopen(conf->fileNameLog,"a+");
    fprintf(conf->logFile,"<br>Aplicando mutaciones iniciales...\n");
    //hace maxDist, randomiza y hace las dos mutaciones (DEBE incluir los genomas guardados de reps de cada especie).
    for (i=1; i<((tamPob*2)+conf->spEspecies); i++)
    {

        // si se est� usando mzxDisteancia, se calcula el nuevo nodo m�s distante a todos los existentes
        // debe ir antes de todas las otras mutaciones y es excluyente con userandomization
        if (conf->maxIntentosDistInicial>0)
        {
            genomaMasLejano(i, conf->maxIntentosDistInicial,conf);
        }
        else
        {
            //randomiza pesos de conexiones de cada genoma copiado
            if (useRandomization==1)
            {
                if ((randomizarPesos(i,minPeso,maxPeso,conf))==0)
                {
                    fclose(conf->logFile);
                    conf->logFile=fopen(conf->fileNameLog,"a+");
                    fprintf(conf->logFile,"<br>\nError 37 en Funcion primeraGen() llamando a randomizarPesos(%u,%1.1f,%1.1f)\n",i,minPeso,maxPeso);
                    return(0);
                }
            }
        }

        //Aplica mutaci�n AN
        if (useMutarAN==1)
        {
            numMutacionesAN=(unsigned )floor((((float)randL(conf))*maxMutacionesAN) + 0.5);
            for (j=0; j<numMutacionesAN; j++)
                if (mutarAN(i,conf)==0)
                {
                    fclose(conf->logFile);
                    conf->logFile=fopen(conf->fileNameLog,"a+");
                    fprintf(conf->logFile,"<br>\nError 38 en funcion primeraGen() llamando a mutarAN(%u)\n",i);
                    return(0);
                }
        }

        //Aplica mutaci�n AC
        if (useMutarAC==1)
        {
            numMutacionesAC=(unsigned )floor((((float)randL(conf))*maxMutacionesAC) + 0.5);
            for (j=0; j<numMutacionesAC; j++)
                if (mutarAC(i,maxIntentosMutAC,conf)==0)
                {
                    fclose(conf->logFile);
                    conf->logFile=fopen(conf->fileNameLog,"a+");
                    fprintf(conf->logFile,"<br>\nError 39 en funcion primeraGen() llamando a mutarAC(%u,%u)\n",i,maxIntentosMutAC);
                    return(0);
                }
        }
    }
    return(1);
}

unsigned inicializarPob(unsigned tamPob,unsigned nEntradas,unsigned nSalidas,unsigned nBias, TConfig* conf)  // OPTIMIZADA, NMR,
{
//Inicializa el vector de genomas (conf->poblacci�n)
//tambi�n obtiene memoria para arreglo de nodos y conexiones de tama�o nEntradas*nSalidas*nBias
//Obtiene memoria para cada nueva estructura Genoma.
//Retorna 0 si hay error
    unsigned i=0;
    // Reserva memoria para randList[conf->tamRandList]
    conf->randList=(tRandList*) malloc(conf->tamRandList*sizeof(tRandList));
    printf("RandListInit\n");
    // Inicializa los valores aleatorios entre 0 y 1 en randList
    for(i=0;i<(conf->tamRandList-1);i++)
    {
        conf->randList[i].valor=(float)rand()/RAND_MAX;
        conf->randList[i].next=&(conf->randList[i+1]);//siguiente
    }
    // arregla el siguiente del �ltimo para que sea el primero.
    conf->randList[conf->tamRandList-1].valor=(float)rand()/RAND_MAX;
    conf->randList[conf->tamRandList-1].next=&(conf->randList[0]);//siguiente
    // apunta el puntero global de posici�n en lista de aleatorios al primero.}
    conf->punteroRand=&(conf->randList[0]);
    i=0;
    // reserva memoria para POB
    printf("PobListInit\n");
    if (conf->pob!=NULL) free((void *)conf->pob);
    if ((conf->pob = ( Genoma *) calloc(tamPob,(unsigned  int)sizeof(Genoma)))==NULL)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 28 en funcion inicializarPob(%u,%u,%u,%u) llamando a calloc(%u,%u)\n",tamPob,nEntradas,nSalidas,nBias,tamPob,(unsigned  int)sizeof(Genoma));
        return(0);
    }
    for (i=0; i<tamPob; i++)
    {
        if (conf->pob[i].nodo!=NULL) free((void *)conf->pob[i].nodo);
        if ((conf->pob[i].nodo=(GenNodoF*)calloc(1,sizeof(GenNodoF)*(nEntradas+nBias+nSalidas)))==NULL)
        {
            fclose(conf->logFile);
            conf->logFile=fopen(conf->fileNameLog,"a+");
            fprintf(conf->logFile,"<br>\nError 29 en funcion inicializarPob(%u,%u,%u,%u) llamando a calloc(1,%u)\n",tamPob,nEntradas,nSalidas,nBias,(nEntradas+nBias+nSalidas)*(unsigned int)sizeof(GenNodoF));
            return(0);
        }
        if (conf->pob[i].conex!=NULL) free((void *)conf->pob[i].conex);
        if ((conf->pob[i].conex=(GenConexF*)calloc(1,sizeof(GenConexF)*((nEntradas+nBias)*nSalidas)))==NULL)
        {
            fclose(conf->logFile);
            conf->logFile=fopen(conf->fileNameLog,"a+");
            fprintf(conf->logFile,"<br>\nError 30 en funcion inicializarPob(%u,%u,%u,%u) llamando a calloc(1,%u)\n",tamPob,nEntradas,nSalidas,nBias,(nEntradas+nBias+nSalidas)*(unsigned int)sizeof(GenConexF));
            return(0);
        }

    }
    return(1);
}

int competencia(TConfig* conf)
// Realiza competencia, que copmara con una copia guardada en indexPob+numEspecies+sizePob
// usa variable conf->porcentCompetencia como prob de ganancia del backup si es mejor que el actual,
// si es peor, simplemente se reemplaza.
// si es negativa, no hace competencia.
// solo compara si los correspondientes indexpob son de la misma especie( para evitar acaparamiento)
// se debe ejecutar despu�s de especiacion().
// retorna 0 si hay error, 1 si ok
{
    int i;
    unsigned bkp;
    if (conf->porcentCompetencia<0)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError: porcentCompetencia negativo");
        return(0);
    }
    //para todos los genomas de la poblaci�n:
    for(i=0; i<conf->sizePob; i++)
    {
        bkp=conf->sizePob+i+conf->spEspecies; // posici�n del backup
        // si i y backup son de la misma especie
        if (conf->pob[i].especie==conf->pob[bkp].especie)
        {
            // si backup.fitness>i.fitness
            if (conf->pob[bkp].fitness>conf->pob[i].fitness)
            {
                // prob de porcentCompetencia
                if (((float)randL(conf))<conf->porcentCompetencia)
                {
                    // copia backup a i
                    fclose(conf->logFile);
                    conf->logFile=fopen(conf->fileNameLog,"a+");
                    fprintf(conf->logFile,"<br>S=%u,",i);
                    copiarGenoma(bkp,i,conf);
                }
                else //TODO: PROBAR QUITANDO EL ELSE
                {
                    // copia i malo a bkp (como porque puede tenerse que pasar por etapa de malo durante mutaci�n y luego mejorar)
                    copiarGenoma(i,bkp,conf);
                }

            }
            else
            {
                copiarGenoma(i,bkp,conf);
            }

        }
        else //si son de diferentes especies en el mismo index, copia el nuevo a backup
        {
            copiarGenoma(i,bkp,conf);
        }
    }
    return(1);
}

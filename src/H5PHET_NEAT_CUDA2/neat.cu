#include "hip/hip_runtime.h"
/** H5PHET - NEAT
	Implementaci�n en ANSI C 89 de la t�cnica: "Neuro Evolution of Augmenting Topologies" de Kenneth Stanley (2004).
	Usa archivo de entrada GTD (Generic Training Data) y genera archivo de salida SNN (Simple Neural Network).
	Parte de H5PHET.
	Por Harvey Bastidas.
*/
# include <sys/types.h>
//# include <sys/local.h>
#include "params.h"
#include "auxiliares.h"
#include "genoma.h"
#include "pob.h"


//TODO: pasar c�digo por gnuindent para estandarizar la indentaci�n.
////TODO: Funcion freeAll() libera memoria de todos los arreglos usados. :) TODO: problemas:
//		hay conexiones con el mismo innovnum repetidas en un genoma.
//		hay disminuci�n de fitness entre evaluaciones
//		hay error de punteros cuando numbias=0
//		en la iteraci�n 130 de un experimento se retorn� fitness de 8.4
//		hay que adicionar como par�metro uso de mutar_tSigma y su respectiva probabilidad por genoma para mutar el tSigma de un nod, tambi�n correcciones en calcularValorNodo();
//		perturbaci�n de pesos con variaci�n m�xima para el �ltimo m�nima para nodos de entrada y bias y lineal o cuadr�tica para el unsigned  int�rvalo
//		implementar mutaci�n removerNodo y remover conexi�n (verificar si es posible y en que casos y condiciones deber�a usarse)
//      erroro de punteros despu�s de e my raro(gen 100+)
//      error poco frecuente en we012 (Mirar donde y a que index de cone se hace malloc y ver porque falla al hacer free)
//          tambi�n verificar otras variables que se liberan
//      TODO: c�lculo de dist�ncia m�n para pertenecer a especie a partir de spEspecies , numDisjounsigned (o mejor porcent respecto m�s grande?), numExcess y difPEsos (al menos con esto se puede hacer) max y min como r�ngo de b�squeda
//      funci�n verificarMejor retorna que se decrement� el fitness despu�s de una extinci�n.
//		TODO: es probable que evaluargenoma no funcione bi�n, probarlo con genomaPerfecto.
//		TODO: funci�n para evaluar un genoma en particular.diferente a evaluarGenoma porque usa evaluarpob para el streaming de entradas
//      TODO: despu�s de corregir bug de we012 y errores de crossover hacer variables: maxThreadProc, usarCUDA y numThCuda (mutex?)
//              adem�s colocar

/********* MAIN *********/
int main(int argc, char *argv[])
{

    TConfig conf; //par�metros del sistema

//Versi�n:

    float version=0.71; // H5PHET 0.71 Parte de EVA parte de TGV
                        // Entrada = Archivo GTD y NNP de inicio o para procesamiento distribuido
                        // Salida = Archivo SNN y NNP.

    unsigned j=1;
    unsigned k=1;
    unsigned i;
    unsigned theOne=0;
    //(genera un n�mero entre 0 y 1e3 )
    i=(unsigned  int) time(NULL);
    j=j*(unsigned  int)(1+floor((i-10.0*floor((float)i/10.0))+0.5));
    j=j*(unsigned  int)(floor((i-1000.0*floor(i/1000.0))+0.5));
    j=j+(unsigned  int)floor((i-10.0*floor((float)i/10.0))+0.5)+1;
    j=j+(unsigned  int)floor((i-1000.0*floor(i/1000.0))+0.5);
    //Inicializa el pseudo-random seed con la hora actual.
    srand(j);
  /*  for (i=0; i<j; i++)
    {
        k=(unsigned )rand();
    }
	*/
	k=(unsigned )rand();
    // re-Inicializa el pseudo-random seed con el �ltimo rand obtenido.
    srand(k);
    // inicializa todas las variables de configuraci�n a valores por defecto.
    inicializaciones(&conf);
	//verifica y asigna los par�metros de linea de comandos, si hay alg�n error, sale.
	if (!procParameters(argc,argv,version, &conf))
	{
	    printf("\nError M6 en main() llamando a procParameters\n");
        return(0);
	}
    // abre archivo de logs.
    // unsigned primeraGen(float tamPob, unsigned nEntradas, unsigned nSalidas, unsigned nBias, float minPeso, float maxPeso, unsigned maxMutacionesAN,unsigned maxMutacionesAC,unsigned maxIntentosMutAC, short unsigned useMutarAC, short unsigned useMutarAN,unsigned useRandomization){
    fclose(conf.logFile);conf.logFile=fopen(conf.fileNameLog,"a+");fprintf(conf.logFile,"<br>\nH5PHET - NEAT V%2.2f\nCreando primera generaci�n\n",version);
    if ((primeraGen(conf.sizePob,conf.numEntradas,conf.numSalidas,conf.numBias,conf.pesoMinInicial,conf.pesoMaxInicial,conf.mutacionesPorExterminio,conf.mutacionesPorExterminio,conf.maxIntentosMutarAC,0,0,1,&conf))==0) // TODO: Cuadrar en params.h los par�metros useMutarAN y useMutarAC de esta Funcion.
    {
        fclose(conf.logFile);conf.logFile=fopen(conf.fileNameLog,"a+");fprintf(conf.logFile,"<br>\nError M1 en main() en funci�n primeraGen(%u,%u,%u,%u,%1.1f,%1.1f,%u,%u,%u,%u,%u,%u)",conf.sizePob,conf.numEntradas,conf.numSalidas,conf.numBias,-3.0,3.0,3,1,100,0,0,1);
        fclose(conf.logFile);conf.logFile=fopen(conf.fileNameLog,"a+");fprintf(conf.logFile,"<br>\nGuardando mejor genoma en c:\\mejorGenoma.txt");
        i=guardarGenomaSNN(buscarMejorFitness(&conf),conf.fileNameSNNv1,&conf); //TODO: esto es necesario en este punto?
        fclose(conf.logFile);conf.logFile=fopen(conf.fileNameLog,"a+");fprintf(conf.logFile,"<br>\nSaliendo...");
        return(0);
    }
//ciclo principal
    fclose(conf.logFile);conf.logFile=fopen(conf.fileNameLog,"a+");fprintf(conf.logFile,"<br>Iniciando ciclo principal\n");
    if ((cicloPrincipal(&conf))==UINT_MAX)
    {
        fclose(conf.logFile);conf.logFile=fopen(conf.fileNameLog,"a+");fprintf(conf.logFile,"<br>\nError M2 en main() en funcion cicloPrincipal()");
        fclose(conf.logFile);conf.logFile=fopen(conf.fileNameLog,"a+");fprintf(conf.logFile,"<br>\nGuardando mejor genoma en c:\\mejorGenoma.txt");
        i = guardarGenomaSNN(buscarMejorFitness(&conf),conf.fileNameSNNv1,&conf);
        imprimirGenoma(i,&conf);
        fclose(conf.logFile);conf.logFile=fopen(conf.fileNameLog,"a+");fprintf(conf.logFile,"<br>\nSaliendo...");
        fclose(conf.fOut);
        fclose(conf.fIn);
        return(0);
    }

    theOne=buscarMejorFitness(&conf);

    /*	fclose(conf.logFile);conf.logFile=fopen(conf.fileNameLog,"a+");fprintf(conf.logFile,"<br>unsigned  introduciendo GenomaPerfectocomo elemento 15");
    	if (genomaPerfecto(15,&conf)==0){ //necesaria esta evaluaci�n antes de primera especiaci�n?
    		fclose(conf.logFile);conf.logFile=fopen(conf.fileNameLog,"a+");fprintf(conf.logFile,"<br>\nError 59 en funcion cicloPrincipal() llamando a evaluarPob()");
    		return(UINT_MAX);
    	}
    theOne=15;*/
    fclose(conf.logFile);conf.logFile=fopen(conf.fileNameLog,"a+");fprintf(conf.logFile,"<br>\nLista de Representantes: ");
    for (i=0; i<conf.numEspecies; i++)
    {
        fclose(conf.logFile);conf.logFile=fopen(conf.fileNameLog,"a+");fprintf(conf.logFile,"<br>\nEspecie %u(%u) = %7.7f",i,conf.representantes[i],conf.pob[conf.sizePob+i].fitness);
    }
    fclose(conf.logFile);conf.logFile=fopen(conf.fileNameLog,"a+");fprintf(conf.logFile,"<br>\nGuardando mejor genoma %u con fitness = %11.11f en c:\\mejorGenoma.txt",theOne,conf.pob[theOne].fitness);
    i=guardarGenomaSNN(theOne,conf.fileNameSNNv1,&conf);

/*
    //Prueba la Funcion xor con el mejor genoma
    entr=(float*)malloc(sizeof(float)*8);
    if (entr==NULL)
        return(0);
    salid=(float*)malloc(sizeof(float)*4);
    if (salid==NULL);
    {
        free (entr);
        free (salid);//??para evitar warning de clocwork
        return(0);
    }
    entr[0]=-1;
    entr[1]=-1;
    entr[2]=-1;
    entr[3]=1;
    entr[4]=1;
    entr[5]=-1;
    entr[6]=1;
    entr[7]=1;
    salid[0]=1;
    salid[1]=1;
    salid[2]=1;
    salid[3]=1;
    actualizarPNodos(theOne,&conf);
    i=evaluarGenoma(theOne,0,entr,salid,&conf);
    salid[0]=conf.pob[theOne].nodo[3].valor;
    i=evaluarGenoma(theOne,0,entr+2,salid+1,&conf);
    salid[1]=conf.pob[theOne].nodo[3].valor;
    i=evaluarGenoma(theOne,0,entr+4,salid+2,&conf);
    salid[2]=conf.pob[theOne].nodo[3].valor;
    i=evaluarGenoma(theOne,0,entr+6,salid+3,&conf);
    salid[3]=conf.pob[theOne].nodo[3].valor;

    fclose(conf.logFile);conf.logFile=fopen(conf.fileNameLog,"a+");fprintf(conf.logFile,"<br>\nEvaluaci�n de la tabla de XOR para el mejor genoma: \n");
    fclose(conf.logFile);conf.logFile=fopen(conf.fileNameLog,"a+");fprintf(conf.logFile,"<br> %3.3f XOR %3.3f = %3.3f \n",entr[0],entr[1],salid[0]);
    fclose(conf.logFile);conf.logFile=fopen(conf.fileNameLog,"a+");fprintf(conf.logFile,"<br> %3.3f XOR %3.3f = %3.3f \n",entr[2],entr[3],salid[1]);
    fclose(conf.logFile);conf.logFile=fopen(conf.fileNameLog,"a+");fprintf(conf.logFile,"<br> %3.3f XOR %3.3f = %3.3f \n",entr[4],entr[5],salid[2]);
    fclose(conf.logFile);conf.logFile=fopen(conf.fileNameLog,"a+");fprintf(conf.logFile,"<br> %3.3f XOR %3.3f = %3.3f \n",entr[6],entr[7],salid[3]);
*/

    fclose(conf.logFile);conf.logFile=fopen(conf.fileNameLog,"a+");fprintf(conf.logFile,"<br>\nMemoria Usada=%lu\n",calcularMemoriaUsada(conf.sizePob,&conf)/1024);
    fclose(conf.logFile);conf.logFile=fopen(conf.fileNameLog,"a+");fprintf(conf.logFile,"<br>\nN�mero de especies iniciales: %u\n",conf.numEspecies);
    imprimirGenoma(theOne,&conf);
    fclose(conf.logFile);conf.logFile=fopen(conf.fileNameLog,"a+");fprintf(conf.logFile,"<br>\nH5PHET - NEAT finanizado correctamente.\n");
    ////TODO funcion freeAll(), libera la memoria requerida por cada arreglo de nodos y conexiones de cada genoma para toda la conf.poblaci�n y libera conf.fInalmente *conf.pob
    return(0);
}

//Mashauritaki
/*
VER EVOLVING NEUR... Para parametros de XOR y //TODOntes de perturbaci�n de pesos.
Inicializaciones de par�metros en pag 14 de Evolving Neural networks through augmenting topologies.pdf
La especici�n se hace basado en la dt el threshold de distancia m�nimo para una nueva especie.




*/

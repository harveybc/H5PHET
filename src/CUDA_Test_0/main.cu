#include <stdio.h>
#include <hip/hip_runtime.h>
//#include <stdio.h>
//#include <stdlib.h>
#define BLOCK_SIZE 16
// Es 1000 porque son 3 vectores de float =12kbytes, supuestamente tienen 16kbytes/multiproc (bloque)
#define THREADSPB 96
// kernel de CUDA para sumar dos vectores. (buffsize=tam copiado de global a shared)
__global__ void sumaVec (float* vec1_d, float* vec2_d, float* vec3_d, int numDatos)
{
    int i,j;
    float v1,v2,v3,acum; //registros para cada entrada y salida, acumulador par calculo de media
    // calcula el id del thread
    int idx=blockDim.x*blockIdx.x+threadIdx.x;
    // calcula el n�mero de repeticiones para floor de todos los datos/buffSize
    int numReps=numDatos/THREADSPB;

	//THREADSPB tambi�n es el n�mero de datos completos leidos de global a shared.
    // declara el vector shared para vec1,2 y 3 de tama�o buffsize/(3*sizeof(float))
    __shared__ float vec1_s[THREADSPB];
    __shared__ float vec2_s[THREADSPB];
    __shared__ float vec3_s[THREADSPB];
    // para i=0;i<numReps;i++ , para cada repetici�n
    acum=0;
    for (i=0;i<numReps;i++)
    {
		// coloca en memoria shared los valores de global, los lee en paralelo
		vec1_s[idx]=vec1_d[i*THREADSPB+idx];
		vec2_s[idx]=vec2_d[i*THREADSPB+idx];
		//sincroniza
		__syncthreads();
		// copia de shared a registros
        // para j=0;j<THREADSPB;j++
		for(j=0;j<THREADSPB;j++)
		{
            //sincroniza los threads para leer al mismo tiempo de la memoria shared.
            __syncthreads();
			// coloca el dato j en los registros necesarios
            v1=vec1_s[j];
            v2=vec2_s[j];
            // realiza los calculos con los registros.
            v3=v1*v2;
            // acumula valor de salida de entrenam y calculada para posterior calculo de error
            acum+=v3;
            // escribe en shared[j] los resultados
            vec3_s[j]=v3;
        }
        __syncthreads();
		// copia v3 parcial calculado de shared a global
		vec3_d[i*THREADSPB+idx]=vec3_s[idx];
    }
    // calcula el fitness
    // para i=0;i<numReps;i++
        // si el id==0 (para que solo lo haga un core por block)   NECESARIO?
            // copia de global vec1[i*buffsize], buffsize datos a shared.
        // para j=0;j<buffsize;j++
        // calcula las 3 sumatorias del coeficiente de correlaci�n
}


int main()
// prueba de CUDA: suma dos vectores componente a componente
{
    int i,j;
    float* vec1_h;
    float* vec2_h;
    float* vec3_h;
    float* vec1_d;
    float* vec2_d;
    float* vec3_d;
    int numDatos=1000000;
	int tamVectores=numDatos*sizeof(float);
    // reserva memoria para vector 1,2 y 3 en host
    vec1_h=(float*)malloc(tamVectores);
    vec2_h=(float*)malloc(tamVectores);
    vec3_h=(float*)malloc(tamVectores);
    // llena con n�meros aleatorios los vectores 1 y 2 en host
    printf("Generando vectores\n");
    for (i=0;i<numDatos;i++)
    {
        vec1_h[i]=(float)rand()/RAND_MAX;
        vec2_h[i]=(float)rand()/RAND_MAX;
    }
    printf("Reservando memoria en device\n");
    // reserva memoria para vector 1,2 y 3 en device
    hipMalloc(&vec1_d, tamVectores); //TODO: POSIBLE PROBLEMA: SE MACE MALLOC DE &vec1_d en lugar de vec1_d (es puntero)
    hipMalloc(&vec2_d, tamVectores); //TODO: POSIBLE PROBLEMA: SE MACE MALLOC DE &vec1_d en lugar de vec1_d (es puntero)
    hipMalloc(&vec3_d, tamVectores); //TODO: POSIBLE PROBLEMA: SE MACE MALLOC DE &vec1_d en lugar de vec1_d (es puntero)
    printf("Copiando vectores de host a device\n");
    // copia en la memoria global de device los vec 1 y 2 de host
    hipMemcpy(vec1_d, vec1_h, tamVectores, hipMemcpyHostToDevice);
    hipMemcpy(vec2_d, vec2_h, tamVectores, hipMemcpyHostToDevice);
    // ejecuta kernel de CUDA con los 3 vectores device, su tama�o y ShBuffSize como par�metros
    printf("Ejecutando kernel de CUDA para todos los threads\n");
    int threadsPorBlock=THREADSPB;
    int blocksPorGrid=(1+threadsPorBlock-1)/threadsPorBlock;
    sumaVec<<<blocksPorGrid,threadsPorBlock>>>(vec1_d,vec2_d,vec3_d,numDatos);
    // copia el vector 3 de la memoria global de device a host.
    hipMemcpy(vec3_h, vec3_d, tamVectores, hipMemcpyDeviceToHost);
    // imprime los resultados
    j=0;
    if (numDatos>1000)
        j=numDatos-1000;
    for (i=j;i<numDatos;i++)
    {
        printf("[%i]  %3.3f + %3.3f = %3.3f\n",i,vec1_h[i],vec2_h[i],vec3_h[i]);
    }
	while (1)
	{
	    if ('n' == getchar())
		break;
	}
    return 0;
}


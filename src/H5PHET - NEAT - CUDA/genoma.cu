#include "hip/hip_runtime.h"
/** Redes neuronales para NEAT - H file
	usan/modifican genomas �nicamente (pueden usarse operaciones de genes de gen.h en un genoma).
*/

#ifndef PARAMS_H_INCLUDED
#include "params.h"
#define PARAMS_H_INCLUDED
#endif
#ifndef AUXILIARES_H_INCLUDED
#include "auxiliares.h"
#define AUXILIARES_H_INCLUDED
#endif
#ifndef GEN_H_INCLUDED
#include "gen.h"
#define GEN_H_INCLUDED
#endif
#ifndef ESPECIE_H_INCLUDED
#include "especie.h"
#define ESPECIE_H_INCLUDED
#endif
#include "genoma.h"

float evalGenom(int indexpob,TConfig* conf)
{


    int i=indexpob;
    int j,k,totalNodos,indSalidas,indBias,indInS;
    // float matrixF[200][200]; // mil nodos m�ximo
    float valor[500]; // valor de salida de la neurona
    // float valorAnt[2000]; // valor de salida de la neurona
    float acum;
    // calcula el total de Nodos.
    totalNodos=conf->headerSNN[i].numEntradas+conf->headerSNN[i].numSalidas+conf->headerSNN[i].numBias+conf->headerSNN[i].numHiddens;
    // inicializa el fitness del genoma  en 0
    conf->fitness[i]=0;
    // calcula l�mites de for para optimizaci�n.
    indSalidas=(conf->headerGTD.numEntradas+conf->headerSNN[i].numBias+conf->headerSNN[i].numSalidas);
    indBias=(conf->headerGTD.numEntradas+conf->headerSNN[i].numBias);
    indInS=conf->headerGTD.numEntradas*sizeof(float);
    // reordena la lista de conexiones para que quede simialr a evaluaci�n recursiva.
    // para los que est�n al mismo nivel, debe ir primero el que est� m�s cerca a las entradas.
    // eval�a los datos de GTD en las conex y produce el vector de fitness
    // coloca en 0 valor[] y valorAnt[] superiores a las entradas y bias
    for (k=indBias;k<totalNodos;k++)
    {
        valor[k]=0;
    }
    //coloca bias en 1 ://TODO: falta para varias bias, es necesario? NO
    valor[conf->headerGTD.numEntradas]=1;
    for (j=0;j<conf->numDatos;j++)
    {
       // coloca los valores de entrada de valor[] en valorAnt[] y copia los nuevos a valor[]
// INICIO KERNEL CUDA_1, CAMBIAR  memcpy
// HACER SYNC ANTES DE HACER EL MEMCPY DESDE LOCAL A SHARED de dataGTDf[][] y los acum de Vc y Vt para posterior c�lculo de error
       // memcpy(valorAnt,valor,indInS);
        memcpy(&(valor[0]),conf->dataGTDf[j],indInS);
        // coloca bias con valor 1 .
        valor[conf->headerGTD.numEntradas]=1;
        // calcula el fSigma de las entradas
        for (k=0;k<conf->headerGTD.numEntradas;k++)
        {
            valor[k]=2*((float)exp(-(float)conf->A*((valor[k]-(float)conf->Fthreshold)*(valor[k]-(float)conf->Fthreshold))))-1;;
        }
        // calcula el nuevo valor[] con el genoma i y dataGTD
        acum=0;
        for (k=0;k<conf->tamListaConexPost[i];k++)
        //for (k=0;k<headerSNN[i].numConex;k++)
        {
            acum+=(conf->listaConexData[i][k].peso*valor[conf->listaConexData[i][k].conexIn]);
            // si el conexOut no es el �ltimo
            if (k<(conf->tamListaConexPost[i]-1))
            {
                // si el ConexOut siguiente es diferente al actual
                if (conf->listaConexData[i][k].conexOut!=conf->listaConexData[i][k+1].conexOut)
                {
                    // calcula el sigma y actualiza valorAnt
                    valor[conf->listaConexData[i][k].conexOut]=2*((float)exp(-(float)conf->A*((acum-(float)conf->Fthreshold)*(acum-(float)conf->Fthreshold))))-1;
                    //reinicializa el acumulador
                    acum=0;
                }
            }
        }
        // saca el Fsigma del �ltimo nodo.

        valor[conf->listaConexData[i][conf->tamListaConexPost[i]-1].conexOut]=2*((float)exp(-(float)conf->A*(acum-(float)conf->Fthreshold)*(acum-(float)conf->Fthreshold)))-1;
        // copia valor a valorAnt
//        memcpy(valorAnt,valor,totalNodos*sizeof(float));
        // guarda los valores calculados en el arreglo valoresC[]
        for (k=indBias;k<indSalidas;k++)
        {
           conf->valoresC[j]=valor[k];
        }
        //FALTA: tambi�n calcular acumuladores de Vt y Vc para sacar medias al final desde host y luego llamar otro kernel que
        // calcula los errores. Lo mejor ser�a un kernel que calcule los vectores en un ciclo hasta numDatos/buffarsize
        // incluyendo transferencia desde buffer grande a shared y viceversa, luego
//FIN KERNEL CUDA_1 c, cambiar memcpy
    }
    // calcula el fitness como el coeficiente de correlaci�n de Pearson de los 2 vectores (1 si =es,-1 si inversos, 0 si diferentes)
//INICIO KERNEL CUDA_2 (o parte final de CUDA_1)
    conf->fitness[i]=correlac(conf->valoresC,conf);
//FIN KERNEL CUDA_2 para calculo de fitness.

    return(conf->fitness[i]);
}

unsigned guardarGenomaSNN(unsigned indexpobA, char *filename, TConfig* conf)  // OPTIMIZADA
{
//Escribe un genoma deseado de conf->pob en un archivo
//El formato de salida es (sin separadores): Genoma, Genoma.nodo, genoma.conex las longitudes a escribir
//de cada estructura se basan en el tama�o de Genoma, GenNodoF, GenconexF y en los valores Genoma.totalNodos
//y Genoma.totalConexiones
//Par�metros:	indexpob = indice del arreglo de genomas conf->pob que se va a guardar
//				filename = path y nombre de archivo en el que se guardar� el genoma
//Retorna 0 si hay error, 1 si ok
    int result;
    FILE *fileOut;
    size_t escritos=0;
    // Formato SNN: [header] unsigned conexIn[numConex], unsigned conexOut[numConex], unsigned ,double peso[numConex]
    int i,j,indexpob=indexpobA;
    double tmpPesoD;
    float tmpPesoF;
    char tmpChar;
//    float tmpFit=0;
    //si es un rep y si fitness <que backup, llama a las copiasdel backup
    if (indexpob>conf->sizePob)
    {
        if (conf->pob[indexpob].fitness>conf->pob[conf->representantes[indexpob-conf->sizePob]].fitness)
            copiarGenoma(indexpob,conf->representantes[indexpob-conf->sizePob],conf);
        indexpob=conf->representantes[indexpob-conf->sizePob];
    }
    // inicializa los campos del header de SNN
    conf->headerSNN[indexpob].fileID[0] = 'S';
    conf->headerSNN[indexpob].fileID[1] = 'N';
    conf->headerSNN[indexpob].fileID[2] = 'N';
    conf->headerSNN[indexpob].version = 1;
    conf->headerSNN[indexpob].usarSigned = conf->tSigma>=1000? 1 : 0;
    conf->headerSNN[indexpob].tamRegistros = conf->useFloat==0? 8 : 4;
    conf->headerSNN[indexpob].numEntradas = conf->numEntradas;
    conf->headerSNN[indexpob].numSalidas = conf->numSalidas;
    conf->headerSNN[indexpob].numBias = conf->numBias;
    conf->headerSNN[indexpob].numHiddens = conf->pob[indexpob].totalNodos-(conf->numEntradas+conf->numSalidas+conf->numBias);
    conf->headerSNN[indexpob].numConex = conf->pob[indexpob].totalConexiones;
    conf->headerSNN[indexpob].sigmaFactor = (double)conf->A;
    conf->headerSNN[indexpob].actThreshold = (double) conf->Fthreshold;
    conf->headerSNN[indexpob].lastFitness = (double) conf->pob[indexpob].fitness; // usado para programac distribuida
    // actualiza la lista de conexiones en orden de evaluaci�n
//    tmpFit=evalGenom(indexpob,conf);
    printf("FitnessMejorGuardado=%7.7f\n",conf->pob[indexpob].fitness);
    // abre el archivo de salida para escritura
    if ((fileOut=fopen(filename,"wb"))==NULL)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 49 en funcion guardarMejorFitness(%u,%s) llamando a fopen(%s,\"wb\")\n",indexpob,filename,filename);
        return(0);
    }
    // Escribe el encabezado SNN
    escritos=fwrite(&(conf->headerSNN[indexpob]), sizeof(hdrSNNv1),1,fileOut);
    // Escribe los arreglos en orden: int conexIn[numConex],conexOut[numConex],double peso[numConex]
    // para conexIn
    for (i=0; i<conf->pob[indexpob].totalConexiones; i++)
    {
        // escribe los 3 arrays con los datos de la conex
        escritos=fwrite(&(conf->pob[indexpob].conex[i].indexIn), sizeof(unsigned),1,fileOut);
    }
    // para conexOut
    for (i=0; i<conf->pob[indexpob].totalConexiones; i++)
    {
        // escribe los 3 arrays con los datos de la conex
        escritos=fwrite(&(conf->pob[indexpob].conex[i].indexOut), sizeof(unsigned),1,fileOut);
    }
    // para enableds
    for (i=0; i<conf->pob[indexpob].totalConexiones; i++)
    {
        tmpChar = conf->pob[indexpob].conex[i].enabled;
        escritos = fwrite(&tmpChar, sizeof(char),1,fileOut);
    }
    // tama�o de lista de orden de evaluaci�n de conexiones
    if (conf->tamListaConexPost[indexpob]==0)
    {
        printf("ERROIR");
        exit(0);
    }
    escritos = fwrite(&(conf->tamListaConexPost[indexpob]), sizeof(int),1,fileOut);
    // genera y escribe lista de evaluaci�n de conexiones
    for (i=0;i<conf->tamListaConexPost[indexpob];i++)
    {
        result=-1;
        for(j=0;j<conf->pob[indexpob].totalConexiones;j++)
        {
            if ((conf->listaConexData[indexpob][i].conexIn==conf->pob[indexpob].conex[j].indexIn)&&(conf->listaConexData[indexpob][i].conexOut==conf->pob[indexpob].conex[j].indexOut))
                result=j;
        }
        if (result==-1)
        {
            printf("\nError 56 en guardarGenomaSNN(), conex no encontrada");
            exit(0);
        }
        escritos = fwrite(&result, sizeof(int),1,fileOut);
    }
    //escritos = fwrite(conf->listaConexData[indexpob], conf->tamListaConexPost[indexpob]*sizeof(int),1,fileOut);
    // para peso
    for (i=0; i<conf->pob[indexpob].totalConexiones; i++)
    {
        if (conf->headerSNN[indexpob].tamRegistros==4)//para float
        {
            //hace cast para sacar double a partir de float.
            tmpPesoF=(float)conf->pob[indexpob].conex[i].peso;
            // escribe los 3 arrays con los datos de la conex
            escritos=fwrite(&tmpPesoF, sizeof(float),1,fileOut);
        }
        else //para double
        {
            //hace cast para sacar double a partir de float.
            tmpPesoD=(double)conf->pob[indexpob].conex[i].peso;
            // escribe los 3 arrays con los datos de la conex
            escritos=fwrite(&tmpPesoD, sizeof(double),1,fileOut);
        }
    }
    //cierra el archivo SNN.
    if (fclose(fileOut)!=0)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 50 en funcion guardarMejorFitness(%u,%s) llamando a fclose(fileout))\n",indexpob,filename);
        return(0);
    }
    // libera memoria de los arreglos.
    return(1);
}
////TODO: el fitness se modifica en cada Funcion que modifique el genoma pero se debe usar conf.actualizarEnCambios para controlar esto.
//pero la Funcion debe llamar a evaluarpob y luego actualiza representantes.


int snnDataLoader(const unsigned indexpob, const unsigned especie, FILE* fileIn, TConfig* conf)
// Carga un snn en indexpob desde un archivo fileIn que debe estar abierto reconstruyendo el genoma.
{
    hdrSNNv1 headerSNN;
    unsigned* conexIn;
    unsigned* conexOut;
    char* enabled;
    float* pesosF=NULL;
    double* pesosD=NULL;
    int i, j , tmp, leidos,tmpNumNodos;

    // lee encabezado SNNv1
    leidos=fread(&headerSNN,sizeof(hdrSNNv1),1,fileIn);
    if (leidos<1)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 344.5 en snnDataLoader() llamando a fread()");
        return(1);
    }
    //verifica fileId y version de headerSNN
    if((headerSNN.fileID[0]!='S')||(headerSNN.fileID[1]!='N')||(headerSNN.fileID[2]!='N')||(headerSNN.version!=1)||(headerSNN.numEntradas>32000)||(headerSNN.numBias>32000)||(headerSNN.numConex>4000000)||(headerSNN.numHiddens>32000)||(headerSNN.numSalidas>32000)||(headerSNN.sigmaFactor>10000)||(headerSNN.actThreshold>1000)||(headerSNN.tamRegistros>256)||(headerSNN.usarSigned>100))
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 345 en snnDataLoader() error en encabezado SNNv1 %c%c%c v=%d,ne=%d,%d,%d,%d,%d,%d",headerSNN.fileID[0],headerSNN.fileID[1],headerSNN.fileID[2],headerSNN.version,headerSNN.numEntradas,headerSNN.numBias,headerSNN.numHiddens,headerSNN.numSalidas,headerSNN.numConex,headerSNN.tamRegistros);
        return(0);
    }
    // reserva memoria para los arreglos  a leer o leer secuencialmente? ver formato SNN
    tmp=headerSNN.numConex;
    if (tmp<4000000)
    {
        conexIn = (unsigned *) malloc(tmp*sizeof(unsigned));
        if (conexIn==NULL)
        {
            fclose(conf->logFile);
            conf->logFile=fopen(conf->fileNameLog,"a+");
            fprintf(conf->logFile,"<br>\nError 346 en snnDataLoader() llamando a malloc()");
            return(0);
        }
        conexOut = (unsigned *) malloc(tmp*sizeof(unsigned));
        if (conexOut==NULL)
        {
            fclose(conf->logFile);
            conf->logFile=fopen(conf->fileNameLog,"a+");
            fprintf(conf->logFile,"<br>\nError 347 en snnDataLoader() llamando a malloc()");
            return(0);
        }
        enabled = (char *) malloc(tmp*sizeof(sizeof(char)));
        if (enabled==NULL)
        {
            fclose(conf->logFile);
            conf->logFile=fopen(conf->fileNameLog,"a+");
            fprintf(conf->logFile,"<br>\nError 347.1 en snnDataLoader() llamando a malloc()");
            return(0);
        }
        if (headerSNN.tamRegistros==4)
        {
            pesosF = (float*) malloc(tmp*sizeof(float));
            if (pesosF==NULL)
            {
                fclose(conf->logFile);
                conf->logFile=fopen(conf->fileNameLog,"a+");
                fprintf(conf->logFile,"<br>\nError 348 en snnDataLoader() llamando a malloc()");
                return(0);
            }
        }
        else if (headerSNN.tamRegistros==8)
        {
            pesosD = (double*) malloc(tmp*sizeof(double));
            if (pesosD==NULL)
            {
                fclose(conf->logFile);
                conf->logFile=fopen(conf->fileNameLog,"a+");
                fprintf(conf->logFile,"<br>\nError 349 en snnDataLoader() llamando a malloc()");
                return(0);
            }
        }
        else
        {
            fclose(conf->logFile);
            conf->logFile=fopen(conf->fileNameLog,"a+");
            fprintf(conf->logFile,"<br>\nError 350 en snnDataLoader() temRegistros invalido.");
            return(0);
        }
    }
    //sino imprime error y retorna
    else
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 350.2 en snnDataLoader() numConex>4M.");
        return(1);
    }
    // lee los 4 arreglos.
    leidos=fread(conexIn,sizeof(unsigned),headerSNN.numConex,fileIn);
    leidos+=fread(conexOut,sizeof(unsigned),headerSNN.numConex,fileIn);
    leidos+=fread(enabled,sizeof(char),headerSNN.numConex,fileIn);
    leidos+=fread(&(conf->tamListaConexPost[indexpob]),sizeof(int),1,fileIn);
    leidos+=fread(conf->listaConexData[indexpob],sizeof(int),conf->tamListaConexPost[indexpob],fileIn);
    if (headerSNN.tamRegistros==4)
    {
        leidos+=fread(pesosF,sizeof(float),headerSNN.numConex,fileIn);
    }
    else if (headerSNN.tamRegistros==8)
    {
        leidos+=fread(pesosD,sizeof(double),headerSNN.numConex,fileIn);
    }
    else
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 350.3 en snnDataLoader() tamregistros desconocido");
        free(conexIn); free(conexOut); if (pesosD) free(pesosD); if(pesosF) free(pesosF);
        return(1);
    }
    // verifica si se leyeron correctamente.
    if (leidos!=((4*headerSNN.numConex)+1+conf->tamListaConexPost[indexpob]))
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 350.5 en snnDataLoader() llamando a fread()");
        free(conexIn); free(conexOut); if (pesosD) free(pesosD); if(pesosF) free(pesosF);
        return(1);
    }
    // hasta aqu� se puede retornar sinmodificar el genoma por tanto, se retorna 1.
    // crea nuevo genoma con numEntradas, numSalidas, numBias,etc..
    if (genomaInicial(indexpob, headerSNN.numEntradas, headerSNN.numSalidas, headerSNN.numBias, 0, especie, conf)==0)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 351 en snnDataLoader() llamando a genomaInicial()");
        return(0);
    }
    // calcula el total de nodos del SNN
    tmpNumNodos=headerSNN.numEntradas+headerSNN.numSalidas+headerSNN.numBias+headerSNN.numHiddens;
    // verifica que valor leido no sea demasiado grande o corrupto
    if (tmpNumNodos>640000)
    {
        fclose(conf->logFile);
        conf->logFile=fopen(conf->fileNameLog,"a+");
        fprintf(conf->logFile,"<br>\nError 352 en snnDataLoader() n�mero de nodos de un SNN demasiado grande>64k.");
        return(0);
    }
    // reconstruye el genoma indexpob a partir de los arreglos leidos
    // para cada conexion i
    for (i=0;i<headerSNN.numConex;i++)
    {
        // si conexIn==pob[indexpob].numNodos;
        if (conexIn[i]==conf->pob[indexpob].totalNodos)
        {
            // busca en las conex actuales la que tiene ConexIn[i+1],conexOut[i]
            j=0;
            while ((j<i)&&((conexIn[i+1]!=conf->pob[indexpob].conex[j].indexIn)||(conexOut[i]!=conf->pob[indexpob].conex[j].indexOut)))
                j++;
            // si no la encontr�, imprime error
            if (j==i)
            {
                fclose(conf->logFile);
                conf->logFile=fopen(conf->fileNameLog,"a+");
                fprintf(conf->logFile,"<br>\nError 352.1 en snnDataLoader() conexi�n no encontrada..");
                return(0);
            }
            // nuevoNodo(ConexIn[i+1],conexOut[i])
            if (nuevoNodo(indexpob,j,conf)==0)
            {
                fclose(conf->logFile);
                conf->logFile=fopen(conf->fileNameLog,"a+");
                fprintf(conf->logFile,"<br>\nError 352.1 en snnDataLoader() llamando a nuevoNodo..");
                return(0);
            }
            // actualizar peso y enabled de conex i
            conf->pob[indexpob].conex[i].enabled=enabled[i];
            if (headerSNN.tamRegistros==4)
            {
                conf->pob[indexpob].conex[i].peso = pesosF[i];
            }
            else if (headerSNN.tamRegistros==8)
            {
                conf->pob[indexpob].conex[i].peso = (float)pesosD[i];
            }
        }
        // si el nodo de entrada ya existe,
        else
        {
            // busca en conex actuales la que tenga conexIn[i] y conexOut[i]
            j=0;
            while ((j<i)&&((conexIn[i]!=conf->pob[indexpob].conex[j].indexIn)||(conexOut[i]!=conf->pob[indexpob].conex[j].indexOut)))
                j++;
            //si existeConex(conexin a conexout)
            if (j!=i)
            {
                //actualiza peso y enabled
                conf->pob[indexpob].conex[i].enabled=enabled[i];
                if (headerSNN.tamRegistros==4)
                {
                    conf->pob[indexpob].conex[i].peso = pesosF[i];
                }
                else if (headerSNN.tamRegistros==8)
                {
                    conf->pob[indexpob].conex[i].peso = (float)pesosD[i];
                }
            }
            // si no existe
            else
            {
                // verifica que el nodoIn y el Out Existan
                if ((conexIn[i]<conf->pob[indexpob].totalNodos)&&(conexOut[i]<conf->pob[indexpob].totalNodos))
                {
                    // crea la nueva conex(conexin a conexout)
                    nuevaConex(indexpob,conexIn[i],conexOut[i],((headerSNN.tamRegistros==4)? pesosF[i]: (float)pesosD[i]),0,enabled[i],conf);
                }
            }
        }
    }
    conf->pob[indexpob].fitness=(float)headerSNN.lastFitness;
    free(conexIn);
    free(conexOut);
    free(enabled);
    if (pesosF)
        free(pesosF);
    if (pesosD)
        free (pesosD);
    return(1);
}

 float calcularValorNodo(Genoma* pGenoma, int indexPob,GenNodoF* nodo, int indexNodo, TConfig* conf)  //OPTIMIZADA TODO: deber�a ir en gen.c
{
    // calcula recursivamente valores de salida (despu�s de pasar por fsigma) de un nodo de un genoma indicado.
    // retorna el valor del nodo calculado, tambi�n lo asigna a valor y coloca en 1 valorCalculado.
    // par�metros:  indexPob = index del genoma en la poblaci�n.
    //              nodo = puntero al nodo para el cual se quiere calcular el valor.
    unsigned i;
    unsigned contHijos = nodo->contHijos; // para que no se tenga que que hacer operaci�n de consulta cada vez durante el for durante la condici�n.
    GenNodoF* pNodoHijo; //usado para acelerar el calculo
    float acum = 0;
    // verifica si el valor del nodo ya ha sido calculado.
    /*	if (nodo->estadoC == 2){ // si el valor del nodo ya fu� calculado, solo lo retorna.
            return(nodo->valor);
    	}
    */
    if (nodo->estadoC == 1)  // si el valor del nodo ha empezado ha calcularse, pero a�n no se tiene resultado retona el valor anterior.
    {
        // TODO: Para multiprocesamiento, se debe verificar si el nodo hijo corresponde a alguno de los padres
        // para procesamiento de un solo hilo, se retorna el valor anterior si es solicitado por
        return(nodo->valor);
    }
    // marca el valor del nodo como calculado, debe ir aqu� por si recursivamente es solicitado este nodo, se retorne su valor anterior
    // tambi�n por esta raz�n se debe llamar calcularValorNodo exclusivamente en las salidas para que se calcule hacia abajo.
    nodo->estadoC = 1;
    // llena el valor de conf->ordenEval[indexpob][conf->contO[indexpob]] y incrementa conf->contO[indexpob]
//printf("oE[%d][%d]=%d,",indexPob,conf->contO[indexPob],indexNodo);
    conf->ordenEval[indexPob][conf->contO[indexPob]]=indexNodo;
    conf->contO[indexPob]++;
    // :)
    // FALTA: implementar ordenador de ordenFitness (sirve el que ya hice?)
    // FALTA: modificar guardarGenoma para agregar el arreglo ordenEval[indexpob][contO[indexpob]]
    // FALTA: // verificar si no es necesario para experimento cargarSNN.
    // FALTA: modificar SNNeval
    // para todos los hijos del nodo.
//PROBANDO, la sig linea no iba
    if (nodo->nodeFunction==0)//nodo->valor;
    {
//        if (contHijos==0)
 //           return(nodo->valor);
  //      else
            acum=nodo->valor;
    }
    for (i=0; i<contHijos; i++)
    {
        // si la conex con el hijo est� enabled
        if (nodo->conexHijo[i]->enabled==1)  //TODO: Cuando est� funcionando, no colocar conexiones disableds en actualizarPNodos y quitar esta comprobaci�n
        {
            // si el valor del nodo ya ha sido calculado
            pNodoHijo = &(pGenoma->nodo[nodo->conexHijo[i]->indexIn]);
            if (pNodoHijo->estadoC==1)
            {
                acum += nodo->conexHijo[i]->peso * pNodoHijo->valor;
            }
            else  //Si el valor del nodo hijo no ha sido calculado ERROR, falta INDEXNODO de nodo HIJO
            {
                acum += nodo->conexHijo[i]->peso*calcularValorNodo(pGenoma,indexPob,pNodoHijo,nodo->conexHijo[i]->indexIn,conf);
            }
        }
    }
    // retorna el fsigma.
    return(nodo->valor = fSigma(acum-nodo->thNodo,conf->tSigma,conf->fSigmaD,conf));
}

unsigned actualizarPNodos(unsigned index, TConfig* conf) //OPTIMIZADA
{
    //actualiza los valores de los punteros a conexiones hijo para cada nodo de un genoma.
    //se debe llamar al inicio de evaluarGenoma.
    //retorna 0 si hay error, 1 si ok
    unsigned i;
    unsigned j,k;
    Genoma* pGenoma = &(conf->pob[index]);
    GenNodoF* pNodo; //usado para acelerar los calculos
    //para cada gen nodo del genoma se reserva memoria para arreglo de conexHijo.
    for (i=0; i<pGenoma->totalNodos; i++)
    {
        pNodo = &(pGenoma->nodo[i]);
        if ( pNodo->nodeFunction != 3)
        {
            if ((pNodo->conexHijo = (GenConexF**)malloc(sizeof(GenConexF*)*pNodo->contHijos))==NULL)
            {
                fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 1101 en Funcion actualizarPNodos() llamando a malloc() o contHijos=0");
                return(0);
            }
        }
        else
        {
            pNodo->contHijos=0;
            pNodo->conexHijo=NULL;
        }
        //para cada gen conex del genoma: busca las que tienen como indexOut a i y adiciona un puntero a esta conexi�n como hijo del nodo
        k=0;
        for (j=0; j<pGenoma->totalConexiones; j++)
        {
            if(pGenoma->conex[j].indexOut==i)
            {
                //adiciona al arreglo de conexHijo el ptr a la conex j
                /*				if (pGenoma->nodo[i].contHijos<=k){
                                    fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 503.5 en actualizarPNodos() :pGenoma->nodo[i].contHijos<=k ");
                                    return(0);
                				}
                */
                pGenoma->nodo[i].conexHijo[k] = &(pGenoma->conex[j]);
                k++;
            }
        }
    }

    /*    //para cada gen conex del genoma:
        for (i=0;i<pGenoma->totalConexiones;i++){
            //adiciona al arreglo de conexHijo el ptr a la conex i
            indexNodoPadre=pGenoma->conex[i].indexOut;
            //para cada hijo
            for (j=0;j<conf->pob[index].nodo[indexNodoPadre].contHijos;j++){
                pGenoma->nodo[indexNodoPadre].conexHijo[j] = &(pGenoma->conex[i]);
            }
        }
    */
    return(1);
}

unsigned copiarGenoma(unsigned srcindexpob,unsigned dstindexpob,TConfig* conf)  //OPTIMIZADA
{
//Copia el genoma de origen al genoma de destino (el de destino es borrado)
//SUPONE que ya se ha reservado memoria para el array conf->pob que incluye a los dos elementos: funcion inicializarPob();
//Retorna 0 si hay error, 1 si Ok
    Genoma* pGenomaDst=&(conf->pob[dstindexpob]);
    Genoma* pGenomaSrc=&(conf->pob[srcindexpob]);
    // si fuente y destino son iguales, solo retorna 1
    if (srcindexpob==dstindexpob)
        return(1);
    //copia el contenido de conf->pob[src] al contenido de  conf->pob[dest] (copia un Genoma)
    if (pGenomaDst->nodo!=NULL) free((void *)pGenomaDst->nodo);
    if (pGenomaDst->conex!=NULL) free((void *)pGenomaDst->conex);
    //Copia los valores de la estructura genoma en el indexPob de destino.
    *pGenomaDst=*pGenomaSrc;
    //reserva memoria en el puntero conf->pob[dest].nodo con tama�o conf->pob[dest].totalNodos
    if ((pGenomaDst->nodo=(GenNodoF *) malloc(pGenomaSrc->totalNodos * ((unsigned  int)sizeof(GenNodoF))))==NULL)
    {
        fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 31 en funcion copiarGenoma(%u,%u) llamando a malloc(%u).\n",srcindexpob,dstindexpob,pGenomaSrc->totalNodos * ((unsigned  int)sizeof(GenNodoF)));
        return(0);
    }
    //y copia el contenido del puntero conf->pob[src].nodo al contenido de conf->pob[dest].nodo
    pGenomaDst->nodo=(GenNodoF*)memcpy(pGenomaDst->nodo,pGenomaSrc->nodo, pGenomaSrc->totalNodos * ((unsigned  int)sizeof(GenNodoF)));
    if ((pGenomaDst->conex=(GenConexF *) malloc(pGenomaSrc->totalConexiones * ((unsigned  int)sizeof(GenConexF))))==NULL)
    {
        fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 32 en funcion copiarGenoma(%u,%u) llamando a malloc(%u).\n",srcindexpob,dstindexpob,pGenomaSrc->totalConexiones * ((unsigned  int)sizeof(GenConexF)));
        return(0);
    }
    //y copia el contenido del puntero conf->pob[src].conex al contenido de conf->pob[dest].conex
    pGenomaDst->conex=(GenConexF*)memcpy(pGenomaDst->conex,pGenomaSrc->conex, pGenomaSrc->totalConexiones * ((unsigned  int)sizeof(GenConexF)));
    return(1);

}

/* se puede dejar otra Funcion que calcule el valor solo con indice en la struct GenNodoF para ahorrar memoria(que puede ser limitada) pero sacrificando velocidad.
Estado anterior de unsigned evaluarGenoma(unsigned index ,unsigned primero, float *entradas, float *salidas, unsigned nEntradas, unsigned nSalidas, TConfig* conf){ //OPTIMIZANDO
// Funcion evaluarGenoma() para un genoma i obtiene los valores y los fitness NO ajustados(1-error) para cada NODO de un genoma (incluyendo las salidas)
// tambi�n acumula fitness en pob[index].fitness para ser procesado luego por evaluarPob
// Par�metros:
//				index = indice de genoma por par�metro y
//				primero = especifica si es la primera vez (=1) que se eval�a el genoma para inicializar si no es la primera vez=0
//				entradas = puntero a un arreglo de nEntradas valores float que ser�n las entradas a evaluar
//				salidas = puntero a un arreglo de nSalidas valores float que ser�n las salidas deseadas, respecto a las cuales se obtendr� el error y por tant el fitness = 1-error.
//				nEntradas = n�mero de elementos en el arreglo entradas
//				nSalidas = n�mero de elementos en el arreglo de salidas
//sus valores en 0 excepto los de las entradas.
//salida: retorna el genoma en el indice index evaluado para la entrada con la variable valor de la estructura GenNodoF evaluada.
//retorna 0 si hubo error.
//DEMORADA, probar velocidad haciendo primero funcion crearmatrix(genoma) y evaluarmatrix(), en lugar de evaluargenoma (puede ser otro par�metro de esta funcion)
////TODO:Colocarle nuevo par�metro para evaluar directamente(actual) o evaluar por generaci�n de matrix y evaluaci�n de matrix (como funciones?).
//contadores
	unsigned i=0;
	unsigned j=0;
	float acum=0;//para almacenar entradasoralmente la sumatoria de cada nodo.
	unsigned tmp=0;
	unsigned inicOcultas=(conf->numEntradas+conf->numSalidas+conf->numBias);//posici�n inicial de nodos ocultos
	//si es primera vez (primero=1) hace cero todos los dem�s conf->pob[index].GenNodo[i].Valor.
	if (primero==1){
		fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>Reseteando valores de salida de neuronas");
		for (i=nEntradas;i<conf->pob[index].totalNodos;i++){
			if (conf->pob[index].nodo[i].nodeFunction!=3)
				conf->pob[index].nodo[i].valor=0;
			}
	}
	//Coloca los elementos leidos en los valores de los nodos correspondientes de entrada
	for (i=0;i<nEntradas;i++){
//		conf->pob[index].nodo[i].valor=entradas[i]; QUITADO POR TESTING DE VALOR DE SALIDA DE NEURONAS DE ENTRADA
// //TODO: para cada par�metro (pesos y thresholds) se debe guardar un registro de si el �ltimo cambio aument� el fitness y el signo
//del incremento, si en el �ltimo incremento hubo mejora, continuar con el mismo signo
//sin�, usar el signo contrario para el pr�ximo incremento aleatorio.
		conf->pob[index].nodo[i].valor=fSigma(entradas[i]-conf->pob[index].nodo[i].thNodo,conf->tSigma,conf->fSigmaD);
	}

//TESTING: calcula loa valores de salida de las neuronas de entrada


//Calcula los valores  de salida de las neuronas ocultas.(Las entradas siempre est�n al principio)
	//Para todos los que tengan GenNodo(i).nodeFunction=1 (ocultas) busca todos los que tengan GenNodo(j).nodoOut=i y los suma en acum (multiplicados por el peso de la conexi�n).
	for (i=inicOcultas;i<conf->pob[index].totalNodos;i++){
		acum=0;
		if(conf->pob[index].nodo[i].nodeFunction==1){//para nodos ocultos////TODO ESTA CONDICION SE PUEDE QUITAR
			for (j=0;j<conf->pob[index].totalConexiones;j++){//para cada conex
				if(conf->pob[index].conex[j].enabled==1){ //Si la conexi�n est� enabled
					if(conf->pob[index].conex[j].nodoOut==conf->pob[index].nodo[i].innovNum){//si la conex j tiene como nodo de salida el nodo i.innovnum
						if((tmp=buscarInnovNodo(index,conf->pob[index].conex[j].nodoIn,conf))==UINT_MAX){//busca el index del nodo de entrada de la conex j
							fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 8 en Funcion evaluarGenoma(%u,%u,%u,%u) llamando a buscarInnovNodo(%u,%u)\n",index,primero,nEntradas,nSalidas,index,conf->pob[index].conex[j].nodoIn);
							return(0);
						}
						acum+=(conf->pob[index].conex[j].peso*conf->pob[index].nodo[tmp].valor);
					}
				}
			}
//Coloca en el valor de cada nodo i sigma de acum,threshold.
			conf->pob[index].nodo[i].valor=fSigma(acum-conf->pob[index].nodo[i].thNodo,conf->tSigma,conf->fSigmaD);
		}
	}
//Calcula los valores de las neuronas de salida (Las entradas siempre est�n al principio).
	//Para todos los que tengan GenNodo(i).function=0 (entradas) busca todos los que tengan GenNodo(j).nodoOut=i y los suma en acum (multiplicados por el peso de la conexi�n).
	for (i=(conf->numEntradas+conf->numBias);i<inicOcultas;i++){
		acum=0;
		if(conf->pob[index].nodo[i].nodeFunction==2){//para nodos de salida//ESTA CONDICION SE PUEDE QUITAR //TODO
			for (j=0;j<conf->pob[index].totalConexiones;j++){
				if(conf->pob[index].conex[j].enabled==1){ //Si la conexi�n est� enabled
					if(conf->pob[index].conex[j].nodoOut==conf->pob[index].nodo[i].innovNum){
						if((tmp=buscarInnovNodo(index,conf->pob[index].conex[j].nodoIn,conf))==UINT_MAX){
							fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 9 en Funcion evaluarGenoma(%u,%u) llamando a buscarInnovNodo(%u,%u)\n",index,primero,index,conf->pob[index].conex[j].nodoIn);
							imprimirGenoma(index,conf);
							return(0);
						}
						acum+=(conf->pob[index].conex[j].peso*conf->pob[index].nodo[tmp].valor);
					}
				}
			}
//Coloca en el valor de cada nodo i sigma de acum,threshold.
			conf->pob[index].nodo[i].valor=fSigma(acum-conf->pob[index].nodo[i].thNodo,conf->tSigma,conf->fSigmaD);
		}
	}

//Habiendo calculado las salidas, obtiene el fitness (NO AJUSTADO) de las salidas actuales respecto a las salidas
//el error el es el promedio de errores ABSOLUTOS de todas las salidas respecto a las salidas deseadas.
	acum=0;
	j=0;
	for (i=(conf->numEntradas+conf->numBias);i<(inicOcultas);i++){//para nodos de salida
		if(conf->pob[index].nodo[i].nodeFunction==2){//TODO para nodos de salida//ESTA CONDICION SE PUEDE QUITAR
			if (j==nSalidas){//TODO ESTE BLOQUE SE PUEDE QUITAR //TODO
				fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 10 en funcion evaluarGenoma(%u,%u) n�mero de salidas %u no corresponde con neuronas de salida en el genoma.",index,primero,nSalidas);
				return(0);
			}
			acum=acum+fabs(conf->pob[index].nodo[i].valor-salidas[j]);////TODO , mejorar CALCULO (se quit� normalizaci�n) DE ESTE ERROR  SI salida[j]=0
			j++;
		}
	}

	conf->pob[index].fitness += (acum/(float)conf->numSalidas);

	//Actualiza la lista de representantes (se busca el MENOR fitness debido a que todav�a no se ha calculado el fitness real)
	if(conf->pob[index].fitness<conf->representantes[conf->pob[index].especie]){
		conf->representantes[conf->pob[index].especie]=index;
	}

	//fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>F%1.1f\n",conf->pob[index].fitness);
//	if(conf->pob[index].fitness<0)
//	{
//	fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>Ins=%1.1f,%1.1f, Nout=%1.1f, d=%1.1f fit=%1.1f, acum=",conf->pob[index].nodo[0].valor,conf->pob[index].nodo[1].valor,conf->pob[index].nodo[3].valor,salidas[0],conf->pob[index].fitness);
//		return(0);
//	}
// :)

	return(1);
}*/

unsigned crossover(unsigned indexpob1, unsigned indexpob2, unsigned indexpobOut ,float super, float promediarPob,float porcentEnableds,TConfig* conf) //OPTIMIZADA TODO: deber�a ir en especie
{
//Realiza el cruce entre dos genomas dados sus indexpob y lo coloca en un elemento de conf->pob (puede ser uno de los padres)
//Toma calcula el fitness de los dos genomas, se heredan los matching genes randomly,
//Los disjounsigned y excess se heredan solo del fittest,
//Par�metros: 	indexpob1, indexpob2 = genomas a cruszar
//				indexpobOut = indexpob donde se debe colocar el genoma resultante del cruce.
//				super = float entre 0 y 1, Probabilidad de heredar los exess y disjounsigned  ints del menos apto (aparte de los que se heredan normalmente del m�s apto)
//				promendiarProb = float entre 0 y 1 = probabilidad de que en caso de matching, se promedien los pesos en lugar de
    //seleccionarlos aleatoriamente entre los padres.
//Retona 0 si hubo error, 1 si ok, coloca en la variable global tempGenoma el genoma generado por crossover
    unsigned mejor=indexpob1;
    unsigned peor=indexpob2;
    unsigned i;
    unsigned j;
    Genoma* pGenomaMejor; //usados para acelerar calculoa
    Genoma* pGenomaPeor;

    //TODO QUITAR: verifica que los m�ximos innov num y m�ximos numNodo y conex correspondan con los que se encuentran en los genoma
    if ((verificarGenoma(indexpob1,conf)==0)||(verificarGenoma(indexpob2,conf)==0)||(verificarGenoma(indexpobOut,conf)==0))
    {
        fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 46.0 en funcion crossover(%u,%u,%u,%1.1f,%1.1f) llamando a verificarGenomas()\n",indexpob1, indexpob2, indexpobOut , super, promediarPob);
        return(0);
    }//TODO QUITAR CUANDO NO HAYA ERRORES

    // coloca en mejor el indexpob del que tiene mayor fitness
    if (conf->pob[indexpob2].fitness>conf->pob[indexpob1].fitness)
    {
        mejor=indexpob2;
        peor=indexpob1;
    }
    // si fitness mayor= fitness menor, si super=0 entonces mejor=el que tenga menor totalconexiones, peor el que tenga mayor totalConexiones
    if (conf->pob[indexpob2].fitness==conf->pob[indexpob1].fitness)
    {
        if (conf->pob[indexpob2].totalConexiones<conf->pob[indexpob1].totalConexiones)
        {
            mejor=indexpob2;
            peor=indexpob1;
        }
    }
    // asigna punteros al mejor y peor genoma
    pGenomaMejor = &(conf->pob[mejor]);
    pGenomaPeor = &(conf->pob[peor]);

    // probando
   // fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>%u>%u,", mejor,indexpobOut);
   // copiarGenoma(mejor, indexpobOut,conf);

// TODO:Probando nuevo crossover que solo copia El mejor a hijo si hijo es =! peor  y luego comparando conex a conex (y nodo por fdt?)
//      en caso de genes con ==innovNumCon, se debe escoger uno de los dos pesos al azar para esa conex en el Hijo.
// TODO: PROBANDO POR ERROR DE SC

    // si hijo es != peor
    if (indexpobOut!=peor)
    {
        // copia mejor a hijo
        copiarGenoma(mejor,indexpobOut,conf); //   :)
    }
    // busca en el peor conexiones con =innovnum a alguna de m
    for (i=0;i<conf->pob[peor].totalConexiones;i++) // barre econex del peor
    {
        for (j=0;j<conf->pob[mejor].totalConexiones;j++) // barre conex del mejor
        {
            if (conf->pob[mejor].conex[j].innovNum==conf->pob[peor].conex[i].innovNum) // si los innovnums son iguales
            {
                if (((float)randL(conf))<=0.5){ // escoge al azar entre los dos.
                    // si el hijo no es el peor, asigna el peso del peor, porque originalmente ten�a el del mejor.
                    if (indexpobOut!=peor)
                    {
                        conf->pob[indexpobOut].conex[j].peso=conf->pob[peor].conex[i].peso;
                        //solo si estaba disabled en el mejor la copia act/desact, para evitar no-viables.
                        if (conf->pob[indexpobOut].conex[j].enabled==0)
                            conf->pob[indexpobOut].conex[j].enabled=conf->pob[peor].conex[i].enabled;
                    }
                    // si el hijo es el peor, asigna el peso del mejor, porque originalmente ten�a el del peor.
                    else
                    {
                        conf->pob[indexpobOut].conex[i].peso=conf->pob[mejor].conex[j].peso;
                        //solo si estaba disabled en el hijo(peor) la copia act/desact, para evitar no-viables.
                        if (conf->pob[indexpobOut].conex[i].enabled==0)
                            conf->pob[indexpobOut].conex[i].enabled=conf->pob[mejor].conex[j].enabled;
                    }
                }
            }
        }
    }


/** Probando nuevo crossover que solo copia El mejor a hijo si hijo es =! peor  y luego comparando conex a conex (y nodo por fdt?)
    //en caso de genes con ==innovNumCon, se debe escoger uno de los dos pesos al azar para esa conex en el Hijo.

    // copia com malloc+memcpy el pGenomaMejor->nodo[i]  (uno por uno creo) a tempGenoma.nodo
    if((tempGenoma.nodo=malloc(sizeof(GenNodoF)*pGenomaMejor->totalNodos))==NULL)
    {
        fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 40 en funcion crossover(%u,%u,%u,%1.1f,%1.1f) llamando a malloc(%u)\n", indexpob1, indexpob2, indexpobOut , super, promediarPob,pGenomaMejor->totalNodos*(unsigned int)sizeof(GenNodoF) );
        return(0);
    }
    if ((tempGenoma.nodo=memcpy(tempGenoma.nodo,pGenomaMejor->nodo,sizeof(GenNodoF)*pGenomaMejor->totalNodos))==NULL)
    {
        fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 41 en funcion crossover(%u,%u,%u,%1.1f,%1.1f) llamando a memcpy(), el arreglo de nodos del mejor es NULL\n", indexpob1, indexpob2, indexpobOut , super, promediarPob );
        return(0);
    }
    tempGenoma.totalNodos=pGenomaMejor->totalNodos;
    tempGenoma.maxInnovNumNodo=pGenomaMejor->maxInnovNumNodo;
    tempGenoma.maxInnovNumConex=pGenomaMejor->maxInnovNumConex;
    //para todos los nodos hace el contHijos=0;
    for (i=0; i<tempGenoma.totalNodos; i++)
    {
        tempGenoma.nodo[i].contHijos=0;
        tempGenoma.nodo[i].conexHijo=NULL; //al copiar inicializa este puntero en null para el nuevo elemento(Se configura en actualizarPNodos).
    }
    // actualiza totalconexiones y reserva memoria para nodos.
    tempGenoma.totalConexiones=pGenomaMejor->totalConexiones;
    tempGenoma.conex=(GenConexF *) malloc(sizeof(GenConexF)*tempGenoma.totalConexiones);
    if(tempGenoma.conex==NULL)
    {
       fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 42.1 en funcion crossover(%u,%u,%u,%1.1f,%1.1f) llamando a malloc(%u)\n",indexpob1, indexpob2, indexpobOut , super, promediarPob,tempGenoma.totalConexiones*(unsigned int)sizeof(GenConexF));
       return(0);
    }
    // para cada i desde 0, mientras i<pGenomaMejor->totalConexiones
    for (i=0; i<pGenomaMejor->totalConexiones; i++)
    {
        pTempGenomaConex = &(tempGenoma.conex[i]);
        // busca en conf->pob[peor] , pGenomaMejor->conex[i].innovNum, si lo encuentra, escoge leatoriamente entre mejor y peor y lo copia a tempgenoma
        if((j=buscarInnovConex(peor,pGenomaMejor->conex[i].innovNum,conf))!=UINT_MAX)
        {
            //TODO: verificar si este parametro funciona por encima de cero en redes grandes sin� quitarlo.
            if (((float)randL(conf))<=promediarPob	) //los promedia
            {
                *pTempGenomaConex=pGenomaMejor->conex[i];
                pTempGenomaConex->peso=(pGenomaMejor->conex[i].peso+pGenomaPeor->conex[j].peso)/2;

            }
            else //escoge aleatoriamente entre el mejor y el peor
            {
                //TODO: Probar colocando como par�metro si el porcentaje de probabilidad de heredar de mejor o peor y si influye en redes grandes
                if (((float)randL(conf))<=0.5)  //TODO: PUEDE HABER ERROR AQUI SE PODR�A USAR GENOMA EN INDEXPOB DESPUES DE
                                                    //REPS
                    *pTempGenomaConex=pGenomaMejor->conex[i];
                else
                    *pTempGenomaConex=pGenomaPeor->conex[j];
            }
            if(pGenomaMejor->conex[i].enabled!=pGenomaPeor->conex[j].enabled)
            {
                if(((float)randL(conf))<porcentEnableds)
                    pTempGenomaConex->enabled=pGenomaPeor->conex[j].enabled;
                else
                    pTempGenomaConex->enabled=pGenomaMejor->conex[i].enabled;
            }
        }
        // si no lo encuentra, copia la conexi�n del mas apto a tempGenoma.
        else
        {
            // copia la conexi�n del mejor a tempgenoma
            *pTempGenomaConex=pGenomaMejor->conex[i];
        }
        //incrementa el contHijo del nodo padre de la conexi�n
        tempGenoma.nodo[pTempGenomaConex->indexOut].contHijos++;
    }
    //TODO: Verificar si este par�metro funciona en redes grandes, sin� quitarlo
    if (((float)randL(conf))<=super)
    {
        //Copia a tempGenoma los genes de nodo que no existan en tempGenoma.nodo.
        k=tempGenoma.totalNodos;
        for (i=0; i<pGenomaPeor->totalNodos; i++)
        {
            genFound=0;
            for (j=0; j<k; j++)
            {
                if(tempGenoma.nodo[j].innovNum==pGenomaPeor->nodo[i].innovNum)
                    genFound=1;
            }
            if (genFound==0)  //si no se encontr� en tempGenoma, se le adiciona.
            {
                if ((tempGenoma.nodo=(GenNodoF *)realloc(tempGenoma.nodo,sizeof(GenNodoF)*(tempGenoma.totalNodos+1)))==NULL)
                {
                    fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 43 en funcion crossover(%u,%u,%u,%1.1f,%1.1f) llamando a realloc(tempGenoma.nodo,%u)\n",indexpob1, indexpob2, indexpobOut , super, promediarPob,(tempGenoma.totalNodos+1)*(unsigned int)sizeof(GenNodoF));
                    free(tempGenoma.conex);
                    return(0);
                }
                tempGenoma.nodo[tempGenoma.totalNodos++]=pGenomaPeor->nodo[i];
                if (pGenomaPeor->nodo[i].innovNum>tempGenoma.maxInnovNumNodo)
                    tempGenoma.maxInnovNumNodo=pGenomaPeor->nodo[i].innovNum;
            }
        }
        //Copia a tempGenoma los genes disjounsigned o excess de pGenomaPeor->conex
        k=tempGenoma.totalConexiones;
        for (i=0; i<pGenomaPeor->totalConexiones; i++)
        {
            genFound=0;
            for (j=0; j<k; j++)
            {
                if(tempGenoma.conex[j].innovNum==pGenomaPeor->conex[i].innovNum)
                    genFound=1;
            }
            if (genFound==0)  //si no se encontr� en tempGenoma, se le adiciona.
            {
                if ((tempGenoma.conex=(GenConexF *)realloc(tempGenoma.conex,sizeof(GenConexF)*(tempGenoma.totalConexiones+1)))==NULL)
                {
                    fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 44 en funcion crossover(%u,%u,%u,%1.1f,%1.1f) llamando a realloc(tempGenoma.nodo,%u)\n",indexpob1, indexpob2, indexpobOut , super, promediarPob,(tempGenoma.totalConexiones+1)*(unsigned int)sizeof(GenConexF));
                    return(0);
                }
                tempGenoma.conex[tempGenoma.totalConexiones++]=pGenomaPeor->conex[i];
                if (pGenomaPeor->conex[i].innovNum>tempGenoma.maxInnovNumConex)
                    tempGenoma.maxInnovNumConex=pGenomaPeor->nodo[i].innovNum;
            }
        }
    }
    // coloca la especie igual a la del mejor y fitness en 0
    tempGenoma.especie=pGenomaMejor->especie;
    tempGenoma.fitness=0;
    // libera la mamoria usada por el arreglo de nodos y conexiones del nodo hijo a reemplazar.
    if (pGenomaOut->nodo!=NULL) free((void *)pGenomaOut->nodo);
    if (pGenomaOut->conex!=NULL) free((void *)pGenomaOut->conex);
    // copia el tempGenoma al indexpob genoma de la pob.
    *pGenomaOut=tempGenoma;
    // copia tempGenoma a pGenomaOut->nodo
    if((pGenomaOut->nodo=(GenNodoF *)malloc(sizeof(GenNodoF)*tempGenoma.totalNodos))==NULL)
    {
        fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 45 en funcion crossover(%u,%u,%u,%1.1f,%1.1f) llamando a malloc(%u)\n",indexpob1, indexpob2, indexpobOut , super, promediarPob,(tempGenoma.totalNodos)*(unsigned int)sizeof(GenNodoF));
        free(tempGenoma.conex);
        return(0);
    }
    pGenomaOut->nodo=memcpy(pGenomaOut->nodo,tempGenoma.nodo,sizeof(GenNodoF)*tempGenoma.totalNodos);
    // copia tempGenoma a pGenomaOut->conex
    if((pGenomaOut->conex=(GenConexF *)malloc(sizeof(GenConexF)*tempGenoma.totalConexiones))==NULL)
    {
        fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 46 en funcion crossover(%u,%u,%u,%1.1f,%1.1f) llamando a malloc(%u)\n",indexpob1, indexpob2, indexpobOut , super, promediarPob,(tempGenoma.totalConexiones)*(unsigned int)sizeof(GenConexF));
        free(tempGenoma.conex);
        return(0);
    }
    pGenomaOut->conex=memcpy(pGenomaOut->conex,tempGenoma.conex,sizeof(GenConexF)*tempGenoma.totalConexiones);
    //TODO: QUITAR esta Funcion si no hay errores
    if (verificarGenoma(indexpob1,conf)==0)
    {
        fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 46.5 en funcion crossover() llamando a verificarGenomas()\n");
        free(tempGenoma.conex);
        return(0);
    }
    if (verificarGenoma(indexpob2,conf)==0)
    {
        fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 46.6 en funcion crossover() llamando a verificarGenomas()\n");
        free(tempGenoma.conex);
        return(0);
    }
    if (verificarGenoma(indexpobOut,conf)==0)
    {
        fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 46.7 en funcion crossover() llamando a verificarGenomas()\n");
        free(tempGenoma.conex);
        return(0);
    }

    //calcula el max y min Th para el genoma nuevo.
    calcularLimThOneGenome(indexpobOut,conf);
    //libera la memoria usada por el arreglo de nodos y conex de tempgenoma
    if (tempGenoma.nodo!=NULL) free((void *)tempGenoma.nodo);
    if (tempGenoma.conex!=NULL) free((void *)tempGenoma.conex);
*/
    return(1);

}

unsigned evaluarGenoma(int indexPob, Genoma* pGenoma, unsigned primero, float *entradas, float *salidas, TConfig* conf)  //OPTIMIZADA
{
// Funcion evaluarGenoma() para un genoma i obtiene los valores y los fitness NO ajustados(1-error) para cada NODO de un genoma (incluyendo las salidas)
// tambi�n acumula fitness en pob[index].fitness para ser procesado luego por evaluarPob
// Par�metros:
//				index = indice de genoma por par�metro y
//				primero = especifica si es la primera vez (=1) que se eval�a el genoma para inicializar si no es la primera vez=0
//				entradas = puntero a un arreglo de nEntradas valores float que ser�n las entradas a evaluar
//				salidas = puntero a un arreglo de nSalidas valores float que ser�n las salidas deseadas, respecto a las cuales se obtendr� el error y por tant el fitness = 1-error.
//				nEntradas = n�mero de elementos en el arreglo entradas
//				nSalidas = n�mero de elementos en el arreglo de salidas
// sus valores en 0 excepto los de las entradas.
// salida: retorna el genoma en el indice index evaluado para la entrada con la variable valor de la estructura GenNodoF evaluada.
// retorna 0 si hubo error.
// contadores
    unsigned i=0;
    unsigned j=0;
    float acum=0;
    float tmp;
    GenNodoF* pNodo;
    unsigned numEntradas=conf->numEntradas;
    unsigned numEntradasBias = numEntradas + conf->numBias;
    unsigned inicOcultas=(numEntradasBias+conf->numSalidas);//posici�n inicial de nodos ocultos
    // si es primera vez (primero=1) hace cero todos los dem�s pGenoma->GenNodo[i].Valor y marca valornocalculado.
//PROBANDO PARA VER SI SE PUEDE LLEGAR A 90% CON ENTRADAS EN FSIGMA
    if (primero)
    {
        for (i=0; i<pGenoma->totalNodos; i++)
        {
            if ((pNodo = &(pGenoma->nodo[i]))->nodeFunction!=3)
            {
                pNodo->valor=0;
                pNodo->estadoC=0;
            }
        }
    }
    else  // si no es primero, solo marca como no actualizado el valor de cada nodo del genoma que no sea entrada o bias.
    {
//PROBANDO PARA VER SI SE PUEDE LLEGAR A 90% CON ENTRADAS EN FSIGMA
        //for (i=numEntradasBias; i<pGenoma->totalNodos; i++)
        for (i=0; i<pGenoma->totalNodos; i++)
        {
            if ((pNodo = &(pGenoma->nodo[i]))->nodeFunction!=3)
            {
                pNodo->estadoC=0;
            }
        }
    }
    // inicializa los contO[indexPob]=0;
    conf->contO[indexPob]=0;
    // coloca los elementos leidos en los valores de los nodos correspondientes de entrada
    for (i=0; i<numEntradas; i++)
    {
        //MOdificado en version0.66 para probar entrada lineal
        pGenoma->nodo[i].valor=entradas[i]; //FUNCIONA MEJOR EN REDES GRANDES QUE CON FSIGMA
        // TODO: para cada par�metro (pesos y thresholds) se debe guardar un registro de si el �ltimo cambio aument� el fitness y el signo
        //del incremento, si en el �ltimo incremento hubo mejora, continuar con el mismo signo
        //sin�, usar el signo contrario para el pr�ximo incremento aleatorio.
        //pNodo=&(pGenoma->nodo[i]);
        //pNodo->valor=fSigma(entradas[i]-pNodo->thNodo,conf->tSigma,conf->fSigmaD,conf);
    }
    // calcula los valores de las neuronas de salida (Las entradas siempre est�n al principio).
    // Para todos los que tengan GenNodo(i).function=0 (entradas) busca todos los que tengan GenNodo(j).nodoOut=i y los suma en acum (multiplicados por el peso de la conexi�n).
    for (i=numEntradasBias; i<inicOcultas; i++)
    {
        pGenoma->nodo[i].valor=calcularValorNodo(pGenoma,indexPob,&(pGenoma->nodo[i]),i,conf);
    }
    // habiendo calculado las salidas, obtiene el fitness (NO AJUSTADO) de las salidas actuales respecto a las salidas
    // el error el es el promedio de errores ABSOLUTOS de todas las salidas respecto a las salidas deseadas.
    acum=0;
    j=0;
    //TODO: este for y el anterior se pueden unir pero quitando acum del anterior for.

    for (i=numEntradasBias; i<inicOcultas; i++) //para nodos de salida
    {
        //TODO: hacer funci�n de error(para que acum=acum+fError) de be tener como params el nodo[i].valor y la salida[j], el valor de retorno debe sor positivo entre 0 y 1
        tmp=fError(pGenoma->nodo[i].valor,salidas[j]);
		//fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>OutR=%3.3f, OutC=%3.3f, Error=%3.3f\n", salidas[j],pGenoma->nodo[i].valor, tmp );
        //if (index==conf->representantes[0]) fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>tmp=%3.3f, acum=%3.3f \n",pGenoma->nodo[i].valor, salidas[j]);
        if (tmp==-1)
        {
            fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>Error 345 en evaluarGenoma() llamando a fError, error negativo");
            return(0);
        }
        acum=acum+tmp;////TODO , mejorar CALCULO (se quit� normalizaci�n) DE ESTE ERROR  SI salida[j]=0
        j++;
    }
    // incrementa el fitness con el acumulado de error dividido entre el n�mero de salidas (usado en evaluarpob para calculo de fitness total del genoma)
    pGenoma->fitness += acum;//TODO: verificar si funciona quitando la divisi�n por el n�mero de salidas. antes era(acum/(float)conf->numSalidas)
    return(1);
}
//TODO: Falta adicionar a lista de hijos de padre en funciones de nuevoNodo y NuevaConex, verificar si las dos mutaciones funcionan bi�n con eso
//	tambi�n verificar el genoma inicial y marcar como valorCalculado=1 y numHijos=0 a las neuronas de entrada.
//	tambi�n verificar en crossover si los nodos del hijo quedan con la misma lista de nodos hijo y conexHijo.

unsigned genomaPerfecto(unsigned index, TConfig* conf)
{
//Coloca un genoma perfecto de xor en la posici�n deseada. retorna 0 si hay error, 1 swi ok
//parametros :index
    //genera genoma inicial (todos los th en 0.5)
    if (genomaInicial(index,conf->numEntradas,conf->numSalidas,conf->numBias,0,0,conf)==0)
    {
        fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 34 en Funcion genomaPerfecto() llamando a genomaInicial()\n");
        return(0);
    }
    //agrega nuevo nodo 4 entre 2 y 3
    if((nuevoNodo(index,2,conf))==0) ////TODO CURVAS DE I2 para seleccionar i
    {
        fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 19 en funcion genomaPerfecto() llamando a funcion nuevoNodo()\n");
        return(0);
    }
    //hace peso=0 de �ltima conex no se necesita bias
    conf->pob[index].conex[conf->pob[index].totalConexiones-1].peso=0;
    //agrega nueva conex entre 0 y 4 peso=1
    if(nuevaConex(index,0,4, 1.0, 0, 1,conf)==0)
    {
        //Si hubo error retona 0
        fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\Error 13 en funcion genomaPerecto() llamando a nuevaConex()\n");
        return(0);
    }
    //agrega conexi�n entre 1 y 4 peso =-1
    if(nuevaConex(index,1,4, -1.0, 0, 1,conf)==0)
    {
        //Si hubo error retona 0
        fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\Error 13 en funcion genomaPerecto() llamando a nuevaConex()\n");
        return(0);
    }
    //establece peso de -1 para la conex 0 (0->3)
    conf->pob[index].conex[0].peso=-1.0;
    //establece peso de 2 para la conexi�n 3 (de 4 a 3)
    conf->pob[index].conex[3].peso=2.0;
    //TODO: hacer funci�n de prean�lisis de entradas y salidas para establecer la ganancia de normalizaci�n de las entradas y de denormalizaci�n de salidas
    //TODO: la funci�n ed prean�lisis  tambi�n calcula los thresholds de las neuronas de entrada (para las dem�s es inicialmente 0.5)
    //TODO: verificar cu�l es el mejor threshold inicial ara ocultas y salida? usando evaluaciones de fsigma (es el m�ximo o la mitad del unsigned  intervaloinicialmente?)
    return(1);
}

unsigned mutarAC(unsigned indexpob, unsigned maxIntentos, TConfig* conf)  //OPTIMIZADA
{
//Agrega una conexi�n al azar usando la funcion nuevaconexi�n
//adiciona la nueva conex.
//tambi�n realiza la asignaci�n del innovNum de la conexi�n buscando en la lista(mediante in y out), sino existe, lo adiciona.
//Verifica que la conexi�n entre los nodos resultantes no exista.
//Se debe adem�s verificar que en casos de conexiones recurrentes, el nodo no sea de entrada o bias.
//Se verifica si la conexi�n seleccionada ya existe, si esto ocurre randomiza de  nuevo la entrada y salida (en mutar AC)
//hasta un n�mero m�ximo maxIntentosNuevacon para evitar que se entre en bucle inconf->fInito.
//si no encuentra una nueva conexi�n posible, retorna 0 pero no crea la conexi�n.
//y al haber deswcubierto solo conexiones redundantes, incrementa aleatoriamente el peso de la �ltima conexi�n encontrada??
//Si se crea satisfactoriamemnte la conexi�n se le asigna un peso de 1
//La especie se asigna con la funcion calcularEspecie para la primera generaci�n, luego se recalcula despu�s de cada mutaci�n+cruce.
//Par�metros: indexpob = indice de la conf->poblaci�n del genoma a mutar.
//Retorna 0 si hay error, 1 si ok, 2 si no se pudo agregar por l�mite de conexiones.
    unsigned i=0;
    unsigned j=0;
    unsigned k=0;
    unsigned l=0;
    unsigned innovIn=0;
    unsigned innovOut=0;
    float peso;
    //verifica que la conexi�n no exista:
    // para k=0 hasta maxIntentos:
    for (k=0; k<maxIntentos; k++)
    {
        // selecciona dos nodos random i,j menores a maxNodos El l�mite de neuronas es 32767 (RAND_MAX), se puede aumentar usando rand*rand/1073676289
        i=(unsigned  int)floor((((float)randL(conf))*(conf->pob[indexpob].totalNodos-1))+0.5);
        // NO se permiten conexiones a nodos bias como destino de la conexi�n
        do
        {
            j=(unsigned  int)floor((((float)randL(conf))*(conf->pob[indexpob].totalNodos-1))+0.5);
        }
        while(conf->pob[indexpob].nodo[j].nodeFunction==3);
        innovIn=conf->pob[indexpob].nodo[i].innovNum;
        innovOut=conf->pob[indexpob].nodo[j].innovNum;
        // si buscarInnovConexPorNodos == UINT_MAX
        if((l=buscarInnovConexPorNodos(indexpob,innovIn,innovOut,conf))==UINT_MAX)
        {
            // se debe adem�s verificar que en casos de conexiones recurrentes, el nodo no sea de entrada o bias.
            // 0=entrada,1=oculto,2=salida,3=bias.
            if (i==j)
            {
                if ((conf->pob[indexpob].nodo[i].nodeFunction==1)||(conf->pob[indexpob].nodo[i].nodeFunction==2))
                {
                    // adiciona la nueva conexi�n con la funcion:
                    // unsigned nuevaConex(indexpob,innovIn,innovOut, peso, recurrente, 1);//retorna 0 si hay error
                    peso=(((float)randL(conf))*4)-2; // (-2,2)peso aleatorio para la nueva conex (-2,2)
                    if(nuevaConex(indexpob,i,j, peso, 1, 1,conf)==0)
                    {
                        //Si hubo error retona 0
                        fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\Error 13 en funcion mutarAC(%u,%u) llamando a nuevaConex(%u,%u,%u,%u,%u,%u)\n",indexpob,maxIntentos,indexpob,innovIn,innovOut, 1, 1, 1);
                        return(0);
                    }
                    // retorna 1 si OK
                    /*	if (erificarGenoma(indexpob)==0){
                    		fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 14.4 en funcion MutarAC(%u,%u) llamando a verificarGenomas()\n",indexpob,maxIntentos);
                    		return(0);
                    	}*/
                    return(1);
                }
            }
            else
            {
                peso=(((float)randL(conf))*conf->pob[indexpob].nodo[j].contHijos*4)-(2*conf->pob[indexpob].nodo[j].contHijos); // (-1,1) por cada conthijo peso aleatorio para la nueva conex (-2,2)
                if(nuevaConex(indexpob,i,j, peso, 0, 1,conf)==0)
                {
                    //Si hubo error retona 0
                    fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 14 en funcion mutarAC(%u,%u) llamando a nuevaConex(%u,%u,%u,%u,%u,%u)\n",indexpob,maxIntentos,indexpob,innovIn,innovOut, 1, 0, 1);
                    return(0);
                }
                // retorna 1 si OK
                return(1);
            }
        }
        else
        {
            if(conf->pob[indexpob].conex[l].enabled==0)
            {
                if(((float)randL(conf))<conf->porcentEnableds)
                {
                    conf->pob[indexpob].conex[l].enabled=1;
                    //incrementa conCont del nodoout de la conex
                    calcularLimThOneNode(indexpob,conf->pob[indexpob].conex[l].nodoOut,conf);
                    fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>R(%u)",l);
                    return(1);
                }
            }
        }
    }
    return(2); //No se pudo encontrar una conexi�n no existente.
}

unsigned mutarAN(unsigned indexpob, TConfig* conf)  //OPTIMIZADA
{
//Agrega un nodo al azar usando la funcion nuevoNodo, entran en la selecci�n todas las conexiones existentes.
//reerva memoria para el nuevo tama�o del genoma con realloc y luego adiciona el nuevo nodo.
//�Es posible agregar un nodo bias? si, es necesario?
// Retorna 0 si hubo error , 1 si ok.
// Par�metros:	indexpob	= index de conf->pob que se desea mutar.
    float i=0;
    ////TODO:Sustituir todos los rand por (float)(float)rand()
    //Crea el nuevo nodo con la fuci�n nuevoNodo(unsigned indexpob, unsigned indexElimInnovConex )
    if (conf->pob[indexpob].totalConexiones>conf->minConexMutPeso)
    {
        //Selecciona al azar un n�mero i entre 0 y maxConex-1 (como rand solo va hasta 32k, se usa rand*rand)
        i=(float)randL(conf);
        i*=((float)conf->pob[indexpob].totalConexiones-1);
        if((nuevoNodo(indexpob,(unsigned  int)i,conf))==0) ////TODO CURVAS DE I2 para seleccionar i
        {
            fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 19 en funcion mutarAN(%u) llamando a funcion nuevoNodo(%u,%u)\n",indexpob,indexpob,(unsigned  int)i);
            return(0);
        }
        return (1);
    }
    return (1);
}

unsigned nuevaConex(unsigned indexpob,unsigned indexIn, unsigned indexOut, float peso, short unsigned recurrente, short unsigned enabled, TConfig* conf) //OPTIMIZADA TODO: Va en gen.c no aqu�.
{
    /******************************************/
    /*       funcion  nuevaConex()             */
    /******************************************/
//Adiciona o cobreescribe unaonexi�n a un genoma de la conf->poblaci�n (param = indice del genoma en la conf->pob, todas
//las variables de struct GenConexF excepto innovNum que es una variable global para cada gen.).
//adem�s, incrementa el contador de hijos del nodo indexOut
//Par�metros: indexpob, IndexIn, IndexOut,recurrente, peso, enabled.
//Retorna 0 si hubo error, 1 si ok.
////TODO MODIFICAR LA FUNCION EVALUAR GENOMA PARA EVALUAR POR INNOVNUM DE NODOS y CONEXIONES en lugar de indexes.
////TODO: Verificar si genoma adicionado ya existe en el genoma, si existe, le adiciona su peso. y tiene 50% de chance de ser enabled si
//			una de ellas (existente o nueva)es disabled(manejar conCont del nodo destino).
    //obtener memoria con realloc en conf->pob[indexpob].conex para un tama�o (totalConexiones+1)sizeof(GenNodoF)
    Genoma* pGenoma = &(conf->pob[indexpob]); //usado para acelerar operaciones en el genoma
    GenConexF* pConex; //puntero a la nueva conex creada, usado para acelerar operaciones en el nodo
    unsigned innovIn = pGenoma->nodo[indexIn].innovNum;
    unsigned innovOut = pGenoma->nodo[indexOut].innovNum;
    unsigned i;
    // verifica si la conex ya existe.
    for (i=0;i<conf->pob[indexpob].totalConexiones;i++)
    {
        if ((conf->pob[indexpob].conex[i].indexIn==indexIn)&&(conf->pob[indexpob].conex[i].indexOut==indexOut))
        {
            // si existe, sobreescribe los valores y retorna OK.
            conf->pob[indexpob].conex[i].peso=peso;
            conf->pob[indexpob].conex[i].enabled=enabled;
            conf->pob[indexpob].conex[i].recurrente = ((indexIn==indexOut)? 1: 0);
            return(1);
        }
    }
    // ubica memoria para la nueva conexi�n
    if ((pGenoma->conex=(GenConexF *)realloc(pGenoma->conex, sizeof(GenConexF)*(pGenoma->totalConexiones+1)))==NULL)
    {
        fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 11 en funcion nuevaConex(%u,%u,%u,%1.1f,%u,%u) llamando a realloc(conf->pob[%u].conex,%u)\n",indexpob,indexIn, indexOut, peso, recurrente, enabled,indexpob,(pGenoma->totalConexiones+1)*(unsigned int)sizeof(GenConexF));
        return(0);
    }
    pConex=&(pGenoma->conex[pGenoma->totalConexiones]);
    //Inicializa los valores de la nueva conexi�n
    if ((pConex->innovNum=nuevaInnovCon(innovIn,innovOut,conf))==UINT_MAX)
    {
        fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 12 en funcion nuevaConex(%u,%u,%u,%1.1f,%u,%u) llamando a nuevaInnovCon(%u,%u)\n",indexpob, indexIn, indexOut, peso,recurrente,enabled,innovIn,innovOut);
        return(0);
    }
    //La sig. linea mantiene el innovNum m�ximo para el genoma indexPop
    if (pGenoma->maxInnovNumConex<pConex->innovNum) pGenoma->maxInnovNumConex=pConex->innovNum;
    pConex->nodoIn=innovIn;
    pConex->nodoOut=innovOut;
    pConex->indexOut=indexOut;
    pConex->indexIn=indexIn;
    pConex->peso=peso;
    pConex->recurrente= ((indexIn==indexOut)? 1: 0);
    pConex->enabled=enabled;
    if (enabled==1)
    {
        calcularLimThOneNode(indexpob,indexOut,conf);
    }
    pGenoma->nodo[indexOut].contHijos++;
    //Incrementa el total de nconexiones  en pGenoma->totalConexiones++;
    pGenoma->totalConexiones++;
    return(1);
}

unsigned nuevoNodo(unsigned indexpob, unsigned indexElimConex, TConfig* conf)  //OPTIMIZADA TODO: Va en gen.c no aqu�.
{
// Adiciona un nuevo nodo al genoma unsigned  interrumpiendo la conexi�n indicada y conservando el peso original en la conex nuevo a out y
// colocando peso=1 para la conex in a nuevo y coloca en disabled la conexi�n original.
// Par�metros : 	indexpob = indice de la pob para el genoma al que se adicionar� el nodo
// 					indexElimInnovConex = index del nodo en el arreglo de nodos.
// Retorna 0 si hubo alg�n error, 1 si ok.
// Coloca en disabled la conexi�n con n�mero de innovaci�n indexElimConex
    Genoma* pGenoma = &(conf->pob[indexpob]); //usado para acelerar operaciones en el genoma
    GenNodoF* pNodo; //puntero al nuevo nodo creado, usado para acelerar operaciones en el nodo
    pGenoma->conex[indexElimConex].enabled=0;
    // obtener memoria con realloc en pGenoma->nodo para un tama�o (totalNodos+1)sizeof(GenNodoF)
    if ((pGenoma->nodo=(GenNodoF *)realloc(pGenoma->nodo, sizeof(GenNodoF)*(pGenoma->totalNodos+1)))==NULL)
    {
        fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 15 en fupGenoma->nodo[pGenoma->totalNodos].minTh=0;ncion nuevoNodo(%u,%u) llamando a realloc(pGenoma->nodo,%u).",indexpob,indexElimConex,(pGenoma->totalNodos+1)*(unsigned int)sizeof(GenNodoF));
        return(0);
    }
    //Asigna pNodo al nodo reci�n ubicado en memoria para acelerar calculos.
    pNodo = &(pGenoma->nodo[pGenoma->totalNodos]);
    // inicializa los valores del nuevo nodo. obteniendo el n�mero de innovaci�n para New.
    if ((pNodo->innovNum=nuevaInnovNodo(pGenoma->conex[indexElimConex].nodoIn,pGenoma->conex[indexElimConex].nodoOut,conf))==UINT_MAX)
    {
        fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 16 en funcion nuevoNodo(%u,%u) en funcion nuevaInnovNodo(%u,%u)",indexpob,indexElimConex,pGenoma->conex[indexElimConex].nodoIn,pGenoma->conex[indexElimConex].nodoOut);
        return(0);
    }
    pNodo->nodeFunction=1; //1=oculto
    pNodo->estadoC=0;
    pNodo->valor=0.5;
    pNodo->maxTh=1;
    pNodo->minTh=0;
    pNodo->contHijos=0;
    pNodo->conexHijo=NULL;
    // TODO: que valor de threshold deber�a tener el nuevo nodo?. Por ahora se coloca en el medio del l�mite de maxTh y minTh.
    pNodo->thNodo=conf->Fthreshold;
    if (conf->tSigma>1000)  //si tsigma es -1,1 //TODO: verificar sies necesario esto si se usa mutar th,
    {
        pNodo->minTh=-1.0;
        pNodo->valor=0;
    }
    // mantiene en para el genoma el m�ximo n�mero de innovaci�n de nodo.
    if (pGenoma->maxInnovNumNodo<pNodo->innovNum)
    {
        pGenoma->maxInnovNumNodo=pNodo->innovNum;
    }
    // inicializa los valores de la nueva conexi�n New->Out co peso  igual a la conexi�n eliminada y con el valor de enabled de la anterior?????//TODO
    if(nuevaConex(indexpob,pGenoma->totalNodos, pGenoma->conex[indexElimConex].indexOut, pGenoma->conex[indexElimConex].peso  , 0, 1,conf)==0)
    {
        fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 17 en funcion nuevoNodo(%u,%u) en funcion nuevaConex(%u,%u,%u,%1.1f,%u,%u)",indexpob,indexElimConex,indexpob,pGenoma->totalNodos, pGenoma->conex[indexElimConex].indexOut, pGenoma->conex[indexElimConex].peso  , 0, 1);
        return(0);
    }
    // inicializa los valores de la nueva conexi�n In->New con peso 1
    if(nuevaConex(indexpob,pGenoma->conex[indexElimConex].indexIn,pGenoma->totalNodos,1, 0, 1,conf)==0)
    {
        fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 18 en funcion nuevoNodo(%u,%u) llamando a funcion nuevaConex(%u,%u,%u,%u,%u,%u)",indexpob,indexElimConex,indexpob,pGenoma->conex[indexElimConex].indexIn,pGenoma->totalNodos,1 , 0, 1);
        return(0);
    }
    pGenoma->totalNodos++;
    return(1);
}

unsigned genomaInicial(unsigned index,unsigned nEntradas, unsigned nSalidas, unsigned nBias, unsigned primero, unsigned especie,TConfig* conf)  // OTIMIZADA TODO: optimizar con punteros
{
//Crea un nuevo genoma totalmente conectado asigna la especie 0 y lo ubica en el �ndice index de la poblaci�n.
//par�metros: index=donde queda el genoma inicial, nEntradas, nSalidas, nBias.
//			primero si =1 se bora lista de innovaciones , representantes, conservacion, generaciones sin mejora, etc...
//retorna 0 si hubo error, 1 si la creaci�n fu� exitosa.
    unsigned i=0;
    unsigned j=0;
    unsigned k=0;
    ////TODO
    //Crear todos los nodos de entrada y salida (a mano).
    //tenemos puntero a estructura Genoma, pero tenemos variables de genoma sin inicializar.
    //Se hace calloc para el arreglo de nodos de tama�o bias+entradas+salidassizeof(nodo)
    if (conf->pob[index].nodo!=NULL) free((void *)conf->pob[index].nodo);
    if (!(conf->pob[index].nodo=( GenNodoF *) calloc(1,sizeof( GenNodoF)*(nEntradas+nSalidas+nBias))))
    {
        fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 20 en funcion genomaInicial(%u,%u,%u) llamando a calloc(1,%u).\n",nEntradas,nSalidas,nBias,(nEntradas+nSalidas+nBias)*(unsigned int)sizeof( GenNodoF));
        return(0);
    }
    //Para cada  nodo de salida, crea una conexi�n a cada nodo de entrada y bias con peso 1.
    //Obtiene puntero para conexiones con tama�o nSalidas*(nentradas+nBias)sizeof(GenConexF)
    if (conf->pob[index].conex!=NULL) free((void *)conf->pob[index].conex);
    if ((conf->pob[index].conex=( GenConexF *) calloc(1,sizeof( GenConexF)*nSalidas*(nEntradas+nBias)))==NULL)
    {
        fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 21 en funcion genomaInicial(%u,%u,%u) llamando a calloc(1,%u).\n",nEntradas,nSalidas,nBias,nSalidas*(nEntradas+nBias)*(unsigned int)sizeof( GenConexF));
        return(0);
    }
    if (primero==1)
    {
        //Se reserva memoria para el arreglo de innovacione para nodos.
        if (!(conf->listaInnovNodo=( TListaInnov *) malloc(sizeof( TListaInnov)*(nEntradas+nSalidas+nBias))))
        {
            fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 22 en funcion genomaInicial(%u,%u,%u) llamando a malloc(%u).\n",nEntradas,nSalidas,nBias,(nEntradas+nSalidas+nBias)*(unsigned int)sizeof( TListaInnov));
            return(0);
        }
        fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>PrimerGenoma\n");
        //inicializa los punteros de arreglos de nodos de innovs :
        for(i=0; i<(nEntradas+nSalidas+nBias); i++)
        {
            conf->listaInnovNodo[i].numOut=1;
            if ((conf->listaInnovNodo[i].nodoOut=(TNodoOut*)malloc((unsigned  int)sizeof(TNodoOut)))==NULL)
            {
                fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 22.1 en funcion genomaInicial(%u,%u,%u) llamando a malloc(%u).\n",nEntradas,nSalidas,nBias,(unsigned  int)sizeof(TNodoOut));
                return(0);
            }
            conf->listaInnovNodo[i].nodoOut[0].innovNum=i;
            conf->listaInnovNodo[i].nodoOut[0].nodoOut=i;
        }
        conf->contInnovNodo=(nEntradas+nSalidas+nBias);
        //Se reserva memoria para el arreglo de innovacione para conexiones.
        if (!(conf->listaInnovCon=( TListaInnov *) malloc(sizeof(TListaInnov)*(nEntradas+nSalidas+nBias))))
        {
            fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 23 en funcion genomaInicial(%u,%u,%u) llamando a calloc(1,%u).\n",nEntradas,nSalidas,nBias,nSalidas*(nEntradas+nBias)*(unsigned int)sizeof(TListaInnov));
            return(0);
        }
        //inicializa los punteros de arreglos de conexiones de innovs:
        k=0;
        for (i=0; i<(nEntradas+nBias); i++)
        {
            conf->listaInnovCon[i].numOut=nSalidas;
            if((conf->listaInnovCon[i].nodoOut=(TNodoOut*) malloc(sizeof(TNodoOut)*nSalidas))==NULL)  //porque es totalmente conectado
            {
                fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 23.1 en funcion genomaInicial(%u,%u,%u) llamando a malloc(%u).\n",nEntradas,nSalidas,nBias,(unsigned  int)sizeof(TNodoOut));
                return(0);
            }
            for (j=0; j<nSalidas; j++)
            {
                conf->listaInnovCon[i].nodoOut[j].nodoOut=(nEntradas+nBias)+j;
                conf->listaInnovCon[i].nodoOut[j].innovNum=k;
                k++;
            }
        }
        //completa los valores de los dem�s elementos de entrada de listaInnovCon
        for (i=(nEntradas+nBias); i<(nEntradas+nBias+nSalidas); i++)
        {
            conf->listaInnovCon[i].numOut=0;
            conf->listaInnovCon[i].nodoOut=NULL;
        }
        conf->contInnovCon=nSalidas*(nEntradas+nBias);
// IMPRIME LA LISTA DE INNOVACIONES DE NODO Y CONEX
//        imprimirListasInnov(conf);
        //reserva memoria para el arreglo de conf->representantes de tama�o 1 (trivial)
        if (conf->representantes!=NULL) free((void *)conf->representantes);
        if  ((conf->representantes=(unsigned *) calloc(1,(unsigned  int)sizeof(unsigned  int)))==NULL)
        {
            fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 24 en funcion genomaInicial(%u,%u,%u) llamando a calloc(1,%u).\n",nEntradas,nSalidas,nBias,(unsigned  int)sizeof(unsigned  int));
            return(0);
        }
        //reserva memoria para el arreglo de especies en conservaci�n de tema�o conf->maxEspeciesConservacion
        if (conf->conservacionEsp!=NULL) free((void *)conf->conservacionEsp);
        if  ((conf->conservacionEsp=(unsigned *) calloc(conf->spEspecies,(unsigned  int)sizeof(unsigned  int)))==NULL)
        {
            fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 24.4 en funcion genomaInicial(%u,%u,%u) llamando a calloc(maxEspeciesConservaci�n,%u).\n",nEntradas,nSalidas,nBias,(unsigned  int)sizeof(unsigned  int));
            return(0);
        }
        //reserva memoria para el arreglo de numero de genomas por especie, tama�o=conf->numEspecies
        if (conf->conservacionEsp!=NULL) free((void *)conf->conservacionEsp);
        if  ((conf->numGenomasPorEspecie=(unsigned *) calloc(conf->spEspecies,(unsigned  int)sizeof(unsigned  int)))==NULL)
        {
            fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 24.4 en funcion genomaInicial(%u,%u,%u) llamando a calloc(maxEspeciesConservaci�n,%u).\n",nEntradas,nSalidas,nBias,(unsigned  int)sizeof(unsigned  int));
            return(0);
        }
        //reserva memoria para conf->contGeneracSinMejora
        if (conf->contGeneracSinMejora!=NULL) free((void *)conf->contGeneracSinMejora);
        if  ((conf->contGeneracSinMejora=(unsigned *) calloc(1,(unsigned  int)sizeof(unsigned  int)))==NULL)
        {
            fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 24.5 en funcion genomaInicial(%u,%u,%u) llamando a calloc(1,%u).\n",nEntradas,nSalidas,nBias,(unsigned  int)sizeof(unsigned  int));
            return(0);
        }
    }
    //Se inicializa la funcion y el valor en los correspondientes valores iniciales
    //para entradas:
    for (i=0; i<nEntradas; i++)
    {
        conf->pob[index].nodo[i].nodeFunction=0; //0=entrada,1=oculto,2=salida,3=bias.
        conf->pob[index].nodo[i].valor=0; //Valor de salida de cada nodo(para computar la ann sin matriz de pesos).
        conf->pob[index].nodo[i].estadoC=1; // no se calcula, se tiene de las entradas.
        conf->pob[index].nodo[i].maxTh=1; //TODO: calcular rango de thresholds con prean�lisis
        conf->pob[index].nodo[i].minTh=0;
        conf->pob[index].nodo[i].contHijos=0;
        conf->pob[index].nodo[i].conexHijo=NULL;
        // TODO: que valor de threshold deber�a tener el nuevo nodo?. Por ahora se coloca en el medio del l�mite de maxTh y minTh.
        conf->pob[index].nodo[i].thNodo=conf->Fthreshold;
        if (conf->tSigma>1000)  //si tsigma es -1,1
        {
            conf->pob[index].nodo[i].minTh=-1.0;
            conf->pob[index].nodo[i].valor=0;
        }
        //coloca valores de listainnov para los nodos
        if ((conf->pob[index].nodo[i].innovNum=nuevaInnovNodo(i,i,conf))==UINT_MAX)
        {
            fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 25.1 en funcion genomaInicial(%u,%u,%u) llamando a nuevaInnovNodo(%u,%u)\n",nEntradas,nSalidas,nBias,i,i);
            return(0);
        }
    }
    //para bias:
    for (i=nEntradas; i<(nEntradas+nBias); i++)
    {
        conf->pob[index].nodo[i].nodeFunction=3; //0=entrada,1=oculto,2=salida,3=bias.
        conf->pob[index].nodo[i].valor=1; //Ya que es bias, se inicializa en 1
        conf->pob[index].nodo[i].estadoC=1; //nunca se calcula
        conf->pob[index].nodo[i].maxTh=1; //TODO: calcular rango de thresholds con prean�lisis
        conf->pob[index].nodo[i].minTh=0;
        conf->pob[index].nodo[i].contHijos=0;
        conf->pob[index].nodo[i].conexHijo=NULL;
        // TODO: que valor de threshold deber�a tener el nuevo nodo?. Por ahora se coloca en el medio del l�mite de maxTh y minTh.
        conf->pob[index].nodo[i].thNodo=conf->Fthreshold;
        if (conf->tSigma>1000)  //si tsigma es -1,1
        {
            conf->pob[index].nodo[i].minTh=-1.0;
        }
        if ((conf->pob[index].nodo[i].innovNum=nuevaInnovNodo(i,i,conf))==UINT_MAX)
        {
            fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 25.2 en funcion genomaInicial(%u,%u,%u) llamando a nuevaInnovNodo(%u,%u)\n",nEntradas,nSalidas,nBias,i,i);
            return(0);
        }
    }
    //para salidas:
    for (i=(nEntradas+nBias); i<(nEntradas+nBias+nSalidas); i++)
    {
        conf->pob[index].nodo[i].nodeFunction=2; //0=entrada,1=oculto,2=salida,3=bias.
        conf->pob[index].nodo[i].valor=0; //Valor de salida de cada nodo(para computar la ann sin matriz de pesos).
        conf->pob[index].nodo[i].estadoC=0;
        conf->pob[index].nodo[i].maxTh=1; //TODO: calcular rango de thresholds con prean�lisis
        conf->pob[index].nodo[i].minTh=0;
        conf->pob[index].nodo[i].contHijos=nEntradas+nBias;
        conf->pob[index].nodo[i].conexHijo=NULL;
        // TODO: que valor de threshold deber�a tener el nuevo nodo?. Por ahora se coloca en el medio del l�mite de maxTh y minTh.
        conf->pob[index].nodo[i].thNodo=conf->Fthreshold;
        if (conf->tSigma>1000)  //si tsigma es -1,1
        {
            conf->pob[index].nodo[i].minTh=-1.0;
            conf->pob[index].nodo[i].valor=0;
        }

        if ((conf->pob[index].nodo[i].innovNum=nuevaInnovNodo(i,i,conf))==UINT_MAX)
        {
            fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 25.3 en funcion genomaInicial(%u,%u,%u) llamando a nuevaInnovNodo(%u,%u)\n",nEntradas,nSalidas,nBias,i,i);
            return(0);
        }
    }
    //Se inicializa en los valores necesarios.
    k=0;
    for (j=(nEntradas+nBias); j<(nEntradas+nBias+nSalidas); j++)
    {
        for (i=0; i<(nEntradas+nBias); i++)
        {
            conf->pob[index].conex[k].nodoIn = i;
            conf->pob[index].conex[k].nodoOut = j;
            conf->pob[index].conex[k].indexIn = i;
            conf->pob[index].conex[k].indexOut = j;
            conf->pob[index].conex[k].peso = 1;
            conf->pob[index].conex[k].recurrente = 0;
            conf->pob[index].conex[k].enabled = 1;
            ////TODO: funcion para obtener autom�ticamente el n�mero de innovaci�n correspondiente a la conexi�n
            //basado en la lista de innovaciones y en los nodos de entrada y salida yagregarlo a la lista sino existe.
            //�es posible hacer otra lista de innovaciones de nodos que lleven en su estructura
            //el innovConNum de entradas y salidas? podr�a esto verificarse al momento de hacer el cruce �nicamente?
            //�Puede codificarse en el nombre del nodo, el n�mero de conexiones adyacientes?

            //Al aparecer un nuevo nodo su n�mero(nombre) de innovaci�n puede obtenerse  si se compara con
            //una lista donde se lleva para cada nuevo nodo en la generaci�n actual
            //el nodo de origen y destino de la conexi�n que se unsigned  interrumpi� para crearlo.
            //y ese n�mero se usa como identificador del nodo en la estructura GenConexF y en la lista de innovaciones.
            ////TODO:Crear estructuras y modificaciones para usar esto.
            //Para esta primera generaci�n, como los nodos son conocidos, su n�mero de innovaci�n es el mismo n�mero de nodo.
            //y por tanto,:
            if ((conf->pob[index].conex[k].innovNum = nuevaInnovCon(i,j,conf))==UINT_MAX)
            {
                fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 26 en funcion genomaInicial(%u,%u,%u) en funcion nuevaInnovCon(%u,%u) \n",nEntradas,nSalidas,nBias,i,j);
                return(0);
            }
            //Coloca el n�mero de innovaci�n correspondiente a cada conexi�n
            k++;
        }
    }
    conf->pob[index].maxInnovNumConex=(nSalidas*(nEntradas+nBias))-1;
    conf->pob[index].maxInnovNumNodo=nSalidas+nEntradas+nBias-1;
    //Inicializa los valores restantes del genoma Inicial
    conf->pob[index].especie = especie; //establece la especie inicial.

    if (primero==1)
    {
        conf->numEspecies=1; //adiciona una especie especie (especie 0)
        conf->contGeneracSinMejora[especie]=0;//coloca 0 generaciones sin mejor aen fitness para la especie 0
        //reserva memoria para el nuevo arreglo de especies con tama�o conf->numEspeciessizeof(unsigned  int)
        //reserva memoria para el nuevo arreglo de conf->contGeneracSinMejora
        //coloca al indexpob como representante de la nueva especie
        conf->representantes[conf->numEspecies-1]=index;
        //coloca en 0 el n�mero de generaciones sin mejora para la nueva especie
        conf->contGeneracSinMejora[conf->numEspecies-1]=0;

    }
    conf->pob[index].totalNodos = nEntradas+nSalidas+nBias;
    conf->pob[index].totalConexiones = nSalidas*(nEntradas+nBias);
    conf->pob[index].numHijos=-1;
    conf->pob[index].fitness = 0; // error inicial=100% tambi�n inicializa el contador de fitness
    calcularLimThOneGenome(index,conf);
    return(1);
}

void genomaMasLejano(int indexPob,int intentos, TConfig* conf)
// busca el genoma m�s lejano a todos los de su especie entre la poblaci�n  <intentos> veces y lo coloca en indexPob
// tambi�n debe tener distancia menor a 1/2 de distacia a especie m�s cercana.
{
    int i,j;
    int miEspecie=conf->pob[indexPob].especie;
    unsigned offs=(2*conf->sizePob)+conf->spEspecies;
    float distEspCercana=999999; //si hay una sola especie, se usa este valor
    float distancia[1000];
    float minDist=9999999;
    float increm=(1-conf->porcentMutPeso)/intentos;
    float tmp;
    unsigned foundIndex=0;
    // calcula distancia a la especie m�s cercana
    distEspCercana=distEspecieCercana(indexPob,conf->c1, conf->c2, conf->c3, conf->eG_t,conf)/2;
    for (i=0;i<intentos;i++) //inicializa todas las m�nimas distancias en 0: necesario al final de esta func
    {
        distancia[i]=0;
    }
    // copiarGenoma representante[miEspecie] a indexpob (SE DEBE HACER?)
    for (i=0;i<intentos;i++)
    {
        // copia representante[miEspecie] a los n intentos
        copiarGenoma(conf->representantes[miEspecie],offs+i,conf);
        // aplica perturbaci�n cada vez mayor de pesos empezando en conf->porcentMutPeso terminando terminando en 1
        perturbarPeso(offs+i,increm*(float)i,conf->probMutPeso,0,0,0,0, conf);
        //randomizarPesos(offs+i,conf->pesoMinInicial,conf->pesoMaxInicial,conf);
        //inicializa minDist
        minDist=999999;
        // para cada intento para cada genoma de la pob:
        for (j=0;j<conf->sizePob;j++)
        {
            //solo si es de la misma especie que indexpob
            if (conf->pob[j].especie==miEspecie)
            {
                if (j!=indexPob)
                {
                    // medir distancia a cada uno de los de mi especie exc yo y guardar m�nimo en distancia[i]
                    tmp=calcularDist(offs+i, j, conf->c1, conf->c2, conf->c3, conf->eG_t,conf);
                    if ((tmp<minDist)&&(tmp>0))
                    {
                        minDist=tmp;
                    }
                }
            }
        }
        //guarda la m�nima distancia encontrada en la especie en distancia[i]
        distancia[i]=minDist;

    }

    tmp=0;
   // copiar m�ximo entre los temp[] a indexPob
    for(i=0;i<intentos;i++)
    {
        //si la distancia es m�xima y menor a un medio de la distancia entre reps m�s cercanos.
        if ((distancia[i]>tmp)&&(calcularDist(offs+i, conf->representantes[miEspecie], conf->c1, conf->c2, conf->c3, conf->eG_t,conf)<distEspCercana)){
            tmp=distancia[i];
            foundIndex=i;
        }
    }
    copiarGenoma(offs+foundIndex,indexPob,conf);
}

//No quiero perderme ni un fot�n de tus refl�jos. :] Juli.



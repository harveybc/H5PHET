#include "hip/hip_runtime.h"
/** Funciones para menejo de especies - C file
	usan/modifican genomas �nicamente (pueden usarse operaciones de genes de gen.h en un genoma).
*/

#ifndef PARAMS_H_INCLUDED
#include "params.h"
#define PARAMS_H_INCLUDED
#endif
#ifndef AUXILIARES_H_INCLUDED
#include "auxiliares.h"
#define AUXILIARES_H_INCLUDED
#endif
#ifndef GENOMA_H_INCLUDED
#include "genoma.h"
#define GENOMA_H_INCLUDED
#endif
#include "especie.h"


void imprimirListasInnov(TConfig* conf)
{
    unsigned i,j;
    //Imprime innovaciones de nodo formato: In
    fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nImprime la lista de innovaciones de nodos y conex\nFormato: **i<in>(<contInnov>),o<out>=(<innovnum>,<nodoout>)... ");
    fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nLista de innov Nodos: contInnovNodo=%u<br>",conf->contInnovNodo);
    for (i=0; i<conf->contInnovNodo; i++)
    {
        fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"**i%u(%u),",i,conf->listaInnovNodo[i].numOut);
        for (j=0; j<conf->listaInnovNodo[i].numOut; j++)
            fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"o%u=(%u,%u) ",j,conf->listaInnovNodo[i].nodoOut[j].innovNum,conf->listaInnovNodo[i].nodoOut[j].nodoOut);
    }
    fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nLista de innov Conexiones: contInnovConex=%u<br>",conf->contInnovCon);
    for (i=0; i<conf->contInnovNodo; i++)
    {
        fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"**i%u(%u),",i,conf->listaInnovCon[i].numOut);
        for (j=0; j<conf->listaInnovCon[i].numOut; j++){
            fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"o%u=(%u,%u) ",j,conf->listaInnovCon[i].nodoOut[j].innovNum,conf->listaInnovCon[i].nodoOut[j].nodoOut);
        }
    }

}

unsigned verificarListasInnov(TConfig* conf)
{
    //retorna 0 si hay valores inv�lidos en la lista de innovaci�n y imprime el maxInnovNum
    //TODO: quitar cuando no haya errores de innovnums
    unsigned i,j;
//    fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>miN=%i,miC=%i,",conf->contInnovNodo,conf->contInnovCon);
    for (i=0; i<conf->contInnovNodo; i++)  //para la lista de innovaciones de nodos
    {
        for (j=0; j<conf->listaInnovNodo[i].numOut; j++)  //busca l�mites inferiores para innovnum y nodoOut para cada innovaci�n
        {
            if (conf->listaInnovNodo[i].nodoOut[j].innovNum>=conf->contInnovNodo)  //verifica maximo innovnumNodo
            {
                fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 51 en verificarListasInnov(), liNodo[%u].no[%u]=%u > ciN=%u",i,j,conf->listaInnovNodo[i].nodoOut[j].innovNum,conf->contInnovNodo);
                return(0);
            }
            if (conf->listaInnovNodo[i].nodoOut[j].innovNum<0)  //verifica que el innovnumnodo sea >=0
            {
                fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 52 en verificarListasInnov(), liNodo[%u].no[%u].iN=%u < 0",i,j,conf->listaInnovNodo[i].nodoOut[j].innovNum);
                return(0);
            }
            if (conf->listaInnovNodo[i].nodoOut[j].nodoOut>=conf->contInnovNodo)  //verifica que el innovnumnodo sea >=0
            {
                fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 53 en verificarListasInnov(), liNodo[%u].no[%u].nO=%u > ciN=%u",i,j,conf->listaInnovNodo[i].nodoOut[j].nodoOut,conf->contInnovNodo);
                return(0);
            }
            if (conf->listaInnovNodo[i].nodoOut[j].nodoOut<0)  //verifica que el innovnumnodo sea >=0
            {
                fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 54 en verificarListasInnov(), liNodo[%u].no[%u].nO=%u < 0",i,j,conf->listaInnovNodo[i].nodoOut[j].nodoOut);
                return(0);
            }
        }
    }
    for (i=0; i<conf->contInnovNodo; i++)  //para la lista de innovaciones de conex
    {
        for (j=0; j<conf->listaInnovCon[i].numOut; j++)  //busca l�mites inferiores para innovnum y nodoOut para cada innovaci�n
        {
            if (conf->listaInnovCon[i].nodoOut[j].innovNum>=conf->contInnovCon)  //verifica maximo innovnumNodo
            {
                fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 55 en verificarListasInnov(), liCon[%u].no[%u]=%u > ciC=%u",i,j,conf->listaInnovCon[i].nodoOut[j].innovNum,conf->contInnovCon);
                return(0);
            }
            if (conf->listaInnovCon[i].nodoOut[j].innovNum<0)  //verifica que el innovnumnodo sea >=0
            {
                fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 56 en verificarListasInnov(), liCon[%u].no[%u].iN=%u < 0",i,j,conf->listaInnovCon[i].nodoOut[j].innovNum);
                return(0);
            }
            if (conf->listaInnovCon[i].nodoOut[j].nodoOut>=conf->contInnovNodo)  //verifica que el innovnumnodo sea >=0
            {
                fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 57 en verificarListasInnov(), liCon[%u].no[%u].nO=%u > ciN=%u",i,j,conf->listaInnovCon[i].nodoOut[j].nodoOut,conf->contInnovNodo);
                return(0);
            }
            if (conf->listaInnovCon[i].nodoOut[j].nodoOut<0)  //verifica que el innovnumnodo sea >=0
            {
                fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 58 en verificarListasInnov(), liCon[%u].no[%u].nO=%u < 0",i,j,conf->listaInnovCon[i].nodoOut[j].nodoOut);
                return(0);
            }
        }
    }
    return(1);
}

float calcularDist(unsigned indexpob1,unsigned indexpob2,float c1, float c2, float c3,unsigned eG_t, TConfig* conf)  //OPTIMIZADA
{
//Retorna la distancia entre dos genomas. uusando la formula D=((((C1)*E)/n)+(((C2)*D)/n)+C3*W)
//			Donde: 	C1,C2,C3 = constantes de proporcionalidad (mirar ejemplos para valores iniciales.)
//					n = N�mero de genes en el genoma m�s grande.
//					E = N�mero de genes de exceso
//					D = N�mero de genes disjounsigned  int
//					W = Promedio de diferencia de pesos entre genes correspondientes. Puede ser seteado a 1 para valores que no sean excesivamente grandes.
//Par�metros: 	indexpob1 , indexpob2, = index de los genomas en la conf->pob.
//				c1 = constante de proporcionalidad para Excess genes
//				c2 = constante de proporcionalidad para Disjounsigned genes
//				c3 = constante de proporcionalidad para el promedio de diferencias de pesos.
//				eG_t = threshold para considerar n�mero de genes "excesivamente" grandes (y hacer n=1)
    ////TODO: Quitar inicializaciones innecesarias en todas las funciones.
    unsigned i=0;
    float disjointC=0;
    float excessC=0;
    float wAver=0;
    unsigned tmpInnov1=0;
    unsigned tmpInnov2=0;
    unsigned indexMenorInnovNumConex=indexpob1;
    unsigned indexMayorInnovNumConex=indexpob2;
    unsigned cont=0;
    unsigned n=0;
    unsigned tmpMaxInnovNumConex; //usado para acelerar calculos
    //determinar el mayor y el menor (maxInnovNum)
    if (conf->pob[indexMenorInnovNumConex].maxInnovNumConex>conf->pob[indexMayorInnovNumConex].maxInnovNumConex)
    {
        indexMayorInnovNumConex=indexpob1;
        indexMenorInnovNumConex=indexpob2;
    }
    // excess = buscar todos los innovNums (desde el menor conf->maxInnovNumConex hasta el mayor) en el genoma que tiene el mayor.
    if (conf->pob[indexMenorInnovNumConex].maxInnovNumConex<conf->pob[indexMayorInnovNumConex].maxInnovNumConex)
    {
        tmpMaxInnovNumConex=conf->pob[indexMayorInnovNumConex].maxInnovNumConex; // para acelerar evaluaci�n en for.
        for (i=conf->pob[indexMenorInnovNumConex].maxInnovNumConex+1; i<=conf->pob[indexMayorInnovNumConex].maxInnovNumConex; i++)
            if (buscarInnovConex(indexMayorInnovNumConex,i,conf)!=UINT_MAX) excessC++;
    }
    // disjounsigned  int= contar disjounsigned  ints hasta el menor maxInnovNum entre los dos genes de conexiones
    tmpMaxInnovNumConex=conf->pob[indexMenorInnovNumConex].maxInnovNumConex; // para acelerar evaluaci�n en for.
    for (i=0; i<=tmpMaxInnovNumConex; i++) //MAL
    {
        tmpInnov1=buscarInnovConex(indexpob1,i,conf);
        tmpInnov2=buscarInnovConex(indexpob2,i,conf);
        if(tmpInnov1 == UINT_MAX)
            if (tmpInnov2 != UINT_MAX ) disjointC++;
        if(tmpInnov2 == UINT_MAX)
            if (tmpInnov1 != UINT_MAX ) disjointC++;
        if(tmpInnov1!=UINT_MAX) //obtiene promedio de diferencias
            if (tmpInnov2!=UINT_MAX)
            {
                cont++;
                wAver=wAver+fabs(conf->pob[indexpob1].conex[tmpInnov1].peso-conf->pob[indexpob2].conex[tmpInnov2].peso);
            }
    }
    // TODO: faltan optimizar todos los for del programa para que eval�en una variable local en lugar de un valor referenciado.
    // calcula wAver
    if (cont>0)
        //wAver=floor(wAver/(float)cont); CORREGIDO
        wAver=wAver/(float)cont;
    // calcula n=mayor n�mero de genes entre los dos genomas
    if (conf->pob[indexpob1].totalConexiones>conf->pob[indexpob2].totalConexiones) n=conf->pob[indexpob1].totalConexiones;
    else n=conf->pob[indexpob2].totalConexiones;
    // ((((C1)*E)/n)+(((C2)*D)/n)+C3*W)
    if(n>0) return(((((c1)*excessC)+((c2)*disjointC))/(float)n)+c3*wAver);
    // si error o los genomas son iguales retorna 0;
    return(0);
}

unsigned especieMinDist(unsigned indexpob,float c1, float c2, float c3,unsigned eG_t, TConfig* conf)  //OPTIMIZADA
{
// Retorna la especie del genoma al que se tiene la m�nima distancia entre los conf->representantes
// Par�metros: indexpob = indice del genoma para el cual se busca la mindist.
//				c1 = constante de proporcionalidad para Excess genes
//				c2 = constante de proporcionalidad para Disjounsigned genes
//				c3 = constante de proporcionalidad para el promedio de diferencias de pesos.
//				eG_t = threshold para considerar n�mero de genes "excesivamente" grandes (y hacer n=1)
    unsigned i=0;
    float minDist=99999999;
    float tmpDist=0;
    unsigned foundIndex=0;
    for (i=0; i<conf->numEspecies; i++)
    {
        tmpDist=calcularDist(indexpob,conf->representantes[i],c1,c2,c3,eG_t,conf);
        if (tmpDist<minDist)
        {
            minDist=tmpDist;
            foundIndex=i;
        }
    }
    return (foundIndex);
}

float distEspecieCercana(unsigned indexpob,float c1, float c2, float c3,unsigned eG_t, TConfig* conf)  //OPTIMIZADA
{
//retorna la distancia a la especie m�s cercana diferente a la del indexpob
//Par�metros: indexpob = indice del genoma para el cual se busca la mindist.
//				c1 = constante de proporcionalidad para Excess genes
//				c2 = constante de proporcionalidad para Disjounsigned genes
//				c3 = constante de proporcionalidad para el promedio de diferencias de pesos.
//				eG_t = threshold para considerar n�mero de genes "excesivamente" grandes (y hacer n=1)
    unsigned i=0;
    float minDist=99999999; // retorna este valor si hay una sola especie
    float tmpDist=0;
    for (i=0; i<conf->numEspecies; i++)
    {
        if (conf->pob[indexpob].especie!=i){
            tmpDist=calcularDist(conf->representantes[conf->pob[indexpob].especie],conf->representantes[i],c1,c2,c3,eG_t,conf);
            if (tmpDist<minDist)
            {
                minDist=tmpDist;
            }
        }
    }
    return (minDist);
}

unsigned asignarEspecie(unsigned indexpob, float espThreshold,float c1, float c2, float c3,unsigned eG_t, TConfig* conf)  //OPTIMIZADA
{
// Usando especieMinDist obtiene la especie m�s compatible, y la compara su distancia con el threeshold, si es menor asigna la especie, si es mayor, crea
// una nueva y la asigna al genoma en cuesti�n
// retorna el n�mero de la especie asignada., -1 si hay error
// par�metros:	indexpob	= indice del genoma al que se asignar� la especie
//				threeshold	= l�mite(inferior) de distancia para pertenecier a una especie (//TODO: Variaci�n din�mica de este par�m)
    unsigned especieCercana=0;
    unsigned i=0;
    float minDist=99999999;
    float tmpDist=0;
    // busca especie m�s cercana entre los representantes (como Funcion especieMinDist)
    for (i=0; i<conf->numEspecies; i++)
    {
        tmpDist=calcularDist(indexpob,conf->representantes[i],c1,c2,c3,eG_t,conf);
        if (tmpDist<minDist)
        {
            minDist=tmpDist;
            especieCercana=i;
        }
    }
    //Si la distancia entre la especie m�s cercana y el indexpob es menor que el threshold, retorna el n�mero de la especie (i)
    if (minDist<(conf->threshold+(((randL(conf))-0.5)*conf->threshold*conf->maxDesvThEspecies)))
        return(especieCercana);
    else
    {
        //Si es mayor que el threshold, crea una nueva especie
        ////TODO: Eliminar especies que lleven varias generaciones sin mejorar fitness (Hacer nuevo arreglo de numGeneracsSinCambioEnFitnessconf->representantes).
        //Incrementa el n�mero de especies
        conf->numEspecies++;
        // Si genera nueva especie, incrementa el threshold para mantenerlo en el m�ximo nivel posible.
        conf->threshold*=(1+(2*conf->porcentVarTh));
        //reserva memoria para el nuevo arreglo de especies con tama�o conf->numEspeciessizeof(unsigned  int)
        if((conf->representantes=(unsigned *)realloc(conf->representantes,sizeof(unsigned  int)*conf->numEspecies))==NULL)
        {
            fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\Error 654 en asignarEspecie() llamando a realloc");
            return(UINT_MAX);
        }
        //reserva memoria para el nuevo arreglo de conf->contGeneracSinMejora
        if((conf->contGeneracSinMejora=(unsigned *)realloc(conf->contGeneracSinMejora,sizeof(unsigned  int)*conf->numEspecies))==NULL)
        {
            fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\Error 654 en asignarEspecie() llamando a realloc");
            return(UINT_MAX);
        }
        //coloca al indexpob como representante de la nueva especie
        conf->representantes[conf->numEspecies-1]=indexpob;
        //coloca en 0 el n�mero de generaciones sin mejora para la nueva especie
        conf->contGeneracSinMejora[conf->numEspecies-1]=0;
        //POSIBLE PROBLEMA: se generan demasiadas especies.
        return(conf->numEspecies-1);
    }
    return(1);
}
/*
unsigned evaluarEspecie(unsigned inicializar,unsigned especie,unsigned maxBufferSize,char *fileNameGTDv1, TConfig* conf)  // OPTIMIZADA, NMR
{
// eval�a toda la poblaci�n y deja el valor post fSigma en cada nodo.
// y calcula el fitness basado en el que se va acumulando con cada evaluaci�n de cada genoma.
// Los archivos de entrada y salida deben estar previamente abiertos para lectura binaria br
// retorna 0 si hay error, 1 si ok.
// //TODO: URGENTE PARA FX par�metro repeticiones para pasar los archivos de entrada repetidas veces por las redes neuronales al realizar las evaluaciones.
    unsigned leidosIn=0; // guarda el n�mero de datos leidos del archivo de entrada.
    unsigned mBuffer;
    float* inputs; // arreglo de datos leidos del archivo de entrada
    unsigned menorLeido;
    unsigned tmpSize;
    unsigned i;
    unsigned j;
    unsigned k;
    float* tmpin;
    float* tmpout;
    double pasadas=0;// usado para normalizar el fitness
    hdrGTDv1 header; // usado apra leer el ancabezado de GTDv1
    // Abre archivo GTDv1
    if((conf->fIn=fopen(fileNameGTDv1,"rb"))==NULL)
    {
        fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 57 en funcion evaluarPob() llamando a fopen(%s,\"br\")",fileNameGTDv1);
        return(0);
    }
    // Lee encabezado GTDv1
	if (fread(&header, sizeof(hdrGTDv1), 1, conf->fIn)!=1)
	{
		fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>Error 58 en funci�n evaluarPob() llamando a fread(%s)\n",fileNameGTDv1);
		return(0);
	}
    // calcula cuantos grupos de float debe leer multiplos de nEntradas menores que maxBuffersize
    mBuffer = (unsigned  int)floor((float)maxBufferSize/(float)(header.numEntradas+header.numSalidas));
    // compara los multiplicadores y selecciona el menor como mBuffer
	// selecciona por defecto el de entradas debido a que generalmente se usan m�s entradas que salidas
    // reserva memoria para los arreglos de datos leidos  inputs y outpues
    tmpSize=sizeof(float)*mBuffer*(header.numEntradas+header.numSalidas);
    if (tmpSize<32000*maxBufferSize*4)// verifica que no haya m�s de 32k entradas y salidas especificadas en el archivo de entrada
    {
		if((inputs=malloc(tmpSize))==NULL)
		{
			fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 57 en funcion evaluarEspecie() llamando a malloc(%u)",(unsigned  int)(sizeof(float)*mBuffer*header.numEntradas));
			return(0);
		}
    }
	else
	{
		fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 58 en funcion evaluarEspecie() numEntradas + numSalidas >32k leyendo desde archivo GTD");
		return(0);
	}
	//7am 7328064 avianca pasto maletas# AV986484 y 82 y 83 reporte a nombre de Susana Bastidas
    // coloca los valores de cada neurona en 0 para comenzar la evaluaci�n. si incializar=1;
    if (inicializar == 1)  // no debe ser siempre 1? igual que en evaluarpob?
    {
        for (i=0; i<conf->sizePob; i++)
        {
            if (conf->pob[i].especie == especie )
            {
                // Actualiza los punteros a conecHijo de los nodos del genoma.
                if (actualizarPNodos(i,conf)==0)
                {
                    fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 1150 en fnci�n evaluarPob() llamando a actualizarPNodos()\n");
                    free(inputs);
                    return(0);
                }
                for (j=0; j<conf->pob[i].totalNodos; j++)
                {
                    if (conf->pob[i].nodo[j].nodeFunction!=3)
                    {
                        conf->pob[i].nodo[j].valor=0;
                    }
                }
                conf->pob[i].fitness=0; // Inicializa el acumulador de fitness
            }
        }
    }
    // hace repeticiones deben ser 0 o m�s
    for(k=0; k<=conf->repTrain; k++)
    {
        // eval�a toos los genomas en todas las entradas y salidas
        while(!feof(conf->fIn)) // lee hasta que termine cualquiera de los dos archivos
        {
            if (header.tamRegistros==4)
            {
            	tmpSize=4*(header.numEntradas+header.numSalidas);
            	if (tmpSize<4*32000) leidosIn = (unsigned int)fread(inputs,tmpSize,(size_t)mBuffer,conf->fIn);
				menorLeido=leidosIn;
				for (i=0; i<menorLeido; i++)
				{
				    tmpSize= i*(header.numEntradas+header.numSalidas);
					if (tmpSize<conf->sizePob*32000) tmpin=inputs+(tmpSize);
                    tmpSize=(i*(header.numEntradas+header.numSalidas))+header.numEntradas;
					if (tmpSize<conf->sizePob*32000) tmpout=inputs+(tmpSize);
					pasadas=pasadas+1;
					// eval�a entradas y salidas para todos los genomas HERE!!!!ERROR
					for (j=0; j<conf->sizePob; j++)
					{
						if (conf->pob[j].especie == especie)
						{

							// fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>Ins = %3.3f,%3.3f Out = %3.3f\n",tmpin[0],tmpin[1],tmpout[0]);
							if(evaluarGenoma(j,&(conf->pob[j]),0, tmpin,tmpout,conf)==0) ////TODO: HACER PARAMETRO GLOBAL conf->PRIMERO PARA DIFERENTES APLICACIONES
							{
								fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 59 en funcion evaluarPob() llamando a evaluarGenoma(genoma=%u,iteracion=%u )\n",j,i);
								free(inputs);
								return(0);
							}
						}
                    }
                }
            }
            else //FALTA: implementar manejo para cuendo se usa dato double.
            {
            	fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 60 en funci�n evaluarEspecie() manejo de pesos double a�n no se ha implementado.\n");
            	free(inputs);
            	return(0);
            }
        }
        // env�a al inicio el prt del archivo de entradas y salidas
        rewind(conf->fIn);
    }
    // Calcula los fitness a partir del acumulado de cada genoma
    if (pasadas>0)
	{
        for (i=0; i<conf->sizePob; i++)
        {
            if (conf->pob[i].especie == especie )
            {
                conf->pob[i].fitness/=pasadas;
                conf->pob[i].fitness=1.0-conf->pob[i].fitness;
            }
        }
	}
    // cierra archivos de entrada y salida
    fclose(conf->fIn);
    // libera los punteros usados por los punteros a las conexhijos usados durante eval (deben ser rearmados nuevamente con actualizarPNodos).
    for (i=0; i<conf->sizePob; i++)
    {
        if (conf->pob[i].especie == especie )
        {
            for(j=0; j<conf->pob[i].totalNodos; j++)
            {
                if ((conf->pob[i].nodo[j].conexHijo!=NULL)&&(conf->pob[i].nodo[j].contHijos>0)&&(conf->pob[i].nodo[j].nodeFunction!=3))
                {
                    free(conf->pob[i].nodo[j].conexHijo);
                }
            }
        }
    }
    // libera memoria de las listas inputs y outputs
    if (inputs!=NULL) free(inputs);
    // TODO: hacer Funcion bufferize para leer las entradas y salidas en arreglos globales para que se lean una sola vez del disco durante todo el ciclo ppal
    return(1);
}
*/
unsigned nuevaInnovNodo(unsigned nodoIn, unsigned nodoOut, TConfig* conf)  //OPTIMIZADA
{
//Retorna el n�mero innovaci�n(n�mero de nodo) buscandolo en la lista de innovaciones de Nodo, si no lo encuentr, lo crea.
//Par�metros: nodoIn, nodoOut = innovnum de nodos de conexi�n unsigned  interrumpida para agregar el nuevo nodo.
//RETORNA -1 si hay error, numero de innovaci�n si ok.
//OPTIMIZACI�N: Puede univicarse nuevaInnovNodo y nuevaInnovCon en una sola pasando el puntero a la lista correspondiente como par�metro.
    unsigned i=0;
    TListaInnov* innovNodoIn=NULL;
    TNodoOut* ra_result=NULL;
    ////TODO: VERIFICAR tama�o de conf->listaInnovNodo antes de referenciar nidoIn como index
    if ((nodoIn>=conf->contInnovNodo)||(nodoOut>=conf->contInnovNodo))  //Verifica que los nodos In y Out sean menores que el contInnovIn
    {
        fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\Error 1 en nuevaInnovNodo(%u,%u) : nIn o nOut >maxN (%u)",nodoIn,nodoOut,conf->contInnovNodo);
        return(UINT_MAX);
    }
    if ((innovNodoIn=&(conf->listaInnovNodo[nodoIn]))->numOut==0) //Si es la primeravez que se agrega una innovaci�n para este nodo, la adiciona.
    {
//TODO: probando quitando el free ya que no hay necesidad porque es null
        //if (!=NULL) free((void *)innovNodoIn->nodoOut);
        if((innovNodoIn->nodoOut=(TNodoOut *)calloc(1,(unsigned  int)sizeof(TNodoOut)))==NULL)
        {
            fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 2 en funcion nuevaInnovNodo(%u,%u) llamando a calloc(1,%u)\n",nodoIn,nodoOut,(unsigned  int)sizeof(TNodoOut));
            return(UINT_MAX);
        }
        innovNodoIn->nodoOut[0].nodoOut=nodoOut;
        innovNodoIn->nodoOut[0].innovNum=conf->contInnovNodo;
        innovNodoIn->numOut++;
        // reserva memoria para el nuevo nodo como entrada de la lista de innovcon y nodo.(se debe hacer en las 2)
        if((conf->listaInnovNodo=(TListaInnov*) realloc(conf->listaInnovNodo,(sizeof(TListaInnov)*(conf->contInnovNodo+1))))==NULL)
        {
            fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 2.1 en funcion nuevaInnovNodo(%u,%u) llamando a realloc(%s,%u)\n",nodoIn,nodoOut,innovNodoIn->nodoOut==NULL?"NULL":"iNI->nodoOut",(innovNodoIn->numOut+1)*(unsigned int)sizeof(TNodoOut));
            return(UINT_MAX);
        }
        if((conf->listaInnovCon=(TListaInnov*) realloc(conf->listaInnovCon,(sizeof(TListaInnov)*(conf->contInnovNodo+1))))==NULL)
        {
            fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 2.2 en funcion nuevaInnovNodo(%u,%u) llamando a realloc(%s,%u)\n",nodoIn,nodoOut,innovNodoIn->nodoOut==NULL?"NULL":"iNI->nodoOut",(innovNodoIn->numOut+1)*(unsigned int)sizeof(TNodoOut));
            return(UINT_MAX);
        }
        //inicializa los nuevos elementos de las listas (el index del arreglo para las dos es el mismo)
        conf->listaInnovNodo[conf->contInnovNodo].numOut=0;
        conf->listaInnovNodo[conf->contInnovNodo].nodoOut=NULL;
        conf->listaInnovCon[conf->contInnovNodo].numOut=0;
        conf->listaInnovCon[conf->contInnovNodo].nodoOut=NULL;
        conf->contInnovNodo++;
        return(conf->contInnovNodo-1);
    }
    else //Si no es la primera vez, busca LA innovaci�n en la lista y la retorna si la encuentra
    {
        for (i=0; i<innovNodoIn->numOut; i++)
        {
            if(innovNodoIn->nodoOut[i].nodoOut==nodoOut)
            {
                return(innovNodoIn->nodoOut[i].innovNum);
            }
        }//Si no la encuentra, en la lista, la crea con realloc para obntener m�s memoria para el arreglo de salidas.
        if((ra_result=(TNodoOut *) realloc(innovNodoIn->nodoOut,(sizeof(TNodoOut)*(innovNodoIn->numOut+1))))==NULL)
        {
            fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 3 en funcion nuevaInnovNodo(%u,%u) llamando a realloc(%s,%u)\n",nodoIn,nodoOut,innovNodoIn->nodoOut==NULL?"NULL":"iNI->nodoOut",(innovNodoIn->numOut+1)*(unsigned int)sizeof(TNodoOut));
            return(UINT_MAX);
        }
        innovNodoIn->nodoOut=ra_result;
        innovNodoIn->nodoOut[innovNodoIn->numOut].nodoOut=nodoOut;
        innovNodoIn->nodoOut[innovNodoIn->numOut].innovNum=conf->contInnovNodo;
        innovNodoIn->numOut++;
        // reserva memoria para el nuevo nodo como entrada de la lista de innovcon y nodo.(se debe hacer en las 2)
        if((conf->listaInnovNodo=(TListaInnov*) realloc(conf->listaInnovNodo,(sizeof(TListaInnov)*(conf->contInnovNodo+1))))==NULL)
        {
            fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 2.1 en funcion nuevaInnovNodo(%u,%u) llamando a realloc(%s,%u)\n",nodoIn,nodoOut,innovNodoIn->nodoOut==NULL?"NULL":"iNI->nodoOut",(innovNodoIn->numOut+1)*(unsigned int)sizeof(TNodoOut));
            return(UINT_MAX);
        }
        if((conf->listaInnovCon=(TListaInnov*) realloc(conf->listaInnovCon,(sizeof(TListaInnov)*(conf->contInnovNodo+1))))==NULL)
        {
            fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 2.2 en funcion nuevaInnovNodo(%u,%u) llamando a realloc(%s,%u)\n",nodoIn,nodoOut,innovNodoIn->nodoOut==NULL?"NULL":"iNI->nodoOut",(innovNodoIn->numOut+1)*(unsigned int)sizeof(TNodoOut));
            return(UINT_MAX);
        }
        //inicializa los nuevos elementos de las listas (el index del arreglo para las dos es el mismo)
        conf->listaInnovNodo[conf->contInnovNodo].numOut=0;
        conf->listaInnovNodo[conf->contInnovNodo].nodoOut=NULL;
        conf->listaInnovCon[conf->contInnovNodo].numOut=0;
        conf->listaInnovCon[conf->contInnovNodo].nodoOut=NULL;
        conf->contInnovNodo++;
        return(conf->contInnovNodo-1);
    }

}

unsigned nuevaInnovCon(unsigned nodoIn, unsigned nodoOut, TConfig* conf)  // OPTIMIZADA
{
//Retorna el n�mero de innovaci�n para una conexi�n buscandolo en la lista de innovaciones de conex. si no lo encuentra lo crea.
//retrona -1 si hay error
//Par�metros: nodoIn,nodoOut = nodos (innovNums ) de origen y destino de la conex.
//OPTIMIZACI�N: Puede univicarse nuevaInnovNodo y nuevaInnovCon en una sola pasando el puntero a la lista correspondiente como par�metro.
    unsigned i=0;
    TListaInnov* innovNodoIn=NULL;
    TNodoOut* ra_result=NULL;
    if ((nodoIn>=conf->contInnovNodo)||(nodoOut>=conf->contInnovNodo))  //Verifica que los nodos In y Out sean menores que el contInnovIn
    {
        fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\Error 4 en nuevaInnovCon(%u,%u) : nIn o nOut >maxN (%u)",nodoIn,nodoOut,conf->contInnovNodo);
        return(UINT_MAX);
    }
////TODO: cada vex que se haga malloc o realloc hacer inicializaci�n de los elementos especialmente listas y sus elementos y sublistas.
//y probablemente haciendo realloc de la conf->listaInnovNodo actual a una nueva de tama�o i si i >conf->maxInnovNumConex
    innovNodoIn=&(conf->listaInnovCon[nodoIn]);
    if (innovNodoIn->numOut>0)
    {
        if ((innovNodoIn->nodoOut[innovNodoIn->numOut-1].innovNum>=conf->contInnovCon)||(innovNodoIn->nodoOut[innovNodoIn->numOut-1].innovNum<0))
        {
            fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 59.45 en nuevaInnovCon(%u,%u) innovNum de lastConexIN=%u, maxIC=%u",nodoIn,nodoOut,innovNodoIn->nodoOut[innovNodoIn->numOut-1].innovNum,conf->contInnovCon);
            return(UINT_MAX);
        }
    }
    if (innovNodoIn->numOut==0) //Si es la primeravez que se agrega una innovaci�n para este nodo, la adiciona.
    {
//TODO: probando quitando el free ya que no hay necesidad porque es null
        //if (!=NULL) free((void *)innovNodoIn->nodoOut);
        if((innovNodoIn->nodoOut=(TNodoOut *)calloc(1,(unsigned  int)sizeof(TNodoOut)))==NULL)
        {
            fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 5 en funcion nuevaInnovCon(%u,%u) llamando a calloc(1,%u)",nodoIn,nodoOut,(unsigned  int)sizeof(TNodoOut));
            return(UINT_MAX);
        }
        innovNodoIn->nodoOut[0].nodoOut=nodoOut;
        innovNodoIn->nodoOut[0].innovNum=conf->contInnovCon;
        innovNodoIn->numOut=1;
        conf->contInnovCon++;
        return(conf->contInnovCon-1);
    }
    else //Si no es la primera vez, busca LA innovaci�n en la lista y la retorna si la encuentra
    {

        for (i=0; i<innovNodoIn->numOut; i++)
        {
            //fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nHello9.5 i=%u, numout=%u,nodoIn=%u,nodoOut=%u",i,innovNodoIn->numOut,nodoIn,nodoOut); //Problema en arreglo nodoOut para index 0 parece que es NULL
            if(innovNodoIn->nodoOut[i].nodoOut==nodoOut)
            {
                return(innovNodoIn->nodoOut[i].innovNum);
            }
        }//Si no la encuentra, en la lista, la crea con realloc para obntener m�s memoria para el arreglode salidas.
        if((ra_result=(TNodoOut*)malloc(sizeof(TNodoOut)*(innovNodoIn->numOut+1)))==NULL)
        {
            fclose(conf->logFile);conf->logFile=fopen(conf->fileNameLog,"a+");fprintf(conf->logFile,"<br>\nError 6 en funcion nuevaInnovCon(%u,%u) llamando a realloc(%s,%u)\n",nodoIn,nodoOut,ra_result==NULL? "NULL":"iNI->nodoOut",(innovNodoIn->numOut+1)*(unsigned int)sizeof(TNodoOut));
            return(UINT_MAX);
        }
        ra_result=(TNodoOut*)memcpy(ra_result,innovNodoIn->nodoOut,sizeof(TNodoOut)*innovNodoIn->numOut);
        if (ra_result!=NULL) free(innovNodoIn->nodoOut);
        innovNodoIn->nodoOut=ra_result;
        innovNodoIn->nodoOut[innovNodoIn->numOut].nodoOut=nodoOut;
        innovNodoIn->nodoOut[innovNodoIn->numOut].innovNum=conf->contInnovCon;
        innovNodoIn->numOut++;
        conf->contInnovCon++;
        return(conf->contInnovCon-1);
    }
}
//Mi nombre es exterminio.



